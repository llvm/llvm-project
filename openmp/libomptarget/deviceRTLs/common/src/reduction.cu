#include "hip/hip_runtime.h"
//===---- reduction.cu - GPU OpenMP reduction implementation ----- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of reduction with KMPC interface.
//
//===----------------------------------------------------------------------===//

#include "common/omptarget.h"
#include "common/target_atomic.h"
#include "target_impl.h"

EXTERN
void __kmpc_nvptx_end_reduce(int32_t global_tid) {}

EXTERN
void __kmpc_nvptx_end_reduce_nowait(int32_t global_tid) {}

EXTERN int32_t __kmpc_shuffle_int32(int32_t val, int16_t delta, int16_t size) {
  return __kmpc_impl_shfl_down_sync(__kmpc_impl_all_lanes, val, delta, size);
}

EXTERN int64_t __kmpc_shuffle_int64(int64_t val, int16_t delta, int16_t size) {
   uint32_t lo, hi;
   __kmpc_impl_unpack(val, lo, hi);
   hi = __kmpc_impl_shfl_down_sync(__kmpc_impl_all_lanes, hi, delta, size);
   lo = __kmpc_impl_shfl_down_sync(__kmpc_impl_all_lanes, lo, delta, size);
   return __kmpc_impl_pack(lo, hi);
}

INLINE static void gpu_regular_warp_reduce(void *reduce_data,
                                           kmp_ShuffleReductFctPtr shflFct) {
  for (uint32_t mask = WARPSIZE / 2; mask > 0; mask /= 2) {
    shflFct(reduce_data, /*LaneId - not used= */ 0,
            /*Offset = */ mask, /*AlgoVersion=*/0);
  }
}

INLINE static void gpu_irregular_warp_reduce(void *reduce_data,
                                             kmp_ShuffleReductFctPtr shflFct,
                                             uint32_t size, uint32_t tid) {
  uint32_t curr_size;
  uint32_t mask;
  curr_size = size;
  mask = curr_size / 2;
  while (mask > 0) {
    shflFct(reduce_data, /*LaneId = */ tid, /*Offset=*/mask, /*AlgoVersion=*/1);
    curr_size = (curr_size + 1) / 2;
    mask = curr_size / 2;
  }
}

INLINE static uint32_t
gpu_irregular_simd_reduce(void *reduce_data, kmp_ShuffleReductFctPtr shflFct) {
  uint32_t size, remote_id, physical_lane_id;
  physical_lane_id = GetThreadIdInBlock() % WARPSIZE;
  __kmpc_impl_lanemask_t lanemask_lt = __kmpc_impl_lanemask_lt();
  __kmpc_impl_lanemask_t Liveness = __kmpc_impl_activemask();
  uint32_t logical_lane_id = __kmpc_impl_popc(Liveness & lanemask_lt) * 2;
  __kmpc_impl_lanemask_t lanemask_gt = __kmpc_impl_lanemask_gt();
  do {
    Liveness = __kmpc_impl_activemask();
    remote_id = __kmpc_impl_ffs(Liveness & lanemask_gt);
    size = __kmpc_impl_popc(Liveness);
    logical_lane_id /= 2;
    shflFct(reduce_data, /*LaneId =*/logical_lane_id,
            /*Offset=*/remote_id - 1 - physical_lane_id, /*AlgoVersion=*/2);
  } while (logical_lane_id % 2 == 0 && size > 1);
  return (logical_lane_id == 0);
}

EXTERN
int32_t __kmpc_nvptx_simd_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                        size_t reduce_size, void *reduce_data,
                                        kmp_ShuffleReductFctPtr shflFct,
                                        kmp_InterWarpCopyFctPtr cpyFct) {
  __kmpc_impl_lanemask_t Liveness = __kmpc_impl_activemask();
  if (Liveness == __kmpc_impl_all_lanes) {
    gpu_regular_warp_reduce(reduce_data, shflFct);
    return GetThreadIdInBlock() % WARPSIZE ==
           0; // Result on lane 0 of the simd warp.
  } else {
    return gpu_irregular_simd_reduce(
        reduce_data, shflFct); // Result on the first active lane.
  }
}

INLINE
static int32_t nvptx_parallel_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    bool isSPMDExecutionMode, bool isRuntimeUninitialized) {
  uint32_t BlockThreadId = GetLogicalThreadIdInBlock(isSPMDExecutionMode);
  uint32_t NumThreads = GetNumberOfOmpThreads(isSPMDExecutionMode);
  if (NumThreads == 1)
    return 1;
  /*
   * This reduce function handles reduction within a team. It handles
   * parallel regions in both L1 and L2 parallelism levels. It also
   * supports Generic, SPMD, and NoOMP modes.
   *
   * 1. Reduce within a warp.
   * 2. Warp master copies value to warp 0 via shared memory.
   * 3. Warp 0 reduces to a single value.
   * 4. The reduced value is available in the thread that returns 1.
   */

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = BlockThreadId / WARPSIZE;

  // Volta execution model:
  // For the Generic execution mode a parallel region either has 1 thread and
  // beyond that, always a multiple of 32. For the SPMD execution mode we may
  // have any number of threads.
  if ((NumThreads % WARPSIZE == 0) || (WarpId < WarpsNeeded - 1))
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (NumThreads > 1) // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/NumThreads % WARPSIZE,
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);
  }
  return BlockThreadId == 0;
#else
  __kmpc_impl_lanemask_t Liveness = __kmpc_impl_activemask();
  if (Liveness == __kmpc_impl_all_lanes) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (!(Liveness & (Liveness + 1))) // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__kmpc_impl_popc(Liveness),
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);
  else if (!isRuntimeUninitialized) // Dispersed lanes. Only threads in L2
                                    // parallel region may enter here; return
                                    // early.
    return gpu_irregular_simd_reduce(reduce_data, shflFct);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = BlockThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);

    return BlockThreadId == 0;
  } else if (isRuntimeUninitialized /* Never an L2 parallel region without the OMP runtime */) {
    return BlockThreadId == 0;
  }

  // Get the OMP thread Id. This is different from BlockThreadId in the case of
  // an L2 parallel region.
  return global_tid == 0;
#endif // __CUDA_ARCH__ >= 700
}

EXTERN __attribute__((deprecated)) int32_t __kmpc_nvptx_parallel_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(global_tid, num_vars, reduce_size,
                                      reduce_data, shflFct, cpyFct,
                                      isSPMDMode(), isRuntimeUninitialized());
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_v2(
    kmp_Ident *loc, int32_t global_tid, int32_t num_vars, size_t reduce_size,
    void *reduce_data, kmp_ShuffleReductFctPtr shflFct,
    kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      checkSPMDMode(loc), checkRuntimeUninitialized(loc));
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      /*isSPMDExecutionMode=*/true, /*isRuntimeUninitialized=*/true);
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(
      global_tid, num_vars, reduce_size, reduce_data, shflFct, cpyFct,
      /*isSPMDExecutionMode=*/false, /*isRuntimeUninitialized=*/true);
}

INLINE
static int32_t nvptx_teams_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                         size_t reduce_size, void *reduce_data,
                                         kmp_ShuffleReductFctPtr shflFct,
                                         kmp_InterWarpCopyFctPtr cpyFct,
                                         kmp_CopyToScratchpadFctPtr scratchFct,
                                         kmp_LoadReduceFctPtr ldFct,
                                         bool isSPMDExecutionMode) {
  uint32_t ThreadId = GetLogicalThreadIdInBlock(isSPMDExecutionMode);
  // In non-generic mode all workers participate in the teams reduction.
  // In generic mode only the team master participates in the teams
  // reduction because the workers are waiting for parallel work.
  uint32_t NumThreads =
      isSPMDExecutionMode ? GetNumberOfOmpThreads(/*isSPMDExecutionMode=*/true)
                          : /*Master thread only*/ 1;
  uint32_t TeamId = GetBlockIdInKernel();
  uint32_t NumTeams = GetNumberOfBlocksInKernel();
  static SHARED volatile bool IsLastTeam;

  // Team masters of all teams write to the scratchpad.
  if (ThreadId == 0) {
    unsigned int *timestamp = GetTeamsReductionTimestamp();
    char *scratchpad = GetTeamsReductionScratchpad();

    scratchFct(reduce_data, scratchpad, TeamId, NumTeams);
    __kmpc_impl_threadfence();

    // atomicInc increments 'timestamp' and has a range [0, NumTeams-1].
    // It resets 'timestamp' back to 0 once the last team increments
    // this counter.
    unsigned val = __kmpc_atomic_inc(timestamp, NumTeams - 1);
    IsLastTeam = val == NumTeams - 1;
  }

  // We have to wait on L1 barrier because in GENERIC mode the workers
  // are waiting on barrier 0 for work.
  //
  // If we guard this barrier as follows it leads to deadlock, probably
  // because of a compiler bug: if (!IsGenericMode()) __syncthreads();
  uint16_t SyncWarps = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  __kmpc_impl_named_sync(L1_BARRIER, SyncWarps * WARPSIZE);

  // If this team is not the last, quit.
  if (/* Volatile read by all threads */ !IsLastTeam)
    return 0;

    //
    // Last team processing.
    //

    // Threads in excess of #teams do not participate in reduction of the
    // scratchpad values.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t ActiveThreads = NumThreads;
  if (NumTeams < NumThreads) {
    ActiveThreads =
        (NumTeams < WARPSIZE) ? 1 : NumTeams & ~((uint16_t)WARPSIZE - 1);
  }
  if (ThreadId >= ActiveThreads)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = ActiveThreads + ThreadId; i < NumTeams; i += ActiveThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = ThreadId / WARPSIZE;

  // Reduce across warps to the warp master.
  if ((ActiveThreads % WARPSIZE == 0) ||
      (WarpId < WarpsNeeded - 1)) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (ActiveThreads > 1) // Partial warp but contiguous lanes
    // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/ActiveThreads % WARPSIZE,
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  if (ActiveThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#else
  if (ThreadId >= NumTeams)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = NumThreads + ThreadId; i < NumTeams; i += NumThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  // Reduce across warps to the warp master.
  __kmpc_impl_lanemask_t Liveness = __kmpc_impl_activemask();
  if (Liveness == __kmpc_impl_all_lanes) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__kmpc_impl_popc(Liveness),
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  uint32_t ActiveThreads = NumTeams < NumThreads ? NumTeams : NumThreads;
  if (ActiveThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = ThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#endif // __CUDA_ARCH__ >= 700

  return ThreadId == 0;
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                         size_t reduce_size, void *reduce_data,
                                         kmp_ShuffleReductFctPtr shflFct,
                                         kmp_InterWarpCopyFctPtr cpyFct,
                                         kmp_CopyToScratchpadFctPtr scratchFct,
                                         kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, isSPMDMode());
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, /*isSPMDExecutionMode=*/true);
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, /*isSPMDExecutionMode=*/false);
}

EXTERN int32_t __kmpc_nvptx_teams_reduce_nowait_simple(kmp_Ident *loc,
                                                       int32_t global_tid,
                                                       kmp_CriticalName *crit) {
  if (checkSPMDMode(loc) && GetThreadIdInBlock() != 0)
    return 0;
  // The master thread of the team actually does the reduction.
  while (__kmpc_atomic_cas((uint32_t *)crit, 0u, 1u))
    ;
  return 1;
}

EXTERN void
__kmpc_nvptx_teams_end_reduce_nowait_simple(kmp_Ident *loc, int32_t global_tid,
                                            kmp_CriticalName *crit) {
  __kmpc_impl_threadfence_system();
  (void)__kmpc_atomic_exchange((uint32_t *)crit, 0u);
}

INLINE static bool isMaster(kmp_Ident *loc, uint32_t ThreadId) {
  return checkGenericMode(loc) || IsTeamMaster(ThreadId);
}

INLINE static uint32_t roundToWarpsize(uint32_t s) {
  if (s < WARPSIZE)
    return 1;
  return (s & ~(unsigned)(WARPSIZE - 1));
}

DEVICE static volatile uint32_t IterCnt = 0;
DEVICE static volatile uint32_t Cnt = 0;
EXTERN int32_t __kmpc_nvptx_teams_reduce_nowait_v2(
    kmp_Ident *loc, int32_t global_tid, void *global_buffer,
    int32_t num_of_records, void *reduce_data, kmp_ShuffleReductFctPtr shflFct,
    kmp_InterWarpCopyFctPtr cpyFct, kmp_ListGlobalFctPtr lgcpyFct,
    kmp_ListGlobalFctPtr lgredFct, kmp_ListGlobalFctPtr glcpyFct,
    kmp_ListGlobalFctPtr glredFct) {

  // Terminate all threads in non-SPMD mode except for the master thread.
  if (checkGenericMode(loc) && GetThreadIdInBlock() != GetMasterThreadID())
    return 0;

  uint32_t ThreadId = GetLogicalThreadIdInBlock(checkSPMDMode(loc));

  // In non-generic mode all workers participate in the teams reduction.
  // In generic mode only the team master participates in the teams
  // reduction because the workers are waiting for parallel work.
  uint32_t NumThreads =
      checkSPMDMode(loc) ? GetNumberOfOmpThreads(/*isSPMDExecutionMode=*/true)
                         : /*Master thread only*/ 1;
  uint32_t TeamId = GetBlockIdInKernel();
  uint32_t NumTeams = GetNumberOfBlocksInKernel();
  static SHARED unsigned Bound;
  static SHARED unsigned ChunkTeamCount;

  // Block progress for teams greater than the current upper
  // limit. We always only allow a number of teams less or equal
  // to the number of slots in the buffer.
  bool IsMaster = isMaster(loc, ThreadId);
  while (IsMaster) {
    // Atomic read
    Bound = __kmpc_atomic_add((uint32_t *)&IterCnt, 0u);
    if (TeamId < Bound + num_of_records)
      break;
  }

  if (IsMaster) {
    int ModBockId = TeamId % num_of_records;
    if (TeamId < num_of_records)
      lgcpyFct(global_buffer, ModBockId, reduce_data);
    else
      lgredFct(global_buffer, ModBockId, reduce_data);
    __kmpc_impl_threadfence_system();

    // Increment team counter.
    // This counter is incremented by all teams in the current
    // BUFFER_SIZE chunk.
    ChunkTeamCount = __kmpc_atomic_inc((uint32_t *)&Cnt, num_of_records - 1u);
  }
  // Synchronize
  if (checkSPMDMode(loc))
    __kmpc_barrier(loc, global_tid);

  // reduce_data is global or shared so before being reduced within the
  // warp we need to bring it in local memory:
  // local_reduce_data = reduce_data[i]
  //
  // Example for 3 reduction variables a, b, c (of potentially different
  // types):
  //
  // buffer layout (struct of arrays):
  // a, a, ..., a, b, b, ... b, c, c, ... c
  // |__________|
  //     num_of_records
  //
  // local_data_reduce layout (struct):
  // a, b, c
  //
  // Each thread will have a local struct containing the values to be
  // reduced:
  //      1. do reduction within each warp.
  //      2. do reduction across warps.
  //      3. write the final result to the main reduction variable
  //         by returning 1 in the thread holding the reduction result.

  // Check if this is the very last team.
  unsigned NumRecs = __kmpc_impl_min(NumTeams, uint32_t(num_of_records));
  if (ChunkTeamCount == NumTeams - Bound - 1) {
    //
    // Last team processing.
    //
    if (ThreadId >= NumRecs)
      return 0;
    NumThreads = roundToWarpsize(__kmpc_impl_min(NumThreads, NumRecs));
    if (ThreadId >= NumThreads)
      return 0;

    // Load from buffer and reduce.
    glcpyFct(global_buffer, ThreadId, reduce_data);
    for (uint32_t i = NumThreads + ThreadId; i < NumRecs; i += NumThreads)
      glredFct(global_buffer, i, reduce_data);

    // Reduce across warps to the warp master.
    if (NumThreads > 1) {
      gpu_regular_warp_reduce(reduce_data, shflFct);

      // When we have more than [warpsize] number of threads
      // a block reduction is performed here.
      uint32_t ActiveThreads = __kmpc_impl_min(NumRecs, NumThreads);
      if (ActiveThreads > WARPSIZE) {
        uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
        // Gather all the reduced values from each warp
        // to the first warp.
        cpyFct(reduce_data, WarpsNeeded);

        uint32_t WarpId = ThreadId / WARPSIZE;
        if (WarpId == 0)
          gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                    ThreadId);
      }
    }

    if (IsMaster) {
      Cnt = 0;
      IterCnt = 0;
      return 1;
    }
    return 0;
  }
  if (IsMaster && ChunkTeamCount == num_of_records - 1) {
    // Allow SIZE number of teams to proceed writing their
    // intermediate results to the global buffer.
    __kmpc_atomic_add((uint32_t *)&IterCnt, uint32_t(num_of_records));
  }

  return 0;
}

