//===--- omptarget.cu - OpenMP GPU initialization ---------------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//

#include "common/omptarget.h"
#include "target_impl.h"

////////////////////////////////////////////////////////////////////////////////
// global data tables
////////////////////////////////////////////////////////////////////////////////

extern DEVICE
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime) {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f\n",
        OMPTARGET_NVPTX_VERSION);
  ASSERT0(LT_FUSSY, RequiresOMPRuntime,
          "Generic always requires initialized runtime.");
  setExecutionParameters(Generic, RuntimeInitialized);
  for (int I = 0; I < MAX_THREADS_PER_TEAM / WARPSIZE; ++I)
    parallelLevel[I] = 0;

  int threadIdInBlock = GetThreadIdInBlock();
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = __kmpc_impl_smid() % MAX_SM;
  usedSlotIdx = slot;
  omptarget_nvptx_threadPrivateContext =
      omptarget_nvptx_device_State[slot].Dequeue();

  // init thread private
  int threadId = GetLogicalThreadIdInBlock(/*isSPMDExecutionMode=*/false);
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr();
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(threadId);
  nThreads = GetNumberOfThreadsInBlock();
  threadLimit = ThreadLimit;
}

EXTERN void __kmpc_kernel_deinit(int16_t IsOMPRuntimeInitialized) {
  PRINT0(LD_IO, "call to __kmpc_kernel_deinit\n");
  ASSERT0(LT_FUSSY, IsOMPRuntimeInitialized,
          "Generic always requires initialized runtime.");
  // Enqueue omp state object for use by another team.
  int slot = usedSlotIdx;
  omptarget_nvptx_device_State[slot].Enqueue(
      omptarget_nvptx_threadPrivateContext);
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
}

EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime,
                                    int16_t RequiresDataSharing) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  setExecutionParameters(Spmd, RequiresOMPRuntime ? RuntimeInitialized
                                                  : RuntimeUninitialized);
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    usedSlotIdx = __kmpc_impl_smid() % MAX_SM;
    parallelLevel[0] =
        1 + (GetNumberOfThreadsInBlock() > 1 ? OMP_ACTIVE_PARALLEL_LEVEL : 0);
  } else if (GetLaneId() == 0) {
    parallelLevel[GetWarpId()] =
        1 + (GetNumberOfThreadsInBlock() > 1 ? OMP_ACTIVE_PARALLEL_LEVEL : 0);
  }
  if (!RequiresOMPRuntime) {
    // Runtime is not required - exit.
    __kmpc_impl_syncthreads();
    return;
  }

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  if (threadId == 0) {
    // Get a state object from the queue.
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[usedSlotIdx].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
  }
  __kmpc_impl_syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(currTeamDescr.LevelZeroTaskDescr());
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        (int)newTaskDescr->ThreadId(), (int)ThreadLimit);

  if (RequiresDataSharing && GetLaneId() == 0) {
    // Warp master initializes data sharing environment.
    unsigned WID = threadId / WARPSIZE;
    __kmpc_data_sharing_slot *RootS = currTeamDescr.RootS(
        WID, WID == WARPSIZE - 1);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

EXTERN void __kmpc_spmd_kernel_deinit_v2(int16_t RequiresOMPRuntime) {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  if (!RequiresOMPRuntime)
    return;

  __kmpc_impl_syncthreads();
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Enqueue omp state object for use by another team.
    int slot = usedSlotIdx;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}

// Return true if the current target region is executed in SPMD mode.
EXTERN int8_t __kmpc_is_spmd_exec_mode() {
  PRINT0(LD_IO | LD_PAR, "call to __kmpc_is_spmd_exec_mode\n");
  return isSPMDMode();
}
