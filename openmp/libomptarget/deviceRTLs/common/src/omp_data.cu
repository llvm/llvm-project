//===------------ omp_data.cu - OpenMP GPU objects --------------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the data objects used on the GPU device.
//
//===----------------------------------------------------------------------===//
#pragma omp declare target

#include "common/omptarget.h"
#include "common/device_environment.h"

////////////////////////////////////////////////////////////////////////////////
// global device environment
////////////////////////////////////////////////////////////////////////////////

DEVICE omptarget_device_environmentTy omptarget_device_environment;

////////////////////////////////////////////////////////////////////////////////
// global data holding OpenMP state information
////////////////////////////////////////////////////////////////////////////////

DEVICE
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

DEVICE omptarget_nvptx_SimpleMemoryManager
    omptarget_nvptx_simpleMemoryManager;
DEVICE SHARED uint32_t usedMemIdx;
DEVICE SHARED uint32_t usedSlotIdx;

DEVICE SHARED uint8_t parallelLevel[MAX_THREADS_PER_TEAM / WARPSIZE];
DEVICE SHARED uint16_t threadLimit;
DEVICE SHARED uint16_t threadsInTeam;
DEVICE SHARED uint16_t nThreads;
// Pointer to this team's OpenMP state object
DEVICE SHARED
    omptarget_nvptx_ThreadPrivateContext *omptarget_nvptx_threadPrivateContext;

////////////////////////////////////////////////////////////////////////////////
// The team master sets the outlined parallel function in this variable to
// communicate with the workers.  Since it is in shared memory, there is one
// copy of these variables for each kernel, instance, and team.
////////////////////////////////////////////////////////////////////////////////
volatile DEVICE SHARED omptarget_nvptx_WorkFn omptarget_nvptx_workFn;

////////////////////////////////////////////////////////////////////////////////
// OpenMP kernel execution parameters
////////////////////////////////////////////////////////////////////////////////
DEVICE SHARED uint32_t execution_param;

////////////////////////////////////////////////////////////////////////////////
// Data sharing state
////////////////////////////////////////////////////////////////////////////////
DEVICE SHARED DataSharingStateTy DataSharingState;

////////////////////////////////////////////////////////////////////////////////
// Scratchpad for teams reduction.
////////////////////////////////////////////////////////////////////////////////
DEVICE SHARED void *ReductionScratchpadPtr;

////////////////////////////////////////////////////////////////////////////////
// Data sharing related variables.
////////////////////////////////////////////////////////////////////////////////
DEVICE SHARED omptarget_nvptx_SharedArgs omptarget_nvptx_globalArgs;

#pragma omp end declare target
