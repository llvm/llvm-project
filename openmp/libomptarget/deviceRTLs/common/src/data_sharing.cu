//===----- data_sharing.cu - OpenMP GPU data sharing ------------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of data sharing environments
//
//===----------------------------------------------------------------------===//
#include "common/omptarget.h"
#include "target_impl.h"

// Return true if this is the master thread.
INLINE static bool IsMasterThread(bool isSPMDExecutionMode) {
  return !isSPMDExecutionMode && GetMasterThreadID() == GetThreadIdInBlock();
}

/// Return the provided size aligned to the size of a pointer.
INLINE static size_t AlignVal(size_t Val) {
  const size_t Align = (size_t)sizeof(void *);
  if (Val & (Align - 1)) {
    Val += Align;
    Val &= ~(Align - 1);
  }
  return Val;
}

#define DSFLAG 0
#define DSFLAG_INIT 0
#define DSPRINT(_flag, _str, _args...)                                         \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x, _args);*/          \
    }                                                                          \
  }
#define DSPRINT0(_flag, _str)                                                  \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x);*/                 \
    }                                                                          \
  }

// Initialize the shared data structures. This is expected to be called for the
// master thread and warp masters. \param RootS: A pointer to the root of the
// data sharing stack. \param InitialDataSize: The initial size of the data in
// the slot.
EXTERN void
__kmpc_initialize_data_sharing_environment(__kmpc_data_sharing_slot *rootS,
                                           size_t InitialDataSize) {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Runtime must be initialized.");
  DSPRINT0(DSFLAG_INIT,
           "Entering __kmpc_initialize_data_sharing_environment\n");

  unsigned WID = GetWarpId();
  DSPRINT(DSFLAG_INIT, "Warp ID: %u\n", WID);

  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();
  __kmpc_data_sharing_slot *RootS =
      teamDescr->RootS(WID, IsMasterThread(isSPMDMode()));

  DataSharingState.SlotPtr[WID] = RootS;
  DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];

  // We don't need to initialize the frame and active threads.

  DSPRINT(DSFLAG_INIT, "Initial data size: %08x \n", (unsigned)InitialDataSize);
  DSPRINT(DSFLAG_INIT, "Root slot at: %016llx \n", (unsigned long long)RootS);
  DSPRINT(DSFLAG_INIT, "Root slot data-end at: %016llx \n",
          (unsigned long long)RootS->DataEnd);
  DSPRINT(DSFLAG_INIT, "Root slot next at: %016llx \n",
          (unsigned long long)RootS->Next);
  DSPRINT(DSFLAG_INIT, "Shared slot ptr at: %016llx \n",
          (unsigned long long)DataSharingState.SlotPtr[WID]);
  DSPRINT(DSFLAG_INIT, "Shared stack ptr at: %016llx \n",
          (unsigned long long)DataSharingState.StackPtr[WID]);

  DSPRINT0(DSFLAG_INIT, "Exiting __kmpc_initialize_data_sharing_environment\n");
}

EXTERN void *__kmpc_data_sharing_environment_begin(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, __kmpc_impl_lanemask_t *SavedActiveThreads,
    size_t SharingDataSize, size_t SharingDefaultDataSize,
    int16_t IsOMPRuntimeInitialized) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_begin\n");

  // If the runtime has been elided, used shared memory for master-worker
  // data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  DSPRINT(DSFLAG, "Data Size %016llx\n", (unsigned long long)SharingDataSize);
  DSPRINT(DSFLAG, "Default Data Size %016llx\n",
          (unsigned long long)SharingDefaultDataSize);

  unsigned WID = GetWarpId();
  __kmpc_impl_lanemask_t CurActiveThreads = __kmpc_impl_activemask();

  __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
  void *&StackP = DataSharingState.StackPtr[WID];
  void * volatile &FrameP = DataSharingState.FramePtr[WID];
  __kmpc_impl_lanemask_t &ActiveT = DataSharingState.ActiveThreads[WID];

  DSPRINT0(DSFLAG, "Save current slot/stack values.\n");
  // Save the current values.
  *SavedSharedSlot = SlotP;
  *SavedSharedStack = StackP;
  *SavedSharedFrame = FrameP;
  *SavedActiveThreads = ActiveT;

  DSPRINT(DSFLAG, "Warp ID: %u\n", WID);
  DSPRINT(DSFLAG, "Saved slot ptr at: %016llx \n", (unsigned long long)SlotP);
  DSPRINT(DSFLAG, "Saved stack ptr at: %016llx \n", (unsigned long long)StackP);
  DSPRINT(DSFLAG, "Saved frame ptr at: %016llx \n", (long long)FrameP);
  DSPRINT(DSFLAG, "Active threads: %08x \n", (unsigned)ActiveT);

  // Only the warp active master needs to grow the stack.
  if (__kmpc_impl_is_first_active_thread()) {
    // Save the current active threads.
    ActiveT = CurActiveThreads;

    // Make sure we use aligned sizes to avoid rematerialization of data.
    SharingDataSize = AlignVal(SharingDataSize);
    // FIXME: The default data size can be assumed to be aligned?
    SharingDefaultDataSize = AlignVal(SharingDefaultDataSize);

    // Check if we have room for the data in the current slot.
    const uintptr_t CurrentStartAddress = (uintptr_t)StackP;
    const uintptr_t CurrentEndAddress = (uintptr_t)SlotP->DataEnd;
    const uintptr_t RequiredEndAddress =
        CurrentStartAddress + (uintptr_t)SharingDataSize;

    DSPRINT(DSFLAG, "Data Size %016llx\n", (unsigned long long)SharingDataSize);
    DSPRINT(DSFLAG, "Default Data Size %016llx\n",
            (unsigned long long)SharingDefaultDataSize);
    DSPRINT(DSFLAG, "Current Start Address %016llx\n",
            (unsigned long long)CurrentStartAddress);
    DSPRINT(DSFLAG, "Current End Address %016llx\n",
            (unsigned long long)CurrentEndAddress);
    DSPRINT(DSFLAG, "Required End Address %016llx\n",
            (unsigned long long)RequiredEndAddress);
    DSPRINT(DSFLAG, "Active Threads %08x\n", (unsigned)ActiveT);

    // If we require a new slot, allocate it and initialize it (or attempt to
    // reuse one). Also, set the shared stack and slot pointers to the new
    // place. If we do not need to grow the stack, just adapt the stack and
    // frame pointers.
    if (CurrentEndAddress < RequiredEndAddress) {
      size_t NewSize = (SharingDataSize > SharingDefaultDataSize)
                           ? SharingDataSize
                           : SharingDefaultDataSize;
      __kmpc_data_sharing_slot *NewSlot = 0;

      // Attempt to reuse an existing slot.
      if (__kmpc_data_sharing_slot *ExistingSlot = SlotP->Next) {
        uintptr_t ExistingSlotSize = (uintptr_t)ExistingSlot->DataEnd -
                                     (uintptr_t)(&ExistingSlot->Data[0]);
        if (ExistingSlotSize >= NewSize) {
          DSPRINT(DSFLAG, "Reusing stack slot %016llx\n",
                  (unsigned long long)ExistingSlot);
          NewSlot = ExistingSlot;
        } else {
          DSPRINT(DSFLAG, "Cleaning up -failed reuse - %016llx\n",
                  (unsigned long long)SlotP->Next);
          SafeFree(ExistingSlot, "Failed reuse");
        }
      }

      if (!NewSlot) {
        NewSlot = (__kmpc_data_sharing_slot *)SafeMalloc(
            sizeof(__kmpc_data_sharing_slot) + NewSize,
            "Warp master slot allocation");
        DSPRINT(DSFLAG, "New slot allocated %016llx (data size=%016llx)\n",
                (unsigned long long)NewSlot, NewSize);
      }

      NewSlot->Next = 0;
      NewSlot->DataEnd = &NewSlot->Data[NewSize];

      SlotP->Next = NewSlot;
      SlotP = NewSlot;
      StackP = &NewSlot->Data[SharingDataSize];
      FrameP = &NewSlot->Data[0];
    } else {

      // Clean up any old slot that we may still have. The slot producers, do
      // not eliminate them because that may be used to return data.
      if (SlotP->Next) {
        DSPRINT(DSFLAG, "Cleaning up - old not required - %016llx\n",
                (unsigned long long)SlotP->Next);
        SafeFree(SlotP->Next, "Old slot not required");
        SlotP->Next = 0;
      }

      FrameP = StackP;
      StackP = (void *)RequiredEndAddress;
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __kmpc_impl_threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_begin\n");

  // All the threads in this warp get the frame they should work with.
  return FrameP;
}

EXTERN void __kmpc_data_sharing_environment_end(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, __kmpc_impl_lanemask_t *SavedActiveThreads,
    int32_t IsEntryPoint) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_end\n");

  unsigned WID = GetWarpId();

  if (IsEntryPoint) {
    if (__kmpc_impl_is_first_active_thread()) {
      DSPRINT0(DSFLAG, "Doing clean up\n");

      // The master thread cleans the saved slot, because this is an environment
      // only for the master.
      __kmpc_data_sharing_slot *S = IsMasterThread(isSPMDMode())
                                        ? *SavedSharedSlot
                                        : DataSharingState.SlotPtr[WID];

      if (S->Next) {
        SafeFree(S->Next, "Sharing environment end");
        S->Next = 0;
      }
    }

    DSPRINT0(DSFLAG, "Exiting Exiting __kmpc_data_sharing_environment_end\n");
    return;
  }

  __kmpc_impl_lanemask_t CurActive = __kmpc_impl_activemask();

  // Only the warp master can restore the stack and frame information, and only
  // if there are no other threads left behind in this environment (i.e. the
  // warp diverged and returns in different places). This only works if we
  // assume that threads will converge right after the call site that started
  // the environment.
  if (__kmpc_impl_is_first_active_thread()) {
    __kmpc_impl_lanemask_t &ActiveT = DataSharingState.ActiveThreads[WID];

    DSPRINT0(DSFLAG, "Before restoring the stack\n");
    // Zero the bits in the mask. If it is still different from zero, then we
    // have other threads that will return after the current ones.
    ActiveT &= ~CurActive;

    DSPRINT(DSFLAG, "Active threads: %08x; New mask: %08x\n",
            (unsigned)CurActive, (unsigned)ActiveT);

    if (!ActiveT) {
      // No other active threads? Great, lets restore the stack.

      __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
      void *&StackP = DataSharingState.StackPtr[WID];
      void * volatile &FrameP = DataSharingState.FramePtr[WID];

      SlotP = *SavedSharedSlot;
      StackP = *SavedSharedStack;
      FrameP = *SavedSharedFrame;
      ActiveT = *SavedActiveThreads;

      DSPRINT(DSFLAG, "Restored slot ptr at: %016llx \n",
              (unsigned long long)SlotP);
      DSPRINT(DSFLAG, "Restored stack ptr at: %016llx \n",
              (unsigned long long)StackP);
      DSPRINT(DSFLAG, "Restored frame ptr at: %016llx \n",
              (unsigned long long)FrameP);
      DSPRINT(DSFLAG, "Active threads: %08x \n", (unsigned)ActiveT);
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __kmpc_impl_threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_end\n");
  return;
}

EXTERN void *
__kmpc_get_data_sharing_environment_frame(int32_t SourceThreadID,
                                          int16_t IsOMPRuntimeInitialized) {
  DSPRINT0(DSFLAG, "Entering __kmpc_get_data_sharing_environment_frame\n");

  // If the runtime has been elided, use shared memory for master-worker
  // data sharing.  We're reusing the statically allocated data structure
  // that is used for standard data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  // Get the frame used by the requested thread.

  unsigned SourceWID = SourceThreadID / WARPSIZE;

  DSPRINT(DSFLAG, "Source  warp: %u\n", SourceWID);

  void * volatile P = DataSharingState.FramePtr[SourceWID];
  DSPRINT0(DSFLAG, "Exiting __kmpc_get_data_sharing_environment_frame\n");
  return P;
}

////////////////////////////////////////////////////////////////////////////////
// Runtime functions for trunk data sharing scheme.
////////////////////////////////////////////////////////////////////////////////

INLINE static void data_sharing_init_stack_common() {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Runtime must be initialized.");
  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();

  for (int WID = 0; WID < WARPSIZE; WID++) {
    __kmpc_data_sharing_slot *RootS = teamDescr->GetPreallocatedSlotAddr(WID);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

// Initialize data sharing data structure. This function needs to be called
// once at the beginning of a data sharing context (coincides with the kernel
// initialization). This function is called only by the MASTER thread of each
// team in non-SPMD mode.
EXTERN void __kmpc_data_sharing_init_stack() {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Runtime must be initialized.");
  // This function initializes the stack pointer with the pointer to the
  // statically allocated shared memory slots. The size of a shared memory
  // slot is pre-determined to be 256 bytes.
  data_sharing_init_stack_common();
  omptarget_nvptx_globalArgs.Init();
}

// Initialize data sharing data structure. This function needs to be called
// once at the beginning of a data sharing context (coincides with the kernel
// initialization). This function is called in SPMD mode only.
EXTERN void __kmpc_data_sharing_init_stack_spmd() {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Runtime must be initialized.");
  // This function initializes the stack pointer with the pointer to the
  // statically allocated shared memory slots. The size of a shared memory
  // slot is pre-determined to be 256 bytes.
  if (GetThreadIdInBlock() == 0)
    data_sharing_init_stack_common();

  __kmpc_impl_threadfence_block();
}

INLINE static void* data_sharing_push_stack_common(size_t PushSize) {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Expected initialized runtime.");

  // Only warp active master threads manage the stack.
  bool IsWarpMaster = (GetThreadIdInBlock() % WARPSIZE) == 0;

  // Add worst-case padding to DataSize so that future stack allocations are
  // correctly aligned.
  const size_t Alignment = 8;
  PushSize = (PushSize + (Alignment - 1)) / Alignment * Alignment;

  // Frame pointer must be visible to all workers in the same warp.
  const unsigned WID = GetWarpId();
  void *FrameP = 0;
  __kmpc_impl_lanemask_t CurActive = __kmpc_impl_activemask();

  if (IsWarpMaster) {
    // SlotP will point to either the shared memory slot or an existing
    // global memory slot.
    __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
    void *&StackP = DataSharingState.StackPtr[WID];

    // Check if we have room for the data in the current slot.
    const uintptr_t StartAddress = (uintptr_t)StackP;
    const uintptr_t EndAddress = (uintptr_t)SlotP->DataEnd;
    const uintptr_t RequestedEndAddress = StartAddress + (uintptr_t)PushSize;

    // If we requested more data than there is room for in the rest
    // of the slot then we need to either re-use the next slot, if one exists,
    // or create a new slot.
    if (EndAddress < RequestedEndAddress) {
      __kmpc_data_sharing_slot *NewSlot = 0;
      size_t NewSize = PushSize;

      // Allocate at least the default size for each type of slot.
      // Master is a special case and even though there is only one thread,
      // it can share more things with the workers. For uniformity, it uses
      // the full size of a worker warp slot.
      size_t DefaultSlotSize = DS_Worker_Warp_Slot_Size;
      if (DefaultSlotSize > NewSize)
        NewSize = DefaultSlotSize;
      NewSlot = (__kmpc_data_sharing_slot *) SafeMalloc(
          sizeof(__kmpc_data_sharing_slot) + NewSize,
          "Global memory slot allocation.");

      NewSlot->Next = 0;
      NewSlot->Prev = SlotP;
      NewSlot->PrevSlotStackPtr = StackP;
      NewSlot->DataEnd = &NewSlot->Data[0] + NewSize;

      // Make previous slot point to the newly allocated slot.
      SlotP->Next = NewSlot;
      // The current slot becomes the new slot.
      SlotP = NewSlot;
      // The stack pointer always points to the next free stack frame.
      StackP = &NewSlot->Data[0] + PushSize;
      // The frame pointer always points to the beginning of the frame.
      FrameP = DataSharingState.FramePtr[WID] = &NewSlot->Data[0];
    } else {
      // Add the data chunk to the current slot. The frame pointer is set to
      // point to the start of the new frame held in StackP.
      FrameP = DataSharingState.FramePtr[WID] = StackP;
      // Reset stack pointer to the requested address.
      StackP = (void *)RequestedEndAddress;
    }
  }
  // Get address from lane 0.
  int *FP = (int *)&FrameP;
  FP[0] = __kmpc_impl_shfl_sync(CurActive, FP[0], 0);
  if (sizeof(FrameP) == 8)
    FP[1] = __kmpc_impl_shfl_sync(CurActive, FP[1], 0);

  return FrameP;
}

EXTERN void *__kmpc_data_sharing_coalesced_push_stack(size_t DataSize,
                                                      int16_t UseSharedMemory) {
  return data_sharing_push_stack_common(DataSize);
}

// Called at the time of the kernel initialization. This is used to initilize
// the list of references to shared variables and to pre-allocate global storage
// for holding the globalized variables.
//
// By default the globalized variables are stored in global memory. If the
// UseSharedMemory is set to true, the runtime will attempt to use shared memory
// as long as the size requested fits the pre-allocated size.
EXTERN void *__kmpc_data_sharing_push_stack(size_t DataSize,
                                            int16_t UseSharedMemory) {
  // Compute the total memory footprint of the requested data.
  // The master thread requires a stack only for itself. A worker
  // thread (which at this point is a warp master) will require
  // space for the variables of each thread in the warp,
  // i.e. one DataSize chunk per warp lane.
  // TODO: change WARPSIZE to the number of active threads in the warp.
  size_t PushSize = (isRuntimeUninitialized() || IsMasterThread(isSPMDMode()))
                        ? DataSize
                        : WARPSIZE * DataSize;

  // Compute the start address of the frame of each thread in the warp.
  uintptr_t FrameStartAddress =
      (uintptr_t) data_sharing_push_stack_common(PushSize);
  FrameStartAddress += (uintptr_t) (GetLaneId() * DataSize);
  return (void *)FrameStartAddress;
}

// Pop the stack and free any memory which can be reclaimed.
//
// When the pop operation removes the last global memory slot,
// reclaim all outstanding global memory slots since it is
// likely we have reached the end of the kernel.
EXTERN void __kmpc_data_sharing_pop_stack(void *FrameStart) {
  ASSERT0(LT_FUSSY, isRuntimeInitialized(), "Expected initialized runtime.");

  __kmpc_impl_threadfence_block();

  if (GetThreadIdInBlock() % WARPSIZE == 0) {
    unsigned WID = GetWarpId();

    // Current slot
    __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];

    // Pointer to next available stack.
    void *&StackP = DataSharingState.StackPtr[WID];

    // Pop the frame.
    StackP = FrameStart;

    // If the current slot is empty, we need to free the slot after the
    // pop.
    bool SlotEmpty = (StackP == &SlotP->Data[0]);

    if (SlotEmpty && SlotP->Prev) {
      // Before removing the slot we need to reset StackP.
      StackP = SlotP->PrevSlotStackPtr;

      // Remove the slot.
      SlotP = SlotP->Prev;
      SafeFree(SlotP->Next, "Free slot.");
      SlotP->Next = 0;
    }
  }
}

// Begin a data sharing context. Maintain a list of references to shared
// variables. This list of references to shared variables will be passed
// to one or more threads.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_begin_sharing_variables(void ***GlobalArgs, size_t nArgs) {
  omptarget_nvptx_globalArgs.EnsureSize(nArgs);
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}

// End a data sharing context. There is no need to have a list of refs
// to shared variables because the context in which those variables were
// shared has now ended. This should clean-up the list of references only
// without affecting the actual global storage of the variables.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_end_sharing_variables() {
  omptarget_nvptx_globalArgs.DeInit();
}

// This function will return a list of references to global variables. This
// is how the workers will get a reference to the globalized variable. The
// members of this list will be passed to the outlined parallel function
// preserving the order.
// Called by all workers.
EXTERN void __kmpc_get_shared_variables(void ***GlobalArgs) {
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}

// This function is used to init static memory manager. This manager is used to
// manage statically allocated global memory. This memory is allocated by the
// compiler and used to correctly implement globalization of the variables in
// target, teams and distribute regions.
EXTERN void __kmpc_get_team_static_memory(int16_t isSPMDExecutionMode,
                                          const void *buf, size_t size,
                                          int16_t is_shared,
                                          const void **frame) {
  if (is_shared) {
    *frame = buf;
    return;
  }
  if (isSPMDExecutionMode) {
    if (GetThreadIdInBlock() == 0) {
      *frame = omptarget_nvptx_simpleMemoryManager.Acquire(buf, size);
    }
    __kmpc_impl_syncthreads();
    return;
  }
  ASSERT0(LT_FUSSY, GetThreadIdInBlock() == GetMasterThreadID(),
          "Must be called only in the target master thread.");
  *frame = omptarget_nvptx_simpleMemoryManager.Acquire(buf, size);
  __kmpc_impl_threadfence();
}

EXTERN void __kmpc_restore_team_static_memory(int16_t isSPMDExecutionMode,
                                              int16_t is_shared) {
  if (is_shared)
    return;
  if (isSPMDExecutionMode) {
    __kmpc_impl_syncthreads();
    if (GetThreadIdInBlock() == 0) {
      omptarget_nvptx_simpleMemoryManager.Release();
    }
    return;
  }
  __kmpc_impl_threadfence();
  ASSERT0(LT_FUSSY, GetThreadIdInBlock() == GetMasterThreadID(),
          "Must be called only in the target master thread.");
  omptarget_nvptx_simpleMemoryManager.Release();
}

