#include "hip/hip_runtime.h"
//===---------- target_impl.cu - NVPTX OpenMP GPU options ------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// Definitions of target specific functions
//
//===----------------------------------------------------------------------===//
#pragma omp declare target

#include "target_impl.h"
#include "common/debug.h"

#include <hip/hip_runtime.h>

// Forward declaration of CUDA primitives which will be evetually transformed
// into LLVM intrinsics.
extern "C" {
unsigned int __activemask();
unsigned int __ballot(unsigned);
// The default argument here is based on NVIDIA's website
// https://developer.nvidia.com/blog/using-cuda-warp-level-primitives/
int __shfl_sync(unsigned mask, int val, int src_line, int width = WARPSIZE);
int __shfl(int val, int src_line, int width = WARPSIZE);
int __shfl_down(int var, unsigned detla, int width);
int __shfl_down_sync(unsigned mask, int var, unsigned detla, int width);
void __syncwarp(int mask);
void __threadfence();
void __threadfence_block();
void __threadfence_system();
}

DEVICE void __kmpc_impl_unpack(uint64_t val, uint32_t &lo, uint32_t &hi) {
  asm volatile("mov.b64 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "l"(val));
}

DEVICE uint64_t __kmpc_impl_pack(uint32_t lo, uint32_t hi) {
  uint64_t val;
  asm volatile("mov.b64 %0, {%1,%2};" : "=l"(val) : "r"(lo), "r"(hi));
  return val;
}

DEVICE __kmpc_impl_lanemask_t __kmpc_impl_lanemask_lt() {
  __kmpc_impl_lanemask_t res;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(res));
  return res;
}

DEVICE __kmpc_impl_lanemask_t __kmpc_impl_lanemask_gt() {
  __kmpc_impl_lanemask_t res;
  asm("mov.u32 %0, %%lanemask_gt;" : "=r"(res));
  return res;
}

DEVICE uint32_t __kmpc_impl_smid() {
  uint32_t id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  return id;
}

DEVICE double __kmpc_impl_get_wtick() {
  // Timer precision is 1ns
  return ((double)1E-9);
}

DEVICE double __kmpc_impl_get_wtime() {
  unsigned long long nsecs;
  asm("mov.u64  %0, %%globaltimer;" : "=l"(nsecs));
  return (double)nsecs * __kmpc_impl_get_wtick();
}

// In Cuda 9.0, __ballot(1) from Cuda 8.0 is replaced with __activemask().
DEVICE __kmpc_impl_lanemask_t __kmpc_impl_activemask() {
#if CUDA_VERSION >= 9000
  return __activemask();
#else
  return __ballot(1);
#endif
}

// In Cuda 9.0, the *_sync() version takes an extra argument 'mask'.
DEVICE int32_t __kmpc_impl_shfl_sync(__kmpc_impl_lanemask_t Mask, int32_t Var,
                                     int32_t SrcLane) {
#if CUDA_VERSION >= 9000
  return __shfl_sync(Mask, Var, SrcLane);
#else
  return __shfl(Var, SrcLane);
#endif // CUDA_VERSION
}

DEVICE int32_t __kmpc_impl_shfl_down_sync(__kmpc_impl_lanemask_t Mask,
                                          int32_t Var, uint32_t Delta,
                                          int32_t Width) {
#if CUDA_VERSION >= 9000
  return __shfl_down_sync(Mask, Var, Delta, Width);
#else
  return __shfl_down(Var, Delta, Width);
#endif // CUDA_VERSION
}

DEVICE void __kmpc_impl_syncthreads() { __syncthreads(); }

DEVICE void __kmpc_impl_syncwarp(__kmpc_impl_lanemask_t Mask) {
#if CUDA_VERSION >= 9000
  __syncwarp(Mask);
#else
  // In Cuda < 9.0 no need to sync threads in warps.
#endif // CUDA_VERSION
}

// NVPTX specific kernel initialization
DEVICE void __kmpc_impl_target_init() { /* nvptx needs no extra setup */
}

// Barrier until num_threads arrive.
DEVICE void __kmpc_impl_named_sync(uint32_t num_threads) {
  // The named barrier for active parallel threads of a team in an L1 parallel
  // region to synchronize with each other.
  int barrier = 1;
  asm volatile("bar.sync %0, %1;"
               :
               : "r"(barrier), "r"(num_threads)
               : "memory");
}

DEVICE void __kmpc_impl_threadfence() { __threadfence(); }
DEVICE void __kmpc_impl_threadfence_block() { __threadfence_block(); }
DEVICE void __kmpc_impl_threadfence_system() { __threadfence_system(); }

// Calls to the NVPTX layer (assuming 1D layout)
DEVICE int GetThreadIdInBlock() { return __nvvm_read_ptx_sreg_tid_x(); }
DEVICE int GetBlockIdInKernel() { return __nvvm_read_ptx_sreg_ctaid_x(); }
DEVICE int GetNumberOfBlocksInKernel() {
  return __nvvm_read_ptx_sreg_nctaid_x();
}
DEVICE int GetNumberOfThreadsInBlock() { return __nvvm_read_ptx_sreg_ntid_x(); }
DEVICE unsigned GetWarpId() { return GetThreadIdInBlock() / WARPSIZE; }
DEVICE unsigned GetLaneId() { return GetThreadIdInBlock() & (WARPSIZE - 1); }

#define __OMP_SPIN 1000
#define UNSET 0u
#define SET 1u

DEVICE void __kmpc_impl_init_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

DEVICE void __kmpc_impl_destroy_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

DEVICE void __kmpc_impl_set_lock(omp_lock_t *lock) {
  // TODO: not sure spinning is a good idea here..
  while (__kmpc_atomic_cas(lock, UNSET, SET) != UNSET) {
    int32_t start = __nvvm_read_ptx_sreg_clock();
    int32_t now;
    for (;;) {
      now = __nvvm_read_ptx_sreg_clock();
      int32_t cycles = now > start ? now - start : now + (0xffffffff - start);
      if (cycles >= __OMP_SPIN * GetBlockIdInKernel()) {
        break;
      }
    }
  } // wait for 0 to be the read value
}

DEVICE void __kmpc_impl_unset_lock(omp_lock_t *lock) {
  (void)__kmpc_atomic_exchange(lock, UNSET);
}

DEVICE int __kmpc_impl_test_lock(omp_lock_t *lock) {
  return __kmpc_atomic_add(lock, 0u);
}

DEVICE void *__kmpc_impl_malloc(size_t x) { return malloc(x); }
DEVICE void __kmpc_impl_free(void *x) { free(x); }

#pragma omp end declare target
