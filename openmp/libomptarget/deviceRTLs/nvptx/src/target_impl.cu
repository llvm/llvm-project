#include "hip/hip_runtime.h"
//===---------- target_impl.cu - NVPTX OpenMP GPU options ------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// Definitions of target specific functions
//
//===----------------------------------------------------------------------===//
#pragma omp declare target

#include "target_impl.h"
#include "common/debug.h"

DEVICE void __kmpc_impl_unpack(uint64_t val, uint32_t &lo, uint32_t &hi) {
  asm volatile("mov.b64 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "l"(val));
}

DEVICE uint64_t __kmpc_impl_pack(uint32_t lo, uint32_t hi) {
  uint64_t val;
  asm volatile("mov.b64 %0, {%1,%2};" : "=l"(val) : "r"(lo), "r"(hi));
  return val;
}

DEVICE __kmpc_impl_lanemask_t __kmpc_impl_lanemask_lt() {
  __kmpc_impl_lanemask_t res;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(res));
  return res;
}

DEVICE __kmpc_impl_lanemask_t __kmpc_impl_lanemask_gt() {
  __kmpc_impl_lanemask_t res;
  asm("mov.u32 %0, %%lanemask_gt;" : "=r"(res));
  return res;
}

DEVICE uint32_t __kmpc_impl_smid() {
  uint32_t id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  return id;
}

DEVICE double __kmpc_impl_get_wtick() {
  // Timer precision is 1ns
  return ((double)1E-9);
}

DEVICE double __kmpc_impl_get_wtime() {
  unsigned long long nsecs;
  asm("mov.u64  %0, %%globaltimer;" : "=l"(nsecs));
  return (double)nsecs * __kmpc_impl_get_wtick();
}

// In Cuda 9.0, __ballot(1) from Cuda 8.0 is replaced with __activemask().
DEVICE __kmpc_impl_lanemask_t __kmpc_impl_activemask() {
#if CUDA_VERSION < 9020
  return __nvvm_vote_ballot(1);
#else
  unsigned int Mask;
  asm volatile("activemask.b32 %0;" : "=r"(Mask));
  return Mask;
#endif
}

// In Cuda 9.0, the *_sync() version takes an extra argument 'mask'.
DEVICE int32_t __kmpc_impl_shfl_sync(__kmpc_impl_lanemask_t Mask, int32_t Var,
                                     int32_t SrcLane) {
#if CUDA_VERSION >= 9000
  return __nvvm_shfl_sync_idx_i32(Mask, Var, SrcLane, 0x1f);
#else
  return __nvvm_shfl_idx_i32(Var, SrcLane, 0x1f);
#endif // CUDA_VERSION
}

DEVICE int32_t __kmpc_impl_shfl_down_sync(__kmpc_impl_lanemask_t Mask,
                                          int32_t Var, uint32_t Delta,
                                          int32_t Width) {
  int32_t T = ((WARPSIZE - Width) << 8) | 0x1f;
#if CUDA_VERSION >= 9000
  return __nvvm_shfl_sync_down_i32(Mask, Var, Delta, T);
#else
  return __nvvm_shfl_down_i32(Var, Delta, T);
#endif // CUDA_VERSION
}

DEVICE void __kmpc_impl_syncthreads() { __syncthreads(); }

DEVICE void __kmpc_impl_syncwarp(__kmpc_impl_lanemask_t Mask) {
#if CUDA_VERSION >= 9000
  __nvvm_bar_warp_sync(Mask);
#else
  // In Cuda < 9.0 no need to sync threads in warps.
#endif // CUDA_VERSION
}

// NVPTX specific kernel initialization
DEVICE void __kmpc_impl_target_init() { /* nvptx needs no extra setup */
}

// Barrier until num_threads arrive.
DEVICE void __kmpc_impl_named_sync(uint32_t num_threads) {
  // The named barrier for active parallel threads of a team in an L1 parallel
  // region to synchronize with each other.
  int barrier = 1;
  asm volatile("bar.sync %0, %1;"
               :
               : "r"(barrier), "r"(num_threads)
               : "memory");
}

DEVICE void __kmpc_impl_threadfence() { __nvvm_membar_gl(); }
DEVICE void __kmpc_impl_threadfence_block() { __nvvm_membar_cta(); }
DEVICE void __kmpc_impl_threadfence_system() { __nvvm_membar_sys(); }

// Calls to the NVPTX layer (assuming 1D layout)
DEVICE int GetThreadIdInBlock() { return __nvvm_read_ptx_sreg_tid_x(); }
DEVICE int GetBlockIdInKernel() { return __nvvm_read_ptx_sreg_ctaid_x(); }
DEVICE int GetNumberOfBlocksInKernel() {
  return __nvvm_read_ptx_sreg_nctaid_x();
}
DEVICE int GetNumberOfThreadsInBlock() { return __nvvm_read_ptx_sreg_ntid_x(); }
DEVICE unsigned GetWarpId() { return GetThreadIdInBlock() / WARPSIZE; }
DEVICE unsigned GetLaneId() { return GetThreadIdInBlock() & (WARPSIZE - 1); }

// Atomics
DEVICE uint32_t __kmpc_atomic_add(uint32_t *Address, uint32_t Val) {
  return __atomic_fetch_add(Address, Val, __ATOMIC_SEQ_CST);
}
DEVICE uint32_t __kmpc_atomic_inc(uint32_t *Address, uint32_t Val) {
  return __nvvm_atom_inc_gen_ui(Address, Val);
}

DEVICE uint32_t __kmpc_atomic_max(uint32_t *Address, uint32_t Val) {
  return __atomic_fetch_max(Address, Val, __ATOMIC_SEQ_CST);
}

DEVICE uint32_t __kmpc_atomic_exchange(uint32_t *Address, uint32_t Val) {
  uint32_t R;
  __atomic_exchange(Address, &Val, &R, __ATOMIC_SEQ_CST);
  return R;
}

DEVICE uint32_t __kmpc_atomic_cas(uint32_t *Address, uint32_t Compare,
                                  uint32_t Val) {
  (void)__atomic_compare_exchange(Address, &Compare, &Val, false,
                                  __ATOMIC_SEQ_CST, __ATOMIC_SEQ_CST);
  return Compare;
}

DEVICE unsigned long long __kmpc_atomic_exchange(unsigned long long *Address,
                                                 unsigned long long Val) {
  unsigned long long R;
  __atomic_exchange(Address, &Val, &R, __ATOMIC_SEQ_CST);
  return R;
}

DEVICE unsigned long long __kmpc_atomic_add(unsigned long long *Address,
                                            unsigned long long Val) {
  return __atomic_fetch_add(Address, Val, __ATOMIC_SEQ_CST);
}

#define __OMP_SPIN 1000
#define UNSET 0u
#define SET 1u

DEVICE void __kmpc_impl_init_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

DEVICE void __kmpc_impl_destroy_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

DEVICE void __kmpc_impl_set_lock(omp_lock_t *lock) {
  // TODO: not sure spinning is a good idea here..
  while (__kmpc_atomic_cas(lock, UNSET, SET) != UNSET) {
    int32_t start = __nvvm_read_ptx_sreg_clock();
    int32_t now;
    for (;;) {
      now = __nvvm_read_ptx_sreg_clock();
      int32_t cycles = now > start ? now - start : now + (0xffffffff - start);
      if (cycles >= __OMP_SPIN * GetBlockIdInKernel()) {
        break;
      }
    }
  } // wait for 0 to be the read value
}

DEVICE void __kmpc_impl_unset_lock(omp_lock_t *lock) {
  (void)__kmpc_atomic_exchange(lock, UNSET);
}

DEVICE int __kmpc_impl_test_lock(omp_lock_t *lock) {
  return __kmpc_atomic_add(lock, 0u);
}

DEVICE void *__kmpc_impl_malloc(size_t x) { return malloc(x); }
DEVICE void __kmpc_impl_free(void *x) { free(x); }

#pragma omp end declare target
