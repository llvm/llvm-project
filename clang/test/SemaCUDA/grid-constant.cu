#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s
#include "Inputs/hip/hip_runtime.h"

struct S {};

__global__ void kernel_struct(__grid_constant__ const S arg) {}
__global__ void kernel_scalar(__grid_constant__ const int arg) {}

__global__ void gc_kernel_non_const(__grid_constant__ S arg) {} // expected-error {{__grid_constant__ is only allowed on const-qualified kernel parameters}}

void non_kernel(__grid_constant__ S arg) {} // expected-error {{__grid_constant__ is only allowed on const-qualified kernel parameters}}

// templates w/ non-dependent argument types get diagnosed right
// away, without instantiation.
template <typename T>
__global__ void tkernel_nd_const(__grid_constant__ const S arg, T dummy) {}
template <typename T>
__global__ void tkernel_nd_non_const(__grid_constant__ S arg, T dummy) {} // expected-error {{__grid_constant__ is only allowed on const-qualified kernel parameters}}

// dependent arguments get diagnosed after instantiation.
template <typename T>
__global__ void tkernel_const(__grid_constant__ const T arg) {}

template <typename T>
__global__ void tkernel(__grid_constant__ T arg) {} // expected-error {{__grid_constant__ is only allowed on const-qualified kernel parameters}}

void foo() {
  tkernel_const<const S><<<1,1>>>({});
  tkernel_const<S><<<1,1>>>({});
  tkernel<const S><<<1,1>>>({});
  tkernel<S><<<1,1>>>({}); // expected-note {{in instantiation of function template specialization 'tkernel<S>' requested here}}
}
