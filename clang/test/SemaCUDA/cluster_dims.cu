#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -ast-print -x hip -verify=NS,all %s
// RUN: %clang_cc1 -triple nvptx-nvidia-cuda -fcuda-is-device -ast-print -x hip -verify=NS,all %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -target-cpu gfx1250 -fcuda-is-device -ast-print -x hip -verify=amd,common,all %s | FileCheck -check-prefixes=CHECK %s
// RUN: %clang_cc1 -triple nvptx-nvidia-cuda -target-cpu sm_90 -fcuda-is-device -ast-print -x hip -verify=cuda,common,all %s | FileCheck -check-prefixes=CHECK %s
// RUN: %clang_cc1 -triple x86_64-pc-linux-gnu -aux-triple amdgcn-amd-amdhsa -ast-print -x hip -verify=amd,common,all %s | FileCheck -check-prefixes=CHECK %s

#include "Inputs/hip/hip_runtime.h"

const int constint = 4;

// CHECK: __attribute__((global)) __attribute__((cluster_dims(2, 2, 2))) void test_literal_3d()
__global__ void __cluster_dims__(2, 2, 2) test_literal_3d() {} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// CHECK: __attribute__((global)) __attribute__((cluster_dims(2, 2))) void test_literal_2d()
__global__ void __cluster_dims__(2, 2) test_literal_2d() {} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// CHECK: __attribute__((global)) __attribute__((cluster_dims(4))) void test_literal_1d()
__global__ void __cluster_dims__(4) test_literal_1d() {} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// CHECK: __attribute__((global)) __attribute__((cluster_dims(constint, constint / 4, 1))) void test_constant()
__global__ void __cluster_dims__(constint, constint / 4, 1) test_constant() {} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// CHECK: template <int x, int y, int z> void test_template() __attribute__((cluster_dims(x, y, z)))
template <int x, int y, int z>  void test_template(void) __cluster_dims__(x, y, z){} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// CHECK: template <int x, int y, int z> void test_template_expr() __attribute__((cluster_dims(x + constint, y, z)))
template <int x, int y, int z> void test_template_expr(void) __cluster_dims__(x + constint, y, z) {} //NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

//NS-error@+1 {{__cluster_dims__ is not supported for this GPU architecture}}
__global__ void __cluster_dims__(32, 2, 4) test_too_large_dim_0() {} // common-error {{integer constant expression evaluates to value 32 that cannot be represented in a 4-bit unsigned integer type}}

// cuda-error@+2 {{only a maximum of 8 thread blocks in a cluster is supported}}
// amd-error@+1 {{only a maximum of 16 thread blocks in a cluster is supported}}
__global__ void __cluster_dims__(4, 4, 4) test_too_large_dim_1() {} // NS-error {{__cluster_dims__ is not supported for this GPU architecture}}

// cuda-error@+3 {{only a maximum of 8 thread blocks in a cluster is supported}}
// amd-error@+2 {{only a maximum of 16 thread blocks in a cluster is supported}}
template<unsigned a, unsigned b, unsigned c>
__global__ void __cluster_dims__(a, b, c) test_too_large_dim_template() {} // NS-error {{__cluster_dims__ is not supported for this GPU architecture}}
template __global__ void test_too_large_dim_template<4, 4, 4>(); // common-note {{in instantiation of function template specialization 'test_too_large_dim_template<4U, 4U, 4U>' requested here}}

int none_const_int = 4;

//NS-error@+1 {{__cluster_dims__ is not supported for this GPU architecture}}
__global__ void __cluster_dims__(none_const_int, 2, 4) test_non_constant_0() {} // common-error {{'cluster_dims' attribute requires parameter 0 to be an integer constant}}

//NS-error@+1 {{__cluster_dims__ is not supported for this GPU architecture}}
__global__ void __cluster_dims__(8, none_const_int / 2, 4) test_non_constant_1() {} // common-error {{'cluster_dims' attribute requires parameter 1 to be an integer constant}}

//NS-error@+1 {{__cluster_dims__ is not supported for this GPU architecture}}
__global__ void __cluster_dims__(8, 2, none_const_int / 4) test_non_constant_2() {} // common-error {{'cluster_dims' attribute requires parameter 2 to be an integer constant}}

//NS-error@+1 {{__no_cluster__ is not supported for this GPU architecture}}
__global__ void __no_cluster__ test_no_cluster() {}

//NS-error@+2 {{__no_cluster__ is not supported for this GPU architecture}}
//NS-error@+1 {{__cluster_dims__ is not supported for this GPU architecture}}
__global__ void __no_cluster__ __cluster_dims__(2,2,2) test_have_both() {} // common-error {{'cluster_dims' and 'no_cluster' attributes are not compatible}} common-note {{conflicting attribute is here}}

template <int... args>
__cluster_dims__(args) void test_template_variadic_args(void) {} // all-error {{expression contains unexpanded parameter pack 'args'}}

template <int... args>
__cluster_dims__(1, args) void test_template_variadic_args_2(void) {} // all-error {{expression contains unexpanded parameter pack 'args'}}
