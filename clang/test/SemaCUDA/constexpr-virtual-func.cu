// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only \
// RUN:            -fcuda-is-device %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only %s

// expected-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

class A
{
public:
    constexpr virtual int f() = 0;
};

class B : public A
{
public:
    int f() override
    {
        return 42;
    }
};

int test()
{
    B b;
    return b.f();
}
