#include "hip/hip_runtime.h"
// expected-no-diagnostics

// RUN: %clang_cc1 -triple spirv64 -aux-triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -verify -fsyntax-only %s

#define __global__ __attribute__((global))

__attribute__((reqd_work_group_size(128, 1, 1)))
__global__ void reqd_work_group_size_128_1_1() {}

template <unsigned a, unsigned b, unsigned c>
__attribute__((reqd_work_group_size(a, b, c)))
__global__ void reqd_work_group_size_a_b_c() {}

template <>
__global__ void reqd_work_group_size_a_b_c<128,1,1>(void);

__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}

template <unsigned a, unsigned b, unsigned c>
__attribute__((work_group_size_hint(a, b, c)))
__global__ void work_group_size_hint_a_b_c() {}

template <>
__global__ void work_group_size_hint_a_b_c<128,1,1>(void);

__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}

__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}
