#include "hip/hip_runtime.h"
// expected-no-diagnostics

// RUN: %clang_cc1 -triple spirv64 -aux-triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -verify -fsyntax-only %s

#include "Inputs/hip/hip_runtime.h"

__attribute__((reqd_work_group_size(128, 1, 1)))
__global__ void reqd_work_group_size_128_1_1() {}

__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}

__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}

__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}
