
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -aux-triple \
// RUN:   x86_64-pc-windows-msvc -fms-compatibility -fcuda-is-device \
// RUN:   -fsyntax-only -verify -x hip %s
// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -fms-compatibility \
// RUN:   -fsyntax-only -verify -x hip %s

// expected-no-diagnostics

typedef void (__stdcall* funcTy)();
void invoke(funcTy f);

static void __stdcall callee() noexcept {
}

void foo() {
   invoke(callee);
}
