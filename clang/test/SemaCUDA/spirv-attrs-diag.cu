#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple spirv64 -aux-triple x86_64-unknown-linux-gnu \
// RUN:   -fcuda-is-device -verify -fsyntax-only %s

#define __global__ __attribute__((global))

__attribute__((reqd_work_group_size(0x100000000, 1, 1))) // expected-error {{integer constant expression evaluates to value 4294967296 that cannot be represented in a 32-bit unsigned integer type}}
__global__ void TestTooBigArg1(void);

__attribute__((work_group_size_hint(0x100000000, 1, 1))) // expected-error {{integer constant expression evaluates to value 4294967296 that cannot be represented in a 32-bit unsigned integer type}}
__global__ void TestTooBigArg2(void);

template <int... Args>
__attribute__((reqd_work_group_size(Args))) // expected-error {{expression contains unexpanded parameter pack 'Args'}}
__global__ void TestTemplateVariadicArgs1(void) {}

template <int... Args>
__attribute__((work_group_size_hint(Args))) // expected-error {{expression contains unexpanded parameter pack 'Args'}}
__global__ void TestTemplateVariadicArgs2(void) {}

template <class a> // expected-note {{declared here}}
__attribute__((reqd_work_group_size(a, 1, 1))) // expected-error {{'a' does not refer to a value}}
__global__ void TestTemplateArgClass1(void) {}

template <class a> // expected-note {{declared here}}
__attribute__((work_group_size_hint(a, 1, 1))) // expected-error {{'a' does not refer to a value}}
__global__ void TestTemplateArgClass2(void) {}

constexpr int A = 512;

__attribute__((reqd_work_group_size(A, A, A)))
__global__ void TestConstIntArg1(void) {}

__attribute__((work_group_size_hint(A, A, A)))
__global__ void TestConstIntArg2(void) {}

int B = 512;
__attribute__((reqd_work_group_size(B, 1, 1))) // expected-error {{attribute requires parameter 0 to be an integer constant}}
__global__ void TestNonConstIntArg1(void) {}

__attribute__((work_group_size_hint(B, 1, 1))) // expected-error {{attribute requires parameter 0 to be an integer constant}}
__global__ void TestNonConstIntArg2(void) {}

constexpr int C = -512;
__attribute__((reqd_work_group_size(C, 1, 1))) // expected-error {{attribute requires a non-negative integral compile time constant expression}}
__global__ void TestNegativeConstIntArg1(void) {}

__attribute__((work_group_size_hint(C, 1, 1))) // expected-error {{attribute requires a non-negative integral compile time constant expression}}
__global__ void TestNegativeConstIntArg2(void) {}


__attribute__((reqd_work_group_size(A, 0, 1))) // expected-error {{attribute must be greater than 0}}
__global__ void TestZeroArg1(void) {}

__attribute__((work_group_size_hint(A, 0, 1))) // expected-error {{attribute must be greater than 0}}
__global__ void TestZeroArg2(void) {}



