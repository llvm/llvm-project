#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

// Opaque return types used to check that we pick the right overloads.
struct HostReturnTy {};
struct HostReturnTy2 {};
struct DeviceReturnTy {};
struct DeviceReturnTy2 {};
struct HostDeviceReturnTy {};
struct TemplateReturnTy {};

typedef HostReturnTy (*HostFnPtr)();
typedef DeviceReturnTy (*DeviceFnPtr)();
typedef HostDeviceReturnTy (*HostDeviceFnPtr)();
typedef void (*GlobalFnPtr)();  // __global__ functions must return void.

// CurrentReturnTy is {HostReturnTy,DeviceReturnTy} during {host,device}
// compilation.
#ifdef __CUDA_ARCH__
typedef DeviceReturnTy CurrentReturnTy;
#else
typedef HostReturnTy CurrentReturnTy;
#endif

// CurrentFnPtr is a function pointer to a {host,device} function during
// {host,device} compilation.
typedef CurrentReturnTy (*CurrentFnPtr)();

// Host and unattributed functions can't be overloaded.
__host__ void hh() {} // expected-note {{previous definition is here}}
void hh() {} // expected-error {{redefinition of 'hh'}}

// H/D overloading is OK.
__host__ HostReturnTy dh() { return HostReturnTy(); }
__device__ DeviceReturnTy dh() { return DeviceReturnTy(); }

// H/HD and D/HD are not allowed.
__host__ __device__ int hdh() { return 0; } // expected-note {{previous declaration is here}}
__host__ int hdh() { return 0; }
// expected-error@-1 {{__host__ function 'hdh' cannot overload __host__ __device__ function 'hdh'}}

__host__ int hhd() { return 0; }            // expected-note {{previous declaration is here}}
__host__ __device__ int hhd() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'hhd' cannot overload __host__ function 'hhd'}}

__host__ __device__ int hdd() { return 0; } // expected-note {{previous declaration is here}}
__device__ int hdd() { return 0; }
// expected-error@-1 {{__device__ function 'hdd' cannot overload __host__ __device__ function 'hdd'}}

__device__ int dhd() { return 0; }          // expected-note {{previous declaration is here}}
__host__ __device__ int dhd() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'dhd' cannot overload __device__ function 'dhd'}}

// Same tests for extern "C" functions.
extern "C" __host__ int chh() { return 0; } // expected-note {{previous definition is here}}
extern "C" int chh() { return 0; }          // expected-error {{redefinition of 'chh'}}

// H/D overloading is OK.
extern "C" __device__ DeviceReturnTy cdh() { return DeviceReturnTy(); }
extern "C" __host__ HostReturnTy cdh() { return HostReturnTy(); }

// H/HD and D/HD overloading is not allowed.
extern "C" __host__ __device__ int chhd1() { return 0; } // expected-note {{previous declaration is here}}
extern "C" __host__ int chhd1() { return 0; }
// expected-error@-1 {{__host__ function 'chhd1' cannot overload __host__ __device__ function 'chhd1'}}

extern "C" __host__ int chhd2() { return 0; } // expected-note {{previous declaration is here}}
extern "C" __host__ __device__ int chhd2() { return 0; }
// expected-error@-1 {{__host__ __device__ function 'chhd2' cannot overload __host__ function 'chhd2'}}

// Helper functions to verify calling restrictions.
__device__ DeviceReturnTy d() { return DeviceReturnTy(); }
// expected-note@-1 1+ {{'d' declared here}}
// expected-note@-2 1+ {{candidate function not viable: call to __device__ function from __host__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __device__ function from __host__ __device__ function}}

__host__ HostReturnTy h() { return HostReturnTy(); }
// expected-note@-1 1+ {{'h' declared here}}
// expected-note@-2 1+ {{candidate function not viable: call to __host__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
// expected-note@-4 1+ {{candidate function not viable: call to __host__ function from __global__ function}}

__global__ void g() {}
// expected-note@-1 1+ {{'g' declared here}}
// expected-note@-2 1+ {{candidate function not viable: call to __global__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __global__ function from __host__ __device__ function}}
// expected-note@-4 1+ {{candidate function not viable: call to __global__ function from __global__ function}}

extern "C" __device__ DeviceReturnTy cd() { return DeviceReturnTy(); }
// expected-note@-1 1+ {{'cd' declared here}}
// expected-note@-2 1+ {{candidate function not viable: call to __device__ function from __host__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __device__ function from __host__ __device__ function}}

extern "C" __host__ HostReturnTy ch() { return HostReturnTy(); }
// expected-note@-1 1+ {{'ch' declared here}}
// expected-note@-2 1+ {{candidate function not viable: call to __host__ function from __device__ function}}
// expected-note@-3 0+ {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
// expected-note@-4 1+ {{candidate function not viable: call to __host__ function from __global__ function}}

__host__ void hostf() {
  DeviceFnPtr fp_d = d;         // expected-error {{reference to __device__ function 'd' in __host__ function}}
  DeviceReturnTy ret_d = d();   // expected-error {{no matching function for call to 'd'}}
  DeviceFnPtr fp_cd = cd;       // expected-error {{reference to __device__ function 'cd' in __host__ function}}
  DeviceReturnTy ret_cd = cd(); // expected-error {{no matching function for call to 'cd'}}

  HostFnPtr fp_h = h;
  HostReturnTy ret_h = h();
  HostFnPtr fp_ch = ch;
  HostReturnTy ret_ch = ch();

  HostFnPtr fp_dh = dh;
  HostReturnTy ret_dh = dh();
  HostFnPtr fp_cdh = cdh;
  HostReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g;
  g(); // expected-error {{call to global function 'g' not configured}}
  g<<<0, 0>>>();
}

__device__ void devicef() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();

  HostFnPtr fp_h = h;         // expected-error {{reference to __host__ function 'h' in __device__ function}}
  HostReturnTy ret_h = h();   // expected-error {{no matching function for call to 'h'}}
  HostFnPtr fp_ch = ch;       // expected-error {{reference to __host__ function 'ch' in __device__ function}}
  HostReturnTy ret_ch = ch(); // expected-error {{no matching function for call to 'ch'}}

  DeviceFnPtr fp_dh = dh;
  DeviceReturnTy ret_dh = dh();
  DeviceFnPtr fp_cdh = cdh;
  DeviceReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g; // expected-error {{reference to __global__ function 'g' in __device__ function}}
  g(); // expected-error {{no matching function for call to 'g'}}
  g<<<0,0>>>(); // expected-error {{reference to __global__ function 'g' in __device__ function}}
}

__global__ void globalf() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();

  HostFnPtr fp_h = h;         // expected-error {{reference to __host__ function 'h' in __global__ function}}
  HostReturnTy ret_h = h();   // expected-error {{no matching function for call to 'h'}}
  HostFnPtr fp_ch = ch;       // expected-error {{reference to __host__ function 'ch' in __global__ function}}
  HostReturnTy ret_ch = ch(); // expected-error {{no matching function for call to 'ch'}}

  DeviceFnPtr fp_dh = dh;
  DeviceReturnTy ret_dh = dh();
  DeviceFnPtr fp_cdh = cdh;
  DeviceReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g; // expected-error {{reference to __global__ function 'g' in __global__ function}}
  g(); // expected-error {{no matching function for call to 'g'}}
  g<<<0,0>>>(); // expected-error {{reference to __global__ function 'g' in __global__ function}}
}

__host__ __device__ void hostdevicef() {
  DeviceFnPtr fp_d = d;
  DeviceReturnTy ret_d = d();
  DeviceFnPtr fp_cd = cd;
  DeviceReturnTy ret_cd = cd();
#if !defined(__CUDA_ARCH__)
  // expected-error@-5 {{reference to __device__ function 'd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'cd' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __device__ function 'cd' in __host__ __device__ function}}
#endif

  HostFnPtr fp_h = h;
  HostReturnTy ret_h = h();
  HostFnPtr fp_ch = ch;
  HostReturnTy ret_ch = ch();
#if defined(__CUDA_ARCH__)
  // expected-error@-5 {{reference to __host__ function 'h' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __host__ function 'h' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __host__ function 'ch' in __host__ __device__ function}}
  // expected-error@-5 {{reference to __host__ function 'ch' in __host__ __device__ function}}
#endif

  CurrentFnPtr fp_dh = dh;
  CurrentReturnTy ret_dh = dh();
  CurrentFnPtr fp_cdh = cdh;
  CurrentReturnTy ret_cdh = cdh();

  GlobalFnPtr fp_g = g;
#if defined(__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#endif

  g();
#if defined (__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#else
  // expected-error@-4 {{call to global function 'g' not configured}}
#endif

  g<<<0,0>>>();
#if defined(__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
#endif
}

// Test for address of overloaded function resolution in the global context.
HostFnPtr fp_h = h;
HostFnPtr fp_ch = ch;
CurrentFnPtr fp_dh = dh;
CurrentFnPtr fp_cdh = cdh;
GlobalFnPtr fp_g = g;


// Test overloading of destructors
// Can't mix H and unattributed destructors
struct d_h {
  ~d_h() {} // expected-note {{previous definition is here}}
  __host__ ~d_h() {} // expected-error {{destructor cannot be redeclared}}
};

// HD is OK
struct d_hd {
  __host__ __device__ ~d_hd() {}
};

// Test overloading of member functions
struct m_h {
  void operator delete(void *ptr); // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr); // expected-error {{class member cannot be redeclared}}
};

// D/H overloading is OK
struct m_dh {
  __device__ void operator delete(void *ptr);
  __host__ void operator delete(void *ptr);
};

// HD by itself is OK
struct m_hd {
  __device__ __host__ void operator delete(void *ptr);
};

struct m_hhd {
  __host__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ __device__ function 'operator delete' cannot overload __host__ function 'operator delete'}}
};

struct m_hdh {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ function 'operator delete' cannot overload __host__ __device__ function 'operator delete'}}
};

struct m_dhd {
  __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__host__ __device__ function 'operator delete' cannot overload __device__ function 'operator delete'}}
};

struct m_hdd {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __device__ void operator delete(void *ptr) {}
  // expected-error@-1 {{__device__ function 'operator delete' cannot overload __host__ __device__ function 'operator delete'}}
};

// __global__ functions can't be overloaded based on attribute
// difference.
struct G {
  friend void friend_of_g(G &arg); // expected-note {{previous declaration is here}}
private:
  int x; // expected-note {{declared private here}}
};
__global__ void friend_of_g(G &arg) { int x = arg.x; }
// expected-error@-1 {{__global__ function 'friend_of_g' cannot overload __host__ function 'friend_of_g'}}
// expected-error@-2 {{'x' is a private member of 'G'}}
void friend_of_g(G &arg) { int x = arg.x; }

// HD functions are sometimes allowed to call H or D functions -- this
// is an artifact of the source-to-source splitting performed by nvcc
// that we need to mimic. During device mode compilation in nvcc, host
// functions aren't present at all, so don't participate in
// overloading. But in clang, H and D functions are present in both
// compilation modes. Clang normally uses the target attribute as a
// tiebreaker between overloads with otherwise identical priority, but
// in order to match nvcc's behavior, we sometimes need to wholly
// discard overloads that would not be present during compilation
// under nvcc.

template <typename T> TemplateReturnTy template_vs_function(T arg) {
  return TemplateReturnTy();
}
__device__ DeviceReturnTy template_vs_function(float arg) {
  return DeviceReturnTy();
}

// Here we expect to call the templated function during host compilation, even
// if -fcuda-disable-target-call-checks is passed, and even though C++ overload
// rules prefer the non-templated function.
__host__ __device__ void test_host_device_calls_template(void) {
#ifdef __CUDA_ARCH__
  typedef DeviceReturnTy ExpectedReturnTy;
#else
  typedef TemplateReturnTy ExpectedReturnTy;
#endif

  ExpectedReturnTy ret1 = template_vs_function(1.0f);
  ExpectedReturnTy ret2 = template_vs_function(2.0);
}

// Calls from __host__ and __device__ functions should always call the
// overloaded function that matches their mode.
__host__ void test_host_calls_template_fn() {
  TemplateReturnTy ret1 = template_vs_function(1.0f);
  TemplateReturnTy ret2 = template_vs_function(2.0);
}

__device__ void test_device_calls_template_fn() {
  DeviceReturnTy ret1 = template_vs_function(1.0f);
  DeviceReturnTy ret2 = template_vs_function(2.0);
}

// If we have a mix of HD and H-only or D-only candidates in the overload set,
// normal C++ overload resolution rules apply first.
template <typename T> TemplateReturnTy template_vs_hd_function(T arg)
#ifdef __CUDA_ARCH__
//expected-note@-2 {{declared here}}
#endif
{
  return TemplateReturnTy();
}
__host__ __device__ HostDeviceReturnTy template_vs_hd_function(float arg) {
  return HostDeviceReturnTy();
}

__host__ __device__ void test_host_device_calls_hd_template() {
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  TemplateReturnTy ret2 = template_vs_hd_function(1);
#ifdef __CUDA_ARCH__
  // expected-error@-2 {{reference to __host__ function 'template_vs_hd_function<int>' in __host__ __device__ function}}
#endif
}

__host__ void test_host_calls_hd_template() {
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  TemplateReturnTy ret2 = template_vs_hd_function(1);
}

__device__ void test_device_calls_hd_template() {
  HostDeviceReturnTy ret1 = template_vs_hd_function(1.0f);
  // Host-only function template is not callable with strict call checks,
  // so for device side HD function will be the only choice.
  HostDeviceReturnTy ret2 = template_vs_hd_function(1);
}

// Check that overloads still work the same way on both host and
// device side when the overload set contains only functions from one
// side of compilation.
__device__ DeviceReturnTy device_only_function(int arg) { return DeviceReturnTy(); }
__device__ DeviceReturnTy2 device_only_function(float arg) { return DeviceReturnTy2(); }
#ifndef __CUDA_ARCH__
  // expected-note@-3 {{'device_only_function' declared here}}
  // expected-note@-3 {{'device_only_function' declared here}}
#endif
__host__ HostReturnTy host_only_function(int arg) { return HostReturnTy(); }
__host__ HostReturnTy2 host_only_function(float arg) { return HostReturnTy2(); }
#ifdef __CUDA_ARCH__
  // expected-note@-3 {{'host_only_function' declared here}}
  // expected-note@-3 {{'host_only_function' declared here}}
#endif

__host__ __device__ void test_host_device_single_side_overloading() {
  DeviceReturnTy ret1 = device_only_function(1);
  DeviceReturnTy2 ret2 = device_only_function(1.0f);
#ifndef __CUDA_ARCH__
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __device__ function 'device_only_function' in __host__ __device__ function}}
#endif
  HostReturnTy ret3 = host_only_function(1);
  HostReturnTy2 ret4 = host_only_function(1.0f);
#ifdef __CUDA_ARCH__
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
  // expected-error@-3 {{reference to __host__ function 'host_only_function' in __host__ __device__ function}}
#endif
}

// Verify that we allow overloading function templates.
template <typename T> __host__ T template_overload(const T &a) { return a; };
template <typename T> __device__ T template_overload(const T &a) { return a; };

__host__ void test_host_template_overload() {
  template_overload(1); // OK. Attribute-based overloading picks __host__ variant.
}
__device__ void test_device_template_overload() {
  template_overload(1); // OK. Attribute-based overloading picks __device__ variant.
}

// Two classes with `operator-` defined. One of them is device only.
struct C1;
struct C2;
__device__
int operator-(const C1 &x, const C1 &y);
int operator-(const C2 &x, const C2 &y);

template <typename T>
__host__ __device__ int constexpr_overload(const T &x, const T &y) {
  return x - y;
}

// Verify that function overloading doesn't prune candidate wrongly.
int test_constexpr_overload(C2 &x, C2 &y) {
  return constexpr_overload(x, y);
}
