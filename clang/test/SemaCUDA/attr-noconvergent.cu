// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

__device__ float f0(float) __attribute__((noconvergent));
__device__ __attribute__((noconvergent)) float f1(float);
[[clang::noconvergent]] __device__ float f2(float);

__device__ [[clang::noconvergent(1)]] float f3(float);
// expected-error@-1 {{'noconvergent' attribute takes no arguments}}

__device__ [[clang::noconvergent]] float g0;
// expected-warning@-1 {{'noconvergent' attribute only applies to functions and statements}}

__device__ __attribute__((convergent)) __attribute__((noconvergent)) float f4(float);
// expected-error@-1 {{'noconvergent' and 'convergent' attributes are not compatible}}
// expected-note@-2 {{conflicting attribute is here}}

__device__ [[clang::noconvergent]] float f5(float);
__device__ [[clang::convergent]] float f5(float);
// expected-error@-1 {{'convergent' and 'noconvergent' attributes are not compatible}}
// expected-note@-3 {{conflicting attribute is here}}

__device__ float f5(float x) {
  [[clang::noconvergent]] float y;
// expected-warning@-1 {{'noconvergent' attribute only applies to functions and statements}}

  float z;

  [[clang::noconvergent]] z = 1;
// expected-warning@-1 {{'noconvergent' attribute is ignored because there exists no call expression inside the statement}}

  [[clang::noconvergent]] z = f0(x);
}
