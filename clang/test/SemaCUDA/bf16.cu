// REQUIRES: nvptx-registered-target
// REQUIRES: x86-registered-target

// RUN: %clang_cc1 "-triple" "x86_64-unknown-linux-gnu" "-aux-triple" "nvptx64-nvidia-cuda" \
// RUN:    "-target-cpu" "x86-64" -fsyntax-only -verify=scalar %s
// RUN: %clang_cc1 "-aux-triple" "x86_64-unknown-linux-gnu" "-triple" "nvptx64-nvidia-cuda" \
// RUN:    -fcuda-is-device "-aux-target-cpu" "x86-64" -fsyntax-only -verify=scalar %s

#include "Inputs/hip/hip_runtime.h"

__device__ void test(bool b, __bf16 *out, __bf16 in) {
  __bf16 bf16 = in; // No error on using the type itself.

  bf16 + bf16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 - bf16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 * bf16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 / bf16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}

  __fp16 fp16;

  bf16 + fp16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 + bf16; // scalar-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 - fp16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 - bf16; // scalar-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 * fp16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 * bf16; // scalar-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 / fp16; // scalar-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 / bf16; // scalar-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 = fp16; // scalar-error {{assigning to '__bf16' from incompatible type '__fp16'}}
  fp16 = bf16; // scalar-error {{assigning to '__fp16' from incompatible type '__bf16'}}
  bf16 + (b ? fp16 : bf16); // scalar-error {{incompatible operand types ('__fp16' and '__bf16')}}
  *out = bf16;
}
