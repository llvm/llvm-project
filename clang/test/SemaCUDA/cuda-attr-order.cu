#include "hip/hip_runtime.h"
// Verify that we can parse a simple CUDA file with different attributes order.
// RUN: %clang_cc1 "-triple" "nvptx-nvidia-cuda"  -fsyntax-only -verify %s
// expected-no-diagnostics
#include "Inputs/hip/hip_runtime.h"

struct alignas(16) float4 {
    float x, y, z, w;
};

__attribute__((device)) float func() {
    __shared__ alignas(alignof(float4)) float As[4][4];  // Both combinations
    alignas(alignof(float4)) __shared__  float Bs[4][4]; // must be legal

    return As[0][0] + Bs[0][0];
}
