// REQUIRES: amdgpu-registered-target
// REQUIRES: x86-registered-target

// RUN: %clang_cc1 "-triple" "x86_64-unknown-linux-gnu" "-aux-triple" "amdgcn-amd-amdhsa"\
// RUN:    "-target-cpu" "x86-64" -fsyntax-only -verify=amdgcn %s
// RUN: %clang_cc1 "-aux-triple" "x86_64-unknown-linux-gnu" "-triple" "amdgcn-amd-amdhsa"\
// RUN:    -fcuda-is-device "-aux-target-cpu" "x86-64" -fsyntax-only -verify=amdgcn %s

// RUN: %clang_cc1 "-aux-triple" "x86_64-unknown-linux-gnu" "-triple" "r600-unknown-unknown"\
// RUN:    -fcuda-is-device "-aux-target-cpu" "x86-64" -fsyntax-only -verify=amdgcn,r600 %s

// AMDGCN has storage-only support for bf16. R600 does not support it should error out when
// it's the main target.

#include "Inputs/hip/hip_runtime.h"

// There should be no errors on using the type itself, or when loading/storing values for amdgcn.
// r600 should error on all uses of the type.

// r600-error@+1 {{__bf16 is not supported on this target}}
typedef __attribute__((ext_vector_type(2))) __bf16 bf16_x2;
// r600-error@+1 {{__bf16 is not supported on this target}}
typedef __attribute__((ext_vector_type(4))) __bf16 bf16_x4;
// r600-error@+1 {{__bf16 is not supported on this target}}
typedef __attribute__((ext_vector_type(8))) __bf16 bf16_x8;
// r600-error@+1 {{__bf16 is not supported on this target}}
typedef __attribute__((ext_vector_type(16))) __bf16 bf16_x16;

// r600-error@+1 2 {{__bf16 is not supported on this target}}
__device__ void test(bool b, __bf16 *out, __bf16 in) {
  __bf16 bf16 = in;  // r600-error {{__bf16 is not supported on this target}}

  bf16 + bf16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 - bf16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 * bf16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}
  bf16 / bf16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__bf16')}}

  __fp16 fp16;

  bf16 + fp16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 + bf16; // amdgcn-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 - fp16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 - bf16; // amdgcn-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 * fp16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 * bf16; // amdgcn-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 / fp16; // amdgcn-error {{invalid operands to binary expression ('__bf16' and '__fp16')}}
  fp16 / bf16; // amdgcn-error {{invalid operands to binary expression ('__fp16' and '__bf16')}}
  bf16 = fp16; // amdgcn-error {{assigning to '__bf16' from incompatible type '__fp16'}}
  fp16 = bf16; // amdgcn-error {{assigning to '__fp16' from incompatible type '__bf16'}}
  bf16 + (b ? fp16 : bf16); // amdgcn-error {{incompatible operand types ('__fp16' and '__bf16')}}
  *out = bf16;

  // amdgcn-error@+1 {{static_cast from '__bf16' to 'unsigned short' is not allowed}}
  unsigned short u16bf16 = static_cast<unsigned short>(bf16);
  // amdgcn-error@+2 {{C-style cast from 'unsigned short' to '__bf16' is not allowed}}
  // r600-error@+1 {{__bf16 is not supported on this target}}
  bf16 = (__bf16)u16bf16;

  // amdgcn-error@+1 {{static_cast from '__bf16' to 'float' is not allowed}}
  float f32bf16 = static_cast<float>(bf16);
  // amdgcn-error@+2 {{C-style cast from 'float' to '__bf16' is not allowed}}
  // r600-error@+1 {{__bf16 is not supported on this target}}
  bf16 = (__bf16)f32bf16;

  // amdgcn-error@+1 {{static_cast from '__bf16' to 'double' is not allowed}}
  double f64bf16 = static_cast<double>(bf16);
  // amdgcn-error@+2 {{C-style cast from 'double' to '__bf16' is not allowed}}
  // r600-error@+1 {{__bf16 is not supported on this target}}
  bf16 = (__bf16)f64bf16;

  // r600-error@+1 {{__bf16 is not supported on this target}}
  typedef __attribute__((ext_vector_type(2))) __bf16 bf16_x2;
  bf16_x2 vec2_a, vec2_b;
  vec2_a = vec2_b;

  // r600-error@+1 {{__bf16 is not supported on this target}}
  typedef __attribute__((ext_vector_type(4))) __bf16 bf16_x4;
  bf16_x4 vec4_a, vec4_b;
  vec4_a = vec4_b;

  // r600-error@+1 {{__bf16 is not supported on this target}}
  typedef __attribute__((ext_vector_type(8))) __bf16 bf16_x8;
  bf16_x8 vec8_a, vec8_b;
  vec8_a = vec8_b;

  // r600-error@+1 {{__bf16 is not supported on this target}}
  typedef __attribute__((ext_vector_type(16))) __bf16 bf16_x16;
  bf16_x16 vec16_a, vec16_b;
  vec16_a = vec16_b;
}

// r600-error@+1 2 {{__bf16 is not supported on this target}}
__bf16 hostfn(__bf16 a) {
  return a;
}

// r600-error@+2 {{__bf16 is not supported on this target}}
// r600-error@+1 {{vector size not an integral multiple of component size}}
typedef __bf16 foo __attribute__((__vector_size__(16), __aligned__(16)));
