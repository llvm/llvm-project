#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple nvptx -fcuda-is-device \
// RUN:    -emit-llvm -o - %s -fsyntax-only -verify=dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux \
// RUN:    -emit-llvm -o - %s -fsyntax-only -verify=host

// Checks allowed usage of file-scope and function-scope static variables.

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

// Checks static variables are allowed in device functions.

__device__ void f1() {
  const static int b = 123;
  static int a;
}

// Checks static variables are allowd in global functions.

__global__ void k1() {
  const static int b = 123;
  static int a;
}

// Checks static device and constant variables are allowed in device and
// host functions, and static host variables are not allowed in device
// functions.

static __device__ int x;
static __constant__ int y;
static int z;

__global__ void kernel(int *a) {
  a[0] = x;
  a[1] = y;
  a[2] = z;
  // dev-error@-1 {{reference to __host__ variable 'z' in __global__ function}}
}

int* getDeviceSymbol(int *x);

void foo() {
  getDeviceSymbol(&x);
  getDeviceSymbol(&y);
}
