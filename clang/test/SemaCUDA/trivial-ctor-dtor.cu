#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -isystem %S/Inputs  -fsyntax-only -verify %s
// RUN: %clang_cc1 -isystem %S/Inputs -fcuda-is-device -fsyntax-only -verify %s

#include <hip/hip_runtime.h>

// Check trivial ctor/dtor
struct A {
  int x;
  A() {}
  ~A() {}
};

__device__ A a;

// Check trivial ctor/dtor of template class
template<typename T>
struct TA {
  T x;
  TA() {}
  ~TA() {}
};

__device__ TA<int> ta;

// Check non-trivial ctor/dtor in parent template class
template<typename T>
struct TB {
  T x;
  TB() { static int nontrivial_ctor = 1; }
  ~TB() {}
};

template<typename T>
struct TC : TB<T> {
  T x;
  TC() {}
  ~TC() {}
};

template class TC<int>;

__device__ TC<int> tc; //expected-error {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}

// Check trivial ctor specialization
template <typename T>
struct C {
    explicit C() {};
};

template <> C<int>::C() {};
__device__ C<int> ci_d;
C<int> ci_h;

// Check non-trivial ctor specialization
template <> C<float>::C() { static int nontrivial_ctor = 1; }
__device__ C<float> cf_d; //expected-error {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
C<float> cf_h;
