#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -isystem %S/Inputs  -fsyntax-only -verify %s
// RUN: %clang_cc1 -isystem %S/Inputs -fcuda-is-device -fsyntax-only -verify %s

#include <hip/hip_runtime.h>

// Check trivial ctor/dtor
struct A {
  int x;
  A() {}
  ~A() {}
};

__device__ A a;

// Check trivial ctor/dtor of template class
template<typename T>
struct TA {
  T x;
  TA() {}
  ~TA() {}
};

__device__ TA<int> ta;

// Check non-trivial ctor/dtor in parent template class
template<typename T>
struct TB {
  T x;
  TB() { static int nontrivial_ctor = 1; }
  ~TB() {}
};

template<typename T>
struct TC : TB<T> {
  T x;
  TC() {}
  ~TC() {}
};

__device__ TC<int> tc; //expected-error {{dynamic initialization is not supported for __device__, __constant__, __shared__, and __managed__ variables}}
