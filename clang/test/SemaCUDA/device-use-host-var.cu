#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -fcuda-is-device -fsyntax-only -verify=dev %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify=host %s

// host-no-diagnostics

#include "Inputs/hip/hip_runtime.h"

int global_host_var;
__device__ int global_dev_var;
__constant__ int global_constant_var;
__shared__ int global_shared_var;
constexpr int global_constexpr_var = 1;
const int global_const_var = 1;

template<typename F>
__global__ void kernel(F f) { f(); } // dev-note2 {{called by 'kernel<(lambda}}

__device__ void dev_fun(int *out) {
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __device__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

__global__ void global_fun(int *out) {
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __global__ function}}
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __global__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

__host__ __device__ void host_dev_fun(int *out) {
  int &ref_host_var = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

inline __host__ __device__ void inline_host_dev_fun(int *out) {
  int &ref_host_var = global_host_var;
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var;
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
}

void dev_lambda_capture_by_ref(int *out) {
  int &ref_host_var = global_host_var;
  kernel<<<1,1>>>([&]() {
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
                          // dev-error@-1 {{capture host variable 'out' by reference in device or host device lambda function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var; // dev-error {{capture host variable 'ref_host_var' by reference in device or host device lambda function}}
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
  });
}

void dev_lambda_capture_by_copy(int *out) {
  int &ref_host_var = global_host_var;
  kernel<<<1,1>>>([=]() {
  int &ref_dev_var = global_dev_var;
  int &ref_constant_var = global_constant_var;
  int &ref_shared_var = global_shared_var;
  const int &ref_constexpr_var = global_constexpr_var;
  const int &ref_const_var = global_const_var;

  *out = global_host_var; // dev-error {{reference to __host__ variable 'global_host_var' in __host__ __device__ function}}
  *out = global_dev_var;
  *out = global_constant_var;
  *out = global_shared_var;
  *out = global_constexpr_var;
  *out = global_const_var;

  *out = ref_host_var;
  *out = ref_dev_var;
  *out = ref_constant_var;
  *out = ref_shared_var;
  *out = ref_constexpr_var;
  *out = ref_const_var;
  });
}

