
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -E -triple x86_64-linux-gnu %s -o - \
// RUN:   | FileCheck -check-prefix=NOHDT %s
// RUN: %clang_cc1 -E -triple x86_64-linux-gnu %s -o - \
// RUN:   -foffload-implicit-host-device-templates \
// RUN:   | FileCheck -check-prefix=HDT %s

// NOHDT: no_implicit_host_device_templates
// HDT: has_implicit_host_device_templates
#if __has_extension(cuda_implicit_host_device_templates)
int has_implicit_host_device_templates();
#else
int no_implicit_host_device_templates();
#endif
