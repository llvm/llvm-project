#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip %s -o - | FileCheck --check-prefixes=COMMON,CHECK %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip %s -disable-O0-optnone -o - | opt -S -O2 | FileCheck %s --check-prefixes=COMMON,OPT
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm -x hip %s -o - | FileCheck -check-prefix=HOST %s

#include "Inputs/hip/hip_runtime.h"

// Coerced struct from `struct S` without all generic pointers lowered into
// global ones.
// COMMON: %struct.S.coerce = type { i32 addrspace(1)*, float addrspace(1)* }
// COMMON: %struct.T.coerce = type { [2 x float addrspace(1)*] }

// On the host-side compilation, generic pointer won't be coerced.
// HOST-NOT: %struct.S.coerce
// HOST-NOT: %struct.T.coerce

// HOST: define void @_Z22__device_stub__kernel1Pi(i32* %x)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel1Pi(i32 addrspace(1)*{{.*}} %x.coerce)
// CHECK:     = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// CHECK-NOT: = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// OPT: [[VAL:%.*]] = load i32, i32 addrspace(1)* %x.coerce, align 4
// OPT: [[INC:%.*]] = add nsw i32 [[VAL]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* %x.coerce, align 4
// OPT: ret void
__global__ void kernel1(int *x) {
  x[0]++;
}

// HOST: define void @_Z22__device_stub__kernel2Ri(i32* nonnull align 4 dereferenceable(4) %x)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel2Ri(i32 addrspace(1)*{{.*}} nonnull align 4 dereferenceable(4) %x.coerce)
// CHECK:     = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// CHECK-NOT: = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
// OPT: [[VAL:%.*]] = load i32, i32 addrspace(1)* %x.coerce, align 4
// OPT: [[INC:%.*]] = add nsw i32 [[VAL]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* %x.coerce, align 4
// OPT: ret void
__global__ void kernel2(int &x) {
  x++;
}

// HOST: define void @_Z22__device_stub__kernel3PU3AS2iPU3AS1i(i32 addrspace(2)* %x, i32 addrspace(1)* %y)
// CHECK-LABEL: define amdgpu_kernel void  @_Z7kernel3PU3AS2iPU3AS1i(i32 addrspace(2)*{{.*}} %x, i32 addrspace(1)*{{.*}} %y)
// CHECK-NOT: = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
__global__ void kernel3(__attribute__((address_space(2))) int *x,
                        __attribute__((address_space(1))) int *y) {
  y[0] = x[0];
}

// COMMON-LABEL: define void @_Z4funcPi(i32*{{.*}} %x)
// CHECK-NOT: = addrspacecast [[TYPE:.*]] addrspace(1)* %{{.*}} to [[TYPE]]*
__device__ void func(int *x) {
  x[0]++;
}

struct S {
  int *x;
  float *y;
};
// `by-val` struct will be coerced into a similar struct with all generic
// pointers lowerd into global ones.
// HOST: define void @_Z22__device_stub__kernel41S(i32* %s.coerce0, float* %s.coerce1)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel41S(%struct.S.coerce %s.coerce)
// OPT: [[P0:%.*]] = extractvalue %struct.S.coerce %s.coerce, 0
// OPT: [[P1:%.*]] = extractvalue %struct.S.coerce %s.coerce, 1
// OPT: [[V0:%.*]] = load i32, i32 addrspace(1)* [[P0]], align 4
// OPT: [[INC:%.*]] = add nsw i32 [[V0]], 1
// OPT: store i32 [[INC]], i32 addrspace(1)* [[P0]], align 4
// OPT: [[V1:%.*]] = load float, float addrspace(1)* [[P1]], align 4
// OPT: [[ADD:%.*]] = fadd contract float [[V1]], 1.000000e+00
// OPT: store float [[ADD]], float addrspace(1)* [[P1]], align 4
// OPT: ret void
__global__ void kernel4(struct S s) {
  s.x[0]++;
  s.y[0] += 1.f;
}

// If a pointer to struct is passed, only the pointer itself is coerced into the global one.
// HOST: define void @_Z22__device_stub__kernel5P1S(%struct.S* %s)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel5P1S(%struct.S addrspace(1)*{{.*}} %s.coerce)
__global__ void kernel5(struct S *s) {
  s->x[0]++;
  s->y[0] += 1.f;
}

struct T {
  float *x[2];
};
// `by-val` array is also coerced.
// HOST: define void @_Z22__device_stub__kernel61T(float* %t.coerce0, float* %t.coerce1)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel61T(%struct.T.coerce %t.coerce)
// OPT: [[ARR:%.*]] = extractvalue %struct.T.coerce %t.coerce, 0
// OPT: [[P0:%.*]] = extractvalue [2 x float addrspace(1)*] [[ARR]], 0
// OPT: [[P1:%.*]] = extractvalue [2 x float addrspace(1)*] [[ARR]], 1
// OPT: [[V0:%.*]] = load float, float addrspace(1)* [[P0]], align 4
// OPT: [[ADD0:%.*]] = fadd contract float [[V0]], 1.000000e+00
// OPT: store float [[ADD0]], float addrspace(1)* [[P0]], align 4
// OPT: [[V1:%.*]] = load float, float addrspace(1)* [[P1]], align 4
// OPT: [[ADD1:%.*]] = fadd contract float [[V1]], 2.000000e+00
// OPT: store float [[ADD1]], float addrspace(1)* [[P1]], align 4
// OPT: ret void
__global__ void kernel6(struct T t) {
  t.x[0][0] += 1.f;
  t.x[1][0] += 2.f;
}

// Check that coerced pointers retain the noalias attribute when qualified with __restrict.
// HOST: define void @_Z22__device_stub__kernel7Pi(i32* noalias %x)
// COMMON-LABEL: define amdgpu_kernel void @_Z7kernel7Pi(i32 addrspace(1)* noalias{{.*}} %x.coerce)
__global__ void kernel7(int *__restrict x) {
  x[0]++;
}
