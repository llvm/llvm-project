#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -x hip %s -o - | FileCheck %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm -x hip %s -o - | FileCheck -check-prefix=HOST %s

#include "Inputs/hip/hip_runtime.h"

// Coerced struct from `struct S` without all generic pointers lowered into
// global ones.
// CHECK: %struct.S.coerce = type { i32 addrspace(1)*, float addrspace(1)* }
// CHECK: %struct.T.coerce = type { [2 x float addrspace(1)*] }

// On the host-side compilation, generic pointer won't be coerced.
// HOST-NOT: %struct.S.coerce
// HOST-NOT: %struct.T.coerce

// CHECK: define amdgpu_kernel void  @_Z7kernel1Pi(i32 addrspace(1)* %x.coerce)
// HOST: define void @_Z22__device_stub__kernel1Pi(i32* %x)
__global__ void kernel1(int *x) {
  x[0]++;
}

// CHECK: define amdgpu_kernel void  @_Z7kernel2Ri(i32 addrspace(1)* dereferenceable(4) %x.coerce)
// HOST: define void @_Z22__device_stub__kernel2Ri(i32* dereferenceable(4) %x)
__global__ void kernel2(int &x) {
  x++;
}

// CHECK: define amdgpu_kernel void  @_Z7kernel3PU3AS2iPU3AS1i(i32 addrspace(2)* %x, i32 addrspace(1)* %y)
// HOST: define void @_Z22__device_stub__kernel3PU3AS2iPU3AS1i(i32 addrspace(2)* %x, i32 addrspace(1)* %y)
__global__ void kernel3(__attribute__((address_space(2))) int *x,
                        __attribute__((address_space(1))) int *y) {
  y[0] = x[0];
}

// CHECK: define void @_Z4funcPi(i32* %x)
__device__ void func(int *x) {
  x[0]++;
}

struct S {
  int *x;
  float *y;
};
// `by-val` struct will be coerced into a similar struct with all generic
// pointers lowerd into global ones.
// CHECK: define amdgpu_kernel void @_Z7kernel41S(%struct.S.coerce %s.coerce)
// HOST: define void @_Z22__device_stub__kernel41S(i32* %s.coerce0, float* %s.coerce1)
__global__ void kernel4(struct S s) {
  s.x[0]++;
  s.y[0] += 1.f;
}

// If a pointer to struct is passed, only the pointer itself is coerced into the global one.
// CHECK: define amdgpu_kernel void @_Z7kernel5P1S(%struct.S addrspace(1)* %s.coerce)
// HOST: define void @_Z22__device_stub__kernel5P1S(%struct.S* %s)
__global__ void kernel5(struct S *s) {
  s->x[0]++;
  s->y[0] += 1.f;
}

struct T {
  float *x[2];
};
// `by-val` array is also coerced.
// CHECK: define amdgpu_kernel void @_Z7kernel61T(%struct.T.coerce %t.coerce)
// HOST: define void @_Z22__device_stub__kernel61T(float* %t.coerce0, float* %t.coerce1)
__global__ void kernel6(struct T t) {
  t.x[0][0] += 1.f;
  t.x[1][0] += 2.f;
}
