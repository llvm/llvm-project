#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -std=c++17 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -aux-triple amdgcn-amd-amdhsa -std=c++17 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.host

// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=HIP,COMMON %s
// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=COMNEG %s

// RUN: echo "GPU binary" > %t.fatbin

// RUN: %clang_cc1 -triple nvptx -fcuda-is-device -cuid=abc \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -std=c++17 -fgpu-rdc \
// RUN:   -emit-llvm -o - %s > %t.dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -aux-triple nvptx -std=c++17 -fgpu-rdc -fcuda-include-gpubinary %t.fatbin \
// RUN:   -emit-llvm -o - %s > %t.host

// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=CUDA,COMMON %s
// RUN: cat %t.dev %t.host | FileCheck -check-prefixes=COMNEG %s

#include "Inputs/hip/hip_runtime.h"

// HIP-DAG: define weak_odr {{.*}}void @[[KERN:_ZN12_GLOBAL__N_16kernelEv\.intern\.b04fd23c98500190]](
// HIP-DAG: define weak_odr {{.*}}void @[[KTX:_Z2ktIN12_GLOBAL__N_11XEEvT_\.intern\.b04fd23c98500190]](
// HIP-DAG: define weak_odr {{.*}}void @[[KTL:_Z2ktIN12_GLOBAL__N_1UlvE_EEvT_\.intern\.b04fd23c98500190]](
// HIP-DAG: @[[VM:_ZN12_GLOBAL__N_12vmE\.static\.b04fd23c98500190]] = addrspace(1) externally_initialized global
// HIP-DAG: @[[VC:_ZN12_GLOBAL__N_12vcE\.static\.b04fd23c98500190]] = addrspace(4) externally_initialized global
// HIP-DAG: @[[VT:_Z2vtIN12_GLOBAL__N_11XEE\.static\.b04fd23c98500190]] = addrspace(1) externally_initialized global

// CUDA-DAG: define weak_odr {{.*}}void @[[KERN:_ZN12_GLOBAL__N_16kernelEv__intern__b04fd23c98500190]](
// CUDA-DAG: define weak_odr {{.*}}void @[[KTX:_Z2ktIN12_GLOBAL__N_11XEEvT___intern__b04fd23c98500190]](
// CUDA-DAG: define weak_odr {{.*}}void @[[KTL:_Z2ktIN12_GLOBAL__N_1UlvE_EEvT___intern__b04fd23c98500190]](
// CUDA-DAG: @[[VC:_ZN12_GLOBAL__N_12vcE__static__b04fd23c98500190]] = addrspace(4) externally_initialized global
// CUDA-DAG: @[[VT:_Z2vtIN12_GLOBAL__N_11XEE__static__b04fd23c98500190]] = addrspace(1) externally_initialized global

// COMMON-DAG: @_ZN12_GLOBAL__N_12vdE = internal addrspace(1) global
// COMNEG-NOT: @{{.*}} = {{.*}} c"_ZN12_GLOBAL__N_12vdE{{.*}}\00"

// HIP-DAG: @llvm.compiler.used = {{.*}}@[[VM]]{{.*}}@[[VT]]{{.*}}@[[VC]]
// CUDA-DAG: @llvm.compiler.used = {{.*}}@[[VT]]{{.*}}@[[VC]]

// COMMON-DAG: @[[KERNSTR:.*]] = {{.*}} c"[[KERN]]\00"
// COMMON-DAG: @[[KTXSTR:.*]] = {{.*}} c"[[KTX]]\00"
// COMMON-DAG: @[[KTLSTR:.*]] = {{.*}} c"[[KTL]]\00"
// HIP-DAG: @[[VMSTR:.*]] = {{.*}} c"[[VM]]\00"
// COMMON-DAG: @[[VCSTR:.*]] = {{.*}} c"[[VC]]\00"
// COMMON-DAG: @[[VTSTR:.*]] = {{.*}} c"[[VT]]\00"

// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[KERNSTR]]
// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[KTXSTR]]
// COMMON-DAG: call i32 @__{{.*}}RegisterFunction({{.*}}@[[KTLSTR]]
// HIP-DAG: call void @__{{.*}}RegisterManagedVar({{.*}}@[[VMSTR]]
// COMMON-DAG: call void @__{{.*}}RegisterVar({{.*}}@[[VCSTR]]
// COMMON-DAG: call void @__{{.*}}RegisterVar({{.*}}@[[VTSTR]]

template <typename T>
__global__ void kt(T x) {}

template <typename T>
__device__ T vt;

namespace {
  struct X {};
  X x;
  auto lambda = [](){};
#if __HIP__
  __managed__ int vm = 1;
#endif
  __constant__ int vc = 2;

  // C should not be externalized since it is used by device code only.
  __device__ int vd = 3;
  __global__ void kernel() { vd = 4; }
}

template<typename T>
void getSymbol(T *x) {}

void test() {
  kernel<<<1, 1>>>();

  kt<<<1, 1>>>(x);

  kt<<<1, 1>>>(lambda);

  // A, B, and tempVar<X> should be externalized since they are
  // used by host code.
#if __HIP__
  getSymbol(&vm);
#endif
  getSymbol(&vc);
  getSymbol(&vt<X>);
}
