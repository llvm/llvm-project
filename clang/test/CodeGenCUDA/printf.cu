// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm \
// RUN:   -o - %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

extern "C" __device__ int vprintf(const char*, const char*);

// Check a simple call to printf end-to-end.
// CHECK: [[SIMPLE_PRINTF_TY:%[a-zA-Z0-9_]+]] = type { i32, i64, double }
__device__ int CheckSimple() {
  // CHECK: [[BUF:%[a-zA-Z0-9_]+]] = alloca [[SIMPLE_PRINTF_TY]]
  // CHECK: [[FMT:%[0-9]+]] = load{{.*}}%fmt
  const char* fmt = "%d %lld %f";
  // CHECK: [[PTR0:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], ptr [[BUF]], i32 0, i32 0
  // CHECK: store i32 1, ptr [[PTR0]], align 4
  // CHECK: [[PTR1:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], ptr [[BUF]], i32 0, i32 1
  // CHECK: store i64 2, ptr [[PTR1]], align 8
  // CHECK: [[PTR2:%[0-9]+]] = getelementptr inbounds [[SIMPLE_PRINTF_TY]], ptr [[BUF]], i32 0, i32 2
  // CHECK: store double 3.0{{[^,]*}}, ptr [[PTR2]], align 8
  // CHECK: [[RET:%[0-9]+]] = call i32 @vprintf(ptr [[FMT]], ptr [[BUF]])
  // CHECK: ret i32 [[RET]]
  return printf(fmt, 1, 2ll, 3.0);
}

__device__ void CheckNoArgs() {
  // CHECK: call i32 @vprintf({{.*}}, ptr null){{$}}
  printf("hello, world!");
}

// Check that printf's alloca happens in the entry block, not inside the if
// statement.
__device__ bool foo();
__device__ void CheckAllocaIsInEntryBlock() {
  // CHECK: alloca %printf_args
  // CHECK: call {{.*}} @_Z3foov()
  if (foo()) {
    printf("%d", 42);
  }
}
