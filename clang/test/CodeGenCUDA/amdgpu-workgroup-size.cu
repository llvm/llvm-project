// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefix=PRECOV5 %s


// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -mcode-object-version=5 -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefix=COV5 %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -mcode-object-version=none -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefix=COVNONE %s

#include "Inputs/hip/hip_runtime.h"

// PRECOV5-LABEL: test_get_workgroup_size
// PRECOV5: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// PRECOV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// PRECOV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// PRECOV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 6
// PRECOV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// PRECOV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 8
// PRECOV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// COV5-LABEL: test_get_workgroup_size
// COV5: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// COV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// COV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// COV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 14
// COV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// COV5: getelementptr i8, ptr addrspace(4) %{{.*}}, i32 16
// COV5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef


// COVNONE-LABEL: test_get_workgroup_size
// COVNONE: load i32, ptr addrspace(4) @llvm.amdgcn.abi.version
// COVNONE: [[ABI5_X:%.*]] = icmp sge i32 %{{.*}}, 500
// COVNONE: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// COVNONE: [[GEP_5_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// COVNONE: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// COVNONE: [[GEP_4_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// COVNONE: select i1 [[ABI5_X]], ptr addrspace(4) [[GEP_5_X]], ptr addrspace(4) [[GEP_4_X]]
// COVNONE: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// COVNONE: load i32, ptr addrspace(4) @llvm.amdgcn.abi.version
// COVNONE: [[ABI5_Y:%.*]] = icmp sge i32 %{{.*}}, 500
// COVNONE: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// COVNONE: [[GEP_5_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 14
// COVNONE: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// COVNONE: [[GEP_4_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 6
// COVNONE: select i1 [[ABI5_Y]], ptr addrspace(4) [[GEP_5_Y]], ptr addrspace(4) [[GEP_4_Y]]
// COVNONE: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// COVNONE: load i32, ptr addrspace(4) @llvm.amdgcn.abi.version
// COVNONE: [[ABI5_Z:%.*]] = icmp sge i32 %{{.*}}, 500
// COVNONE: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// COVNONE: [[GEP_5_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 16
// COVNONE: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// COVNONE: [[GEP_4_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 8
// COVNONE: select i1 [[ABI5_Z]], ptr addrspace(4) [[GEP_5_Z]], ptr addrspace(4) [[GEP_4_Z]]
// COVNONE: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

__device__ void test_get_workgroup_size(int d, int *out)
{
  switch (d) {
  case 0: *out = __builtin_amdgcn_workgroup_size_x(); break;
  case 1: *out = __builtin_amdgcn_workgroup_size_y(); break;
  case 2: *out = __builtin_amdgcn_workgroup_size_z(); break;
  default: *out = 0;
  }
}

// CHECK-DAG: [[$WS_RANGE]] = !{i16 1, i16 1025}
