#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -std=c++11 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.dev

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -aux-triple amdgcn-amd-amdhsa -std=c++11 -fgpu-rdc \
// RUN:   -emit-llvm -o - -x hip %s > %t.host

// RUN: cat %t.dev %t.host | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: define weak_odr {{.*}}void @[[KERN:_ZN12_GLOBAL__N_16kernelEv\.anon\.b04fd23c98500190]](
// CHECK: @[[STR:.*]] = {{.*}} c"[[KERN]]\00"
// CHECK: call i32 @__hipRegisterFunction({{.*}}@[[STR]]

namespace {
__global__ void kernel() {
}
}

void test() {
  kernel<<<1, 1>>>();
}
