
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -target-cpu gfx942 \
// RUN:   -emit-llvm -xhip %s -o - | FileCheck %s --check-prefix=GCN
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64-amd-amdhsa \
// RUN:   -emit-llvm -xhip %s -o - | FileCheck %s --check-prefix=SPIRV

// GCN: @_ZTV1C = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)], [4 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1C2f1Ev to ptr addrspace(1))], [4 x ptr addrspace(1)] [ptr addrspace(1) inttoptr (i64 -8 to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZThn8_N1C2f1Ev to ptr addrspace(1))] }, comdat, align 8
// GCN: @_ZTV1B = linkonce_odr unnamed_addr addrspace(1) constant { [3 x ptr addrspace(1)] } { [3 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1))] }, comdat, align 8
// GCN: @_ZTV1A = linkonce_odr unnamed_addr addrspace(1) constant { [4 x ptr addrspace(1)] } { [4 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @_ZTV1C = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)], [4 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1B2f2Ev to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1C2f1Ev to ptr addrspace(1))], [4 x ptr addrspace(1)] [ptr addrspace(1) inttoptr (i64 -8 to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZThn8_N1C2f1Ev to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @_ZTV1B = linkonce_odr unnamed_addr addrspace(1) constant { [3 x ptr addrspace(1)] } { [3 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1B2f2Ev to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @_ZTV1A = linkonce_odr unnamed_addr addrspace(1) constant { [4 x ptr addrspace(1)] } { [4 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8

struct A {
  __attribute__((device)) A() { }
  virtual void neither_device_nor_host_f() = 0 ;
  __attribute__((device)) virtual void f1() = 0;
 
};
 
struct B {
  __attribute__((device)) B() { }
  __attribute__((device)) virtual void f2() { };
};
 
struct C : public B, public A {
  __attribute__((device)) C() : B(), A() { }
 
   virtual void neither_device_nor_host_f() override { }
  __attribute__((device)) virtual void f1() override { }
 
};
 
__attribute__((device)) void test() {
  C obj;
}
