
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -target-cpu gfx942 \
// RUN:   -emit-llvm -xhip %s -o - | FileCheck %s

// CHECK: @_ZTV1C = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)], [4 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1C2f1Ev to ptr addrspace(1))], [4 x ptr addrspace(1)] [ptr addrspace(1) inttoptr (i64 -8 to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1C2f1Ev to ptr addrspace(1))] }, comdat, align 8
// CHECK: @_ZTV1B = linkonce_odr unnamed_addr addrspace(1) constant { [3 x ptr addrspace(1)] } { [3 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1))] }, comdat, align 8
// CHECK: @_ZTV1A = linkonce_odr unnamed_addr addrspace(1) constant { [4 x ptr addrspace(1)] } { [4 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8
// CHECK: @__hip_cuid_ = addrspace(1) global i8 0
// CHECK: @llvm.compiler.used = appending addrspace(1) global [1 x ptr] [ptr addrspacecast (ptr addrspace(1) @__hip_cuid_ to ptr)], section "llvm.metadata"
// CHECK: @__oclc_ABI_version = weak_odr hidden local_unnamed_addr addrspace(4) constant i32 500

struct A {
  __attribute__((device)) A() { }
  virtual void neither_device_nor_host_f() = 0 ;
  __attribute__((device)) virtual void f1() = 0;
 
};
 
struct B {
  __attribute__((device)) B() { }
  __attribute__((device)) virtual void f2() { };
};
 
struct C : public B, public A {
  __attribute__((device)) C() : B(), A() { }
 
   virtual void neither_device_nor_host_f() override { }
  __attribute__((device)) virtual void f1() override { }
 
};
 
__attribute__((device)) void test() {
  C obj;
}
