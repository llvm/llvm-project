
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -target-cpu gfx942 \
// RUN:   -emit-llvm -xhip %s -o - | FileCheck %s --check-prefix=GCN
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64-amd-amdhsa \
// RUN:   -emit-llvm -xhip %s -o - | FileCheck %s --check-prefix=SPIRV

// GCN: @_ZTV1C = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)], [4 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1C2f1Ev to ptr addrspace(1))], [4 x ptr addrspace(1)] [ptr addrspace(1) inttoptr (i64 -8 to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1C2f1Ev to ptr addrspace(1))] }, comdat, align 8
// GCN: @_ZTV1B = linkonce_odr unnamed_addr addrspace(1) constant { [3 x ptr addrspace(1)] } { [3 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1B2f2Ev to ptr addrspace(1))] }, comdat, align 8
// GCN: @_ZTV1A = linkonce_odr unnamed_addr addrspace(1) constant { [4 x ptr addrspace(1)] } { [4 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8
// GCN: @__hip_cuid_ = addrspace(1) global i8 0
// GCN: @llvm.compiler.used = appending addrspace(1) global [1 x ptr] [ptr addrspacecast (ptr addrspace(1) @__hip_cuid_ to ptr)], section "llvm.metadata"
// GCN: @__oclc_ABI_version = weak_odr hidden local_unnamed_addr addrspace(4) constant i32 500

// SPIRV: @_ZTV1C = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)], [4 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1B2f2Ev to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1C2f1Ev to ptr addrspace(1))], [4 x ptr addrspace(1)] [ptr addrspace(1) inttoptr (i64 -8 to ptr addrspace(1)), ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1C2f1Ev to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @_ZTV1B = linkonce_odr unnamed_addr addrspace(1) constant { [3 x ptr addrspace(1)] } { [3 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @_ZN1B2f2Ev to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @_ZTV1A = linkonce_odr unnamed_addr addrspace(1) constant { [4 x ptr addrspace(1)] } { [4 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr addrspace(4) @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8
// SPIRV: @__hip_cuid_ = addrspace(1) global i8 0
// SPIRV: @llvm.compiler.used = appending addrspace(1) global [1 x ptr addrspace(4)] [ptr addrspace(4) addrspacecast (ptr addrspace(1) @__hip_cuid_ to ptr addrspace(4))], section "llvm.metadata"

struct A {
  __attribute__((device)) A() { }
  virtual void neither_device_nor_host_f() = 0 ;
  __attribute__((device)) virtual void f1() = 0;
 
};
 
struct B {
  __attribute__((device)) B() { }
  __attribute__((device)) virtual void f2() { };
};
 
struct C : public B, public A {
  __attribute__((device)) C() : B(), A() { }
 
   virtual void neither_device_nor_host_f() override { }
  __attribute__((device)) virtual void f1() override { }
 
};
 
__attribute__((device)) void test() {
  C obj;
}
