#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -emit-llvm %s -o - | FileCheck %s
#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: @_Z16use_dispatch_ptrPi(
// CHECK: %[[PTR:.*]] = call align 4 dereferenceable(64) i8 addrspace(4)* @llvm.amdgcn.dispatch.ptr()
// CHECK: %{{.*}} = addrspacecast i8 addrspace(4)* %[[PTR]] to i8*
__global__ void use_dispatch_ptr(int* out) {
  const int* dispatch_ptr = (const int*)__builtin_amdgcn_dispatch_ptr();
  *out = *dispatch_ptr;
}

// CHECK-LABEL: @_Z12test_ds_fmaxf(
// CHECK: call contract float @llvm.amdgcn.ds.fmax.f32(float addrspace(3)* @_ZZ12test_ds_fmaxfE6shared, float %{{[^,]*}}, i32 0, i32 0, i1 false)
__global__
void test_ds_fmax(float src) {
  __shared__ float shared;
  volatile float x = __builtin_amdgcn_ds_fmaxf(&shared, src, 0, 0, false);
}
