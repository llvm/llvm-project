#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -target-cpu gfx906 \
// RUN:  -aux-triple x86_64-unknown-linux-gnu -fcuda-is-device -emit-llvm %s \
// RUN:  -o - | FileCheck %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -target-cpu gfx906 \
// RUN:  -aux-triple x86_64-pc-windows-msvc -fcuda-is-device -emit-llvm %s \
// RUN:  -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: @_Z16use_dispatch_ptrPi(
// CHECK: %[[PTR:.*]] = call align 4 dereferenceable(64) i8 addrspace(4)* @llvm.amdgcn.dispatch.ptr()
// CHECK: %{{.*}} = addrspacecast i8 addrspace(4)* %[[PTR]] to i8*
__global__ void use_dispatch_ptr(int* out) {
  const int* dispatch_ptr = (const int*)__builtin_amdgcn_dispatch_ptr();
  *out = *dispatch_ptr;
}

// CHECK-LABEL: @_Z12test_ds_fmaxf(
// CHECK: call contract float @llvm.amdgcn.ds.fmax.f32(float addrspace(3)* @_ZZ12test_ds_fmaxfE6shared, float %{{[^,]*}}, i32 0, i32 0, i1 false)
__global__
void test_ds_fmax(float src) {
  __shared__ float shared;
  volatile float x = __builtin_amdgcn_ds_fmaxf(&shared, src, 0, 0, false);
}

// CHECK-LABEL: @_Z6endpgmv(
// CHECK: call void @llvm.amdgcn.endpgm()
__global__ void endpgm() {
  __builtin_amdgcn_endpgm();
}

// Check the 64 bit argument is correctly passed to the intrinsic without truncation or assertion.

// CHECK-LABEL: @_Z14test_uicmp_i64
// CHECK:  store i64* %out, i64** %out.addr.ascast
// CHECK-NEXT:  store i64 %a, i64* %a.addr.ascast
// CHECK-NEXT:  store i64 %b, i64* %b.addr.ascast
// CHECK-NEXT:  %[[V0:.*]] = load i64, i64* %a.addr.ascast
// CHECK-NEXT:  %[[V1:.*]] = load i64, i64* %b.addr.ascast
// CHECK-NEXT:  %[[V2:.*]] = call i64 @llvm.amdgcn.icmp.i64.i64(i64 %0, i64 %1, i32 35)
// CHECK-NEXT:  %[[V3:.*]] = load i64*, i64** %out.addr.ascast
// CHECK-NEXT:  store i64 %[[V2]], i64* %[[V3]]
// CHECK-NEXT:  ret void
__global__ void test_uicmp_i64(unsigned long long *out, unsigned long long a, unsigned long long b)
{
  *out = __builtin_amdgcn_uicmpl(a, b, 30+5);
}

// Check the 64 bit return value is correctly returned without truncation or assertion.

// CHECK-LABEL: @_Z14test_s_memtime
// CHECK: %[[V1:.*]] = call i64 @llvm.amdgcn.s.memtime()
// CHECK-NEXT: %[[PTR:.*]] = load i64*, i64** %out.addr.ascast
// CHECK-NEXT:  store i64 %[[V1]], i64* %[[PTR]]
// CHECK-NEXT:  ret void
__global__ void test_s_memtime(unsigned long long* out)
{
  *out = __builtin_amdgcn_s_memtime();
}
