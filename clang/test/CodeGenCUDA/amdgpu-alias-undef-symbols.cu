// REQUIRES: amdgpu-registered-target

// RUN: %clang -target x86_64-unknown-linux-gnu --offload-arch=gfx906 --cuda-device-only -nogpulib -nogpuinc -x hip -emit-llvm -S -o - %s \
// RUN:   -fgpu-rdc -O3 -mllvm -amdgpu-early-inline-all=true -mllvm -amdgpu-function-calls=false | \
// RUN:   FileCheck %s

#include "Inputs/hip/hip_runtime.h"

struct B {

  // CHECK: @_ZN1BC1Ei = hidden unnamed_addr alias void (ptr, i32), ptr @_ZN1BC2Ei
  __device__ B(int x);
};

__device__ B::B(int x) {
}
