#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST %s

#include "Inputs/hip/hip_runtime.h"

// Test function scope static device variable, which should not be externalized.
// DEV-DAG: @_ZZ6kernelPiPPKiE1w = internal addrspace(4) constant i32 1

// Check a static device variable referenced by host function is externalized.
// DEV-DAG: @_ZL1x = addrspace(1) externally_initialized global i32 0
// HOST-DAG: @_ZL1x = internal global i32 undef
// HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x\00"

static __device__ int x;

// Check a static device variables referenced only by device functions and kernels
// is not externalized.
// DEV-DAG: @_ZL2x2 = internal addrspace(1) global i32 0
static __device__ int x2;

// Check a static device variable referenced by host device function is externalized.
// DEV-DAG: @_ZL2x3 = addrspace(1) externally_initialized global i32 0
static __device__ int x3;

// Check a static device variable referenced in file scope is externalized.
// DEV-DAG: @_ZL2x4 = addrspace(1) externally_initialized global i32 0
static __device__ int x4;
int& x4_ref = x4;

// Check a static device variable in anonymous namespace.
// DEV-DAG: @_ZN12_GLOBAL__N_12x5E = addrspace(1) externally_initialized global i32 0
namespace {
static __device__ int x5;
}

// Check a static constant variable referenced by host is externalized.
// DEV-DAG: @_ZL1y = addrspace(4) externally_initialized global i32 0
// HOST-DAG: @_ZL1y = internal global i32 undef
// HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y\00"

static __constant__ int y;

// Test static host variable, which should not be externalized nor registered.
// HOST-DAG: @_ZL1z = internal global i32 0
// DEV-NOT: @_ZL1z
static int z;

// Test implicit static constant variable, which should not be externalized.
// HOST-DAG: @_ZL2z2 = internal constant i32 456
// DEV-DAG: @_ZL2z2 = internal addrspace(4) constant i32 456

static constexpr int z2 = 456;

// Test static device variable in inline function, which should not be
// externalized nor registered.
// DEV-DAG: @_ZZ6devfunPPKiE1p = linkonce_odr addrspace(4) constant i32 2, comdat

inline __device__ void devfun(const int ** b) {
  const static int p = 2;
  b[0] = &p;
  b[1] = &x2;
}

__global__ void kernel(int *a, const int **b) {
  const static int w = 1;
  a[0] = x;
  a[1] = y;
  a[2] = x2;
  a[3] = x3;
  a[4] = x4;
  a[5] = x5;
  b[0] = &w;
  b[1] = &z2;
  devfun(b);
}

__host__ __device__ void hdf(int *a) {
  a[0] = x3;
}

int* getDeviceSymbol(int *x);

void foo(const int **a) {
  getDeviceSymbol(&x);
  getDeviceSymbol(&x5);
  getDeviceSymbol(&y);
  z = 123;
  a[0] = &z2;
}

// HOST: __hipRegisterVar({{.*}}@_ZL1x {{.*}}@[[DEVNAMEX]]
// HOST: __hipRegisterVar({{.*}}@_ZL1y {{.*}}@[[DEVNAMEY]]
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6kernelPiPPKiE1w
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6devfunPPKiE1p
