// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck -check-prefix=DEV-NEG %s

#include "Inputs/hip/hip_runtime.h"

template <class T>
class A {
    static int h_member;
    __device__ static int d_member;
    __constant__ static int c_member;
    __managed__ static int m_member;
    const static int const_member = 0;
};

template <class T>
int A<T>::h_member;

template <class T>
__device__ int A<T>::d_member;

template <class T>
__constant__ int A<T>::c_member;

template <class T>
__managed__ int A<T>::m_member;

template <class T>
const int A<T>::const_member;

template class A<int>;

//DEV-DAG: @_ZN1AIiE8d_memberE = internal addrspace(1) global i32 0, comdat, align 4
//DEV-DAG: @_ZN1AIiE8c_memberE = internal addrspace(4) constant i32 0, comdat, align 4
//DEV-DAG: @_ZN1AIiE8m_memberE = internal addrspace(1) externally_initialized global ptr addrspace(1) null
//DEV-DAG: @_ZN1AIiE12const_memberE = internal addrspace(4) constant i32 0, comdat, align 4
//DEV-NEG-NOT: @_ZN1AIiE8h_memberE

//HOST-DAG: @_ZN1AIiE8h_memberE = weak_odr global i32 0, comdat, align 4
//HOST-DAG: @_ZN1AIiE8d_memberE = internal global i32 undef, comdat, align 4
//HOST-DAG: @_ZN1AIiE8c_memberE = internal global i32 undef, comdat, align 4
//HOST-DAG: @_ZN1AIiE8m_memberE = internal externally_initialized global ptr null
//HOST-DAG: @_ZN1AIiE12const_memberE = weak_odr constant i32 0, comdat, align 4
