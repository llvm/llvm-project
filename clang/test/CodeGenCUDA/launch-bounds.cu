#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -target-cpu sm_90 -DUSE_MAX_BLOCKS -fcuda-is-device -emit-llvm -o - | FileCheck -check-prefix=CHECK_MAX_BLOCKS %s

#include "Inputs/hip/hip_runtime.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_BLOCKS_PER_MP     2
#ifdef USE_MAX_BLOCKS
#define MAX_BLOCKS_PER_MP     4
#endif

// CHECK: @Kernel1() #[[ATTR0:[0-9]+]]
// CHECK: @{{.*}}Kernel4{{.*}}() #[[ATTR0]]
// CHECK: @{{.*}}Kernel5{{.*}}() #[[ATTR1:[0-9]+]]
// CHECK: @{{.*}}Kernel6{{.*}}() #[[ATTR0]]
// CHECK: @{{.*}}Kernel8{{.*}}() #[[ATTR3:[0-9]+]]

// CHECK: attributes #[[ATTR0]] = {{{.*}} "nvvm.minctasm"="2" {{.*}}}
// CHECK: attributes #[[ATTR1]] = {{{.*}} "nvvm.minctasm"="258" {{.*}}}
// CHECK: attributes #[[ATTR3]] = {{{.*}} "nvvm.minctasm"="12" {{.*}}}

// CHECK_MAX_BLOCKS: @Kernel1_sm_90() #[[ATTR4:[0-9]+]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel4_sm_90{{.*}} #[[ATTR4]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel5_sm_90{{.*}} #[[ATTR5:[0-9]+]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel8_sm_90{{.*}} #[[ATTR6:[0-9]+]]

// CHECK_MAX_BLOCKS: attributes #[[ATTR4]] = {{{.*}} "nvvm.maxclusterrank"="4" "nvvm.minctasm"="2" {{.*}}}
// CHECK_MAX_BLOCKS: attributes #[[ATTR5]] = {{{.*}} "nvvm.maxclusterrank"="260" "nvvm.minctasm"="258" {{.*}}}
// CHECK_MAX_BLOCKS: attributes #[[ATTR6]] = {{{.*}} "nvvm.maxclusterrank"="14" "nvvm.minctasm"="12" {{.*}}}

// Test both max threads per block and Min cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel1()
{
}
}

// CHECK: !{{[0-9]+}} = !{ptr @Kernel1, !"maxntidx", i32 256}

#ifdef USE_MAX_BLOCKS
// Test max threads per block and min/max cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP )
Kernel1_sm_90()
{
}
}

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @Kernel1_sm_90, !"maxntidx", i32 256}
#endif // USE_MAX_BLOCKS

// Test only max threads per block. Min cta per sm defaults to 0, and
// CodeGen doesn't output a zero value for minctasm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK )
Kernel2()
{
}
}

// CHECK: !{{[0-9]+}} = !{ptr @Kernel2, !"maxntidx", i32 256}

template <int max_threads_per_block>
__global__ void
__launch_bounds__(max_threads_per_block)
Kernel3()
{
}

template __global__ void Kernel3<MAX_THREADS_PER_BLOCK>();
// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel3{{.*}}, !"maxntidx", i32 256}

template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp)
Kernel4()
{
}
template __global__ void Kernel4<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4{{.*}}, !"maxntidx", i32 256}

#ifdef USE_MAX_BLOCKS
template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp, max_blocks_per_mp)
Kernel4_sm_90()
{
}
template __global__ void Kernel4_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4_sm_90{{.*}}, !"maxntidx", i32 256}
#endif //USE_MAX_BLOCKS

const int constint = 100;
template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block)
Kernel5()
{
}
template __global__ void Kernel5<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5{{.*}}, !"maxntidx", i32 356}

#ifdef USE_MAX_BLOCKS

template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block,
                  max_blocks_per_mp + max_threads_per_block)
Kernel5_sm_90()
{
}
template __global__ void Kernel5_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5_sm_90{{.*}}, !"maxntidx", i32 356}
#endif //USE_MAX_BLOCKS

// Make sure we don't emit negative launch bounds values.
__global__ void
__launch_bounds__( -MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel6()
{
}
// CHECK-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel6{{.*}}, !"maxntidx",

__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP )
Kernel7()
{
}
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel7{{.*}}, !"maxntidx",
// CHECK-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7{{.*}}, !"minctasm",

#ifdef USE_MAX_BLOCKS
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP, -MAX_BLOCKS_PER_MP )
Kernel7_sm_90()
{
}
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"maxntidx",
// CHECK_MAX_BLOCKS-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"minctasm",
// CHECK_MAX_BLOCKS-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"maxclusterrank",
#endif // USE_MAX_BLOCKS

const char constchar = 12;
__global__ void __launch_bounds__(constint, constchar) Kernel8() {}
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8{{.*}}, !"maxntidx", i32 100

#ifdef USE_MAX_BLOCKS
const char constchar_2 = 14;
__global__ void __launch_bounds__(constint, constchar, constchar_2) Kernel8_sm_90() {}
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8_sm_90{{.*}}, !"maxntidx", i32 100
#endif // USE_MAX_BLOCKS
