#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -target-cpu sm_90 -DUSE_MAX_BLOCKS -fcuda-is-device -emit-llvm -o - | FileCheck -check-prefix=CHECK_MAX_BLOCKS %s

#include "Inputs/hip/hip_runtime.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_BLOCKS_PER_MP     2
#ifdef USE_MAX_BLOCKS
#define MAX_BLOCKS_PER_MP     4
#endif

// Test both max threads per block and Min cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel1()
{
}
}

// CHECK: !{{[0-9]+}} = !{ptr @Kernel1, !"maxntidx", i32 256}
// CHECK: !{{[0-9]+}} = !{ptr @Kernel1, !"minctasm", i32 2}

#ifdef USE_MAX_BLOCKS
// Test max threads per block and min/max cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP )
Kernel1_sm_90()
{
}
}

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @Kernel1_sm_90, !"maxntidx", i32 256}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @Kernel1_sm_90, !"minctasm", i32 2}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @Kernel1_sm_90, !"maxclusterrank", i32 4}
#endif // USE_MAX_BLOCKS

// Test only max threads per block. Min cta per sm defaults to 0, and
// CodeGen doesn't output a zero value for minctasm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK )
Kernel2()
{
}
}

// CHECK: !{{[0-9]+}} = !{ptr @Kernel2, !"maxntidx", i32 256}

template <int max_threads_per_block>
__global__ void
__launch_bounds__(max_threads_per_block)
Kernel3()
{
}

template __global__ void Kernel3<MAX_THREADS_PER_BLOCK>();
// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel3{{.*}}, !"maxntidx", i32 256}

template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp)
Kernel4()
{
}
template __global__ void Kernel4<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4{{.*}}, !"maxntidx", i32 256}
// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4{{.*}}, !"minctasm", i32 2}

#ifdef USE_MAX_BLOCKS
template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp, max_blocks_per_mp)
Kernel4_sm_90()
{
}
template __global__ void Kernel4_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4_sm_90{{.*}}, !"maxntidx", i32 256}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4_sm_90{{.*}}, !"minctasm", i32 2}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel4_sm_90{{.*}}, !"maxclusterrank", i32 4}
#endif //USE_MAX_BLOCKS

const int constint = 100;
template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block)
Kernel5()
{
}
template __global__ void Kernel5<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5{{.*}}, !"maxntidx", i32 356}
// CHECK: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5{{.*}}, !"minctasm", i32 258}

#ifdef USE_MAX_BLOCKS

template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block,
                  max_blocks_per_mp + max_threads_per_block)
Kernel5_sm_90()
{
}
template __global__ void Kernel5_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5_sm_90{{.*}}, !"maxntidx", i32 356}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5_sm_90{{.*}}, !"minctasm", i32 258}
// CHECK_MAX_BLOCKS: !{{[0-9]+}} = !{ptr @{{.*}}Kernel5_sm_90{{.*}}, !"maxclusterrank", i32 260}
#endif //USE_MAX_BLOCKS

// Make sure we don't emit negative launch bounds values.
__global__ void
__launch_bounds__( -MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel6()
{
}
// CHECK-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel6{{.*}}, !"maxntidx",
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel6{{.*}}, !"minctasm",

__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP )
Kernel7()
{
}
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel7{{.*}}, !"maxntidx",
// CHECK-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7{{.*}}, !"minctasm",

#ifdef USE_MAX_BLOCKS
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP, -MAX_BLOCKS_PER_MP )
Kernel7_sm_90()
{
}
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"maxntidx",
// CHECK_MAX_BLOCKS-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"minctasm",
// CHECK_MAX_BLOCKS-NOT: !{{[0-9]+}} = !{ptr @{{.*}}Kernel7_sm_90{{.*}}, !"maxclusterrank",
#endif // USE_MAX_BLOCKS

const char constchar = 12;
__global__ void __launch_bounds__(constint, constchar) Kernel8() {}
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8{{.*}}, !"maxntidx", i32 100
// CHECK:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8{{.*}}, !"minctasm", i32 12

#ifdef USE_MAX_BLOCKS
const char constchar_2 = 14;
__global__ void __launch_bounds__(constint, constchar, constchar_2) Kernel8_sm_90() {}
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8_sm_90{{.*}}, !"maxntidx", i32 100
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8_sm_90{{.*}}, !"minctasm", i32 12
// CHECK_MAX_BLOCKS:     !{{[0-9]+}} = !{ptr @{{.*}}Kernel8_sm_90{{.*}}, !"maxclusterrank", i32 14
#endif // USE_MAX_BLOCKS
