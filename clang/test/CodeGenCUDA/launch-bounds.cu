#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -fcuda-is-device -emit-llvm -o - | FileCheck %s
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -target-cpu sm_90 -DUSE_MAX_BLOCKS -fcuda-is-device -emit-llvm -o - | FileCheck -check-prefix=CHECK_MAX_BLOCKS %s

#include "Inputs/hip/hip_runtime.h"

#define MAX_THREADS_PER_BLOCK 256
#define MIN_BLOCKS_PER_MP     2
#ifdef USE_MAX_BLOCKS
#define MAX_BLOCKS_PER_MP     4
#endif

// CHECK: @Kernel1() #[[ATTR0:[0-9]+]]
// CHECK: @Kernel2() #[[ATTR1:[0-9]+]]
// CHECK: @{{.*}}Kernel3{{.*}}() #[[ATTR1]]
// CHECK: @{{.*}}Kernel4{{.*}}() #[[ATTR0]]
// CHECK: @{{.*}}Kernel5{{.*}}() #[[ATTR2:[0-9]+]]
// CHECK: @{{.*}}Kernel6{{.*}}() #[[ATTR3:[0-9]+]]
// CHECK: @{{.*}}Kernel7{{.*}}() #[[ATTR1]]
// CHECK: @{{.*}}Kernel8{{.*}}() #[[ATTR4:[0-9]+]]

// CHECK-DAG: attributes #[[ATTR0]] = {{{.*}} "nvvm.maxntid"="256" "nvvm.minctasm"="2" {{.*}}}
// CHECK-DAG: attributes #[[ATTR1]] = {{{.*}} "nvvm.maxntid"="256" {{.*}}}
// CHECK-DAG: attributes #[[ATTR2]] = {{{.*}} "nvvm.maxntid"="356" "nvvm.minctasm"="258" {{.*}}}
// CHECK-DAG: attributes #[[ATTR3]] = {{{.*}} "nvvm.minctasm"="2" {{.*}}}
// CHECK-DAG: attributes #[[ATTR4]] = {{{.*}} "nvvm.maxntid"="100" "nvvm.minctasm"="12" {{.*}}}

// CHECK_MAX_BLOCKS: @Kernel1_sm_90() #[[ATTR0:[0-9]+]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel4_sm_90{{.*}} #[[ATTR0]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel5_sm_90{{.*}} #[[ATTR1:[0-9]+]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel7_sm_90{{.*}} #[[ATTR2:[0-9]+]]
// CHECK_MAX_BLOCKS: @{{.*}}Kernel8_sm_90{{.*}} #[[ATTR3:[0-9]+]]

// CHECK_MAX_BLOCKS-DAG: attributes #[[ATTR0]] = {{{.*}} "nvvm.maxclusterrank"="4" "nvvm.maxntid"="256" "nvvm.minctasm"="2" {{.*}}}
// CHECK_MAX_BLOCKS-DAG: attributes #[[ATTR1]] = {{{.*}} "nvvm.maxclusterrank"="260" "nvvm.maxntid"="356" "nvvm.minctasm"="258" {{.*}}}
// CHECK_MAX_BLOCKS-DAG: attributes #[[ATTR2]] = {{{.*}} "nvvm.maxntid"="256" {{.*}}}
// CHECK_MAX_BLOCKS-DAG: attributes #[[ATTR3]] = {{{.*}} "nvvm.maxclusterrank"="14" "nvvm.maxntid"="100" "nvvm.minctasm"="12" {{.*}}}

// Test both max threads per block and Min cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel1()
{
}
}

#ifdef USE_MAX_BLOCKS
// Test max threads per block and min/max cta per sm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP )
Kernel1_sm_90()
{
}
}
#endif // USE_MAX_BLOCKS

// Test only max threads per block. Min cta per sm defaults to 0, and
// CodeGen doesn't output a zero value for minctasm.
extern "C" {
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK )
Kernel2()
{
}
}

template <int max_threads_per_block>
__global__ void
__launch_bounds__(max_threads_per_block)
Kernel3()
{
}

template __global__ void Kernel3<MAX_THREADS_PER_BLOCK>();

template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp)
Kernel4()
{
}
template __global__ void Kernel4<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();


#ifdef USE_MAX_BLOCKS
template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block, min_blocks_per_mp, max_blocks_per_mp)
Kernel4_sm_90()
{
}
template __global__ void Kernel4_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

#endif //USE_MAX_BLOCKS

const int constint = 100;
template <int max_threads_per_block, int min_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block)
Kernel5()
{
}
template __global__ void Kernel5<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP>();

#ifdef USE_MAX_BLOCKS

template <int max_threads_per_block, int min_blocks_per_mp, int max_blocks_per_mp>
__global__ void
__launch_bounds__(max_threads_per_block + constint,
                  min_blocks_per_mp + max_threads_per_block,
                  max_blocks_per_mp + max_threads_per_block)
Kernel5_sm_90()
{
}
template __global__ void Kernel5_sm_90<MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP, MAX_BLOCKS_PER_MP>();

#endif //USE_MAX_BLOCKS

// Make sure we don't emit negative launch bounds values.
__global__ void
__launch_bounds__( -MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP )
Kernel6()
{
}

__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP )
Kernel7()
{
}

#ifdef USE_MAX_BLOCKS
__global__ void
__launch_bounds__( MAX_THREADS_PER_BLOCK, -MIN_BLOCKS_PER_MP, -MAX_BLOCKS_PER_MP )
Kernel7_sm_90()
{
}
#endif // USE_MAX_BLOCKS

const char constchar = 12;
__global__ void __launch_bounds__(constint, constchar) Kernel8() {}

#ifdef USE_MAX_BLOCKS
const char constchar_2 = 14;
__global__ void __launch_bounds__(constint, constchar, constchar_2) Kernel8_sm_90() {}
#endif // USE_MAX_BLOCKS
