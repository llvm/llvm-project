#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip | FileCheck %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -emit-llvm %s -O0 \
// RUN:   -fcuda-include-gpubinary %t -debug-info-kind=limited \
// RUN:   -o - -x hip -fcuda-is-device | FileCheck -check-prefix=DEV %s

#include "Inputs/hip/hip_runtime.h"

extern "C" __global__ void ckernel(int *a) {
  *a = 1;
}

// Device side kernel names
// CHECK: @[[CKERN:[0-9]*]] = {{.*}} c"ckernel\00"

// DEV: define {{.*}}@ckernel{{.*}}!dbg
// DEV:  store {{.*}}!dbg
// DEV:  ret {{.*}}!dbg

// CHECK-NOT: define {{.*}}@__device_stub__ckernel{{.*}}!dbg
// CHECK: define {{.*}}@[[CSTUB:__device_stub__ckernel]]
// CHECK-NOT: call {{.*}}@hipLaunchByPtr{{.*}}!dbg
// CHECK: call {{.*}}@hipLaunchByPtr{{.*}}@[[CSTUB]]
// CHECK-NOT: ret {{.*}}!dbg

// CHECK-LABEL: define {{.*}}@_Z8hostfuncPi{{.*}}!dbg
// CHECK: call void @[[CSTUB]]{{.*}}!dbg
void hostfunc(int *a) {
  ckernel<<<1, 1>>>(a);
}
