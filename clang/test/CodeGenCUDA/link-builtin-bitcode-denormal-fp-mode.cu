
#include <hip/hip_runtime.h>
// Verify the behavior of the denormal-fp-mode attributes in the way that
// rocm-device-libs should be built with. The bitcode should be compiled with
// denormal-fp-math-f32=dynamic, and should be replaced with the denormal mode
// of the final TU.

// Build the fake device library in the way rocm-device-libs should be built.
//
// RUN: %clang_cc1 -x cl -triple amdgcn-amd-amdhsa -fdenormal-fp-math-f32=dynamic \
// RUN:   -mcode-object-version=none -emit-llvm-bc \
// RUN:   %S/Inputs/ocml-sample.cl -o %t.dynamic.f32.bc
//
// RUN: %clang_cc1 -x cl -triple amdgcn-amd-amdhsa -fdenormal-fp-math=dynamic \
// RUN:   -mcode-object-version=none -emit-llvm-bc \
// RUN:   %S/Inputs/ocml-sample.cl -o %t.dynamic.full.bc



// Check the default behavior with no denormal-fp-math arguments.
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 -fcuda-is-device \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc \
// RUN:   -emit-llvm %s -o - | FileCheck -implicit-check-not=denormal-fp-math %s --check-prefixes=CHECK,INTERNALIZE


// Check an explicit full ieee request
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 -fcuda-is-device \
// RUN:    -fdenormal-fp-math=ieee \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc \
// RUN:   -emit-llvm %s -o - | FileCheck -implicit-check-not=denormal-fp-math %s --check-prefixes=CHECK,INTERNALIZE


// Check explicit f32-only flushing request
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math-f32=preserve-sign \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,INTERNALIZE,IEEEF64-PSZF32


// Check explicit flush all request. Only the f32 component of the library is
// dynamic, so the linked functions should use IEEE as the base mode and the new
// functions preserve-sign.
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math=preserve-sign \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,INTERNALIZE,PSZ


// Check explicit f32-only, ieee-other flushing request
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math=ieee -fdenormal-fp-math-f32=preserve-sign \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,INTERNALIZE,IEEEF64-PSZF32


// Check inverse of normal usage. Requesting IEEE f32, with flushed f16/f64
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math=preserve-sign -fdenormal-fp-math-f32=ieee \
// RUN:   -mlink-builtin-bitcode %t.dynamic.f32.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,INTERNALIZE,IEEEF32-PSZF64-DYNF32


// Check backwards from the normal usage where both library components can be
// overridden.
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math=preserve-sign -fdenormal-fp-math-f32=ieee \
// RUN:   -mlink-builtin-bitcode %t.dynamic.full.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,INTERNALIZE,IEEEF32-PSZF64-DYNFULL



// Check the case where no internalization is performed
// RUN: %clang_cc1 -x hip -triple amdgcn-amd-amdhsa -target-cpu gfx803 \
// RUN:   -fcuda-is-device -fdenormal-fp-math=preserve-sign -fdenormal-fp-math-f32=ieee \
// RUN:   -mlink-bitcode-file %t.dynamic.full.bc -emit-llvm %s -o - \
// RUN: | FileCheck -implicit-check-not=denormal-fp-math --enable-var-scope %s --check-prefixes=CHECK,NOINTERNALIZE,NOINTERNALIZE-IEEEF32-PSZF64-DYNFULL



#define __device__ __attribute__((device))
#define __global__ __attribute__((global))

typedef _Float16 half;

extern "C" {
__device__ half do_f16_stuff(half a, half  b, half c);
__device__ float do_f32_stuff(float a, float b, float c);

// Currently all library functions are internalized. Check a weak function in
// case we ever choose to not internalize these. In that case, the safest thing
// to do would likely be to preserve the dynamic denormal-fp-math.
__attribute__((weak)) __device__ float weak_do_f32_stuff(float a, float b, float c);
__device__ double do_f64_stuff(double a, double b, double c);


  // CHECK: kernel_f16({{.*}}) #[[$KERNELATTR:[0-9]+]]
__global__ void kernel_f16(float* out, float* a, float* b, float* c) {
  int id = 0;
  out[id] = do_f16_stuff(a[id], b[id], c[id]);
}

// CHECK: kernel_f32({{.*}}) #[[$KERNELATTR]]
__global__ void kernel_f32(float* out, float* a, float* b, float* c) {
  int id = 0;
  out[id] = do_f32_stuff(a[id], b[id], c[id]);
  out[id] += weak_do_f32_stuff(a[id], b[id], c[id]);
}

// CHECK: kernel_f64({{.*}}) #[[$KERNELATTR]]
__global__ void kernel_f64(double* out, double* a, double* b, double* c) {
  int id = 0;
  out[id] = do_f64_stuff(a[id], b[id], c[id]);
}
}

// INTERNALIZE: define internal half @do_f16_stuff({{.*}}) #[[$FUNCATTR:[0-9]+]]
// INTERNALIZE: define internal float @do_f32_stuff({{.*}}) #[[$FUNCATTR]]
// INTERNALIZE: define internal double @do_f64_stuff({{.*}}) #[[$FUNCATTR]]
// INTERNALIZE: define internal float @weak_do_f32_stuff({{.*}}) #[[$WEAK_FUNCATTR:[0-9]+]]


// NOINTERNALIZE: define dso_local half @do_f16_stuff({{.*}}) #[[$FUNCATTR:[0-9]+]]
// NOINTERNALIZE: define dso_local float @do_f32_stuff({{.*}}) #[[$FUNCATTR]]
// NOINTERNALIZE: define dso_local double @do_f64_stuff({{.*}}) #[[$FUNCATTR]]
// NOINTERNALIZE: define weak float @weak_do_f32_stuff({{.*}}) #[[$WEAK_FUNCATTR:[0-9]+]]



// We should not be littering call sites with the attribute
// Everything should use the default ieee with no explicit attribute

// FIXME: Should check-not "denormal-fp-math" within the denormal-fp-math-f32
// lines.

// Default mode relies on the implicit check-not for the denormal-fp-math.

// PSZ: #[[$KERNELATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign"
// PSZ-SAME: "target-cpu"="gfx803"
// PSZ: #[[$FUNCATTR]] = { {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// PSZ-SAME: "target-cpu"="gfx803"
// PSZ: #[[$WEAK_FUNCATTR]] = { {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// PSZ-SAME: "target-cpu"="gfx803"

// FIXME: Should check-not "denormal-fp-math" within the line
// IEEEF64-PSZF32: #[[$KERNELATTR]] = { {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// IEEEF64-PSZF32-SAME: "target-cpu"="gfx803"
// IEEEF64-PSZF32: #[[$FUNCATTR]] = { {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// IEEEF64-PSZF32-SAME: "target-cpu"="gfx803"
// IEEEF64-PSZF32: #[[$WEAK_FUNCATTR]] = { {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// IEEEF64-PSZF32-SAME: "target-cpu"="gfx803"

// IEEEF32-PSZF64-DYNF32: #[[$KERNELATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign" "denormal-fp-math-f32"="ieee,ieee" {{.*}} "target-cpu"="gfx803" {{.*}}  }
// implicit check-not
// implicit check-not


// IEEEF32-PSZF64-DYNFULL: #[[$KERNELATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign" "denormal-fp-math-f32"="ieee,ieee"
// IEEEF32-PSZF64-DYNFULL-SAME: "target-cpu"="gfx803"
// IEEEF32-PSZF64-DYNFULL: #[[$FUNCATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign" "denormal-fp-math-f32"="ieee,ieee"
// IEEEF32-PSZF64-DYNFULL-SAME: "target-cpu"="gfx803"
// IEEEF32-PSZF64-DYNFULL: #[[$WEAK_FUNCATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign" "denormal-fp-math-f32"="ieee,ieee"
// IEEEF32-PSZF64-DYNFULL-SAME: "target-cpu"="gfx803"

// -mlink-bitcode-file doesn't internalize or propagate attributes.
// NOINTERNALIZE-IEEEF32-PSZF64-DYNFULL: #[[$KERNELATTR]] = { {{.*}} "denormal-fp-math"="preserve-sign,preserve-sign" "denormal-fp-math-f32"="ieee,ieee" {{.*}} "target-cpu"="gfx803" {{.*}} }
// NOINTERNALIZE-IEEEF32-PSZF64-DYNFULL: #[[$FUNCATTR]] = { {{.*}} "denormal-fp-math"="dynamic,dynamic" {{.*}} }
// NOINTERNALIZE-IEEEF32-PSZF64-DYNFULL: #[[$WEAK_FUNCATTR]] = { {{.*}} "denormal-fp-math"="dynamic,dynamic" {{.*}} }
