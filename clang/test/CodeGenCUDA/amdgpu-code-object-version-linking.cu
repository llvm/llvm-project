// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm-bc \
// RUN:   -mcode-object-version=4 -DUSER -x hip -o %t_4.bc %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm-bc \
// RUN:   -mcode-object-version=5 -DUSER -x hip -o %t_5.bc %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm-bc \
// RUN:   -mcode-object-version=none -DDEVICELIB -x hip -o %t_0.bc %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -O3 \
// RUN:   %t_4.bc -mlink-builtin-bitcode %t_0.bc -o - |\
// RUN:   FileCheck -check-prefix=LINKED4 %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm -O3 \
// RUN:   %t_5.bc -mlink-builtin-bitcode %t_0.bc -o - |\
// RUN:   FileCheck -check-prefix=LINKED5 %s

#include "Inputs/hip/hip_runtime.h"

// LINKED4: @llvm.amdgcn.abi.version = weak_odr hidden local_unnamed_addr addrspace(4) constant i32 400
// LINKED4-LABEL: bar
// LINKED4-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED4-NOT: icmp sge i32 %{{.*}}, 500
// LINKED4: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED4: [[GEP_5_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// LINKED4: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED4: [[GEP_4_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// LINKED4: select i1 false, ptr addrspace(4) [[GEP_5_X]], ptr addrspace(4) [[GEP_4_X]]
// LINKED4: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// LINKED4-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED4-NOT: icmp sge i32 %{{.*}}, 500
// LINKED4: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED4: [[GEP_5_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 14
// LINKED4: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED4: [[GEP_4_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 6
// LINKED4: select i1 false, ptr addrspace(4) [[GEP_5_Y]], ptr addrspace(4) [[GEP_4_Y]]
// LINKED4: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// LINKED4-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED4-NOT: icmp sge i32 %{{.*}}, 500
// LINKED4: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED4: [[GEP_5_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 16
// LINKED4: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED4: [[GEP_4_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 8
// LINKED4: select i1 false, ptr addrspace(4) [[GEP_5_Z]], ptr addrspace(4) [[GEP_4_Z]]
// LINKED4: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// LINKED4: "amdgpu_code_object_version", i32 400

// LINKED5: llvm.amdgcn.abi.version = weak_odr hidden local_unnamed_addr addrspace(4) constant i32 500
// LINKED5-LABEL: bar
// LINKED5-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED5-NOT: icmp sge i32 %{{.*}}, 500
// LINKED5: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED5: [[GEP_5_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// LINKED5: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED5: [[GEP_4_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// LINKED5: select i1 true, ptr addrspace(4) [[GEP_5_X]], ptr addrspace(4) [[GEP_4_X]]
// LINKED5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// LINKED5-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED5-NOT: icmp sge i32 %{{.*}}, 500
// LINKED5: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED5: [[GEP_5_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 14
// LINKED5: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED5: [[GEP_4_Y:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 6
// LINKED5: select i1 true, ptr addrspace(4) [[GEP_5_Y]], ptr addrspace(4) [[GEP_4_Y]]
// LINKED5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef

// LINKED5-NOT: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED5-NOT: icmp sge i32 %{{.*}}, 500
// LINKED5: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED5: [[GEP_5_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 16
// LINKED5: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED5: [[GEP_4_Z:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 8
// LINKED5: select i1 true, ptr addrspace(4) [[GEP_5_Z]], ptr addrspace(4) [[GEP_4_Z]]
// LINKED5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// LINKED5: "amdgpu_code_object_version", i32 500

#ifdef DEVICELIB
__device__ void bar(int *x, int *y, int *z)
{
  *x = __builtin_amdgcn_workgroup_size_x();
  *y = __builtin_amdgcn_workgroup_size_y();
  *z = __builtin_amdgcn_workgroup_size_z();
}
#endif

#ifdef USER
__device__ void bar(int *x, int *y, int *z);
__device__ void foo()
{
  int *x, *y, *z;
  bar(x, y, z);
}
#endif
