// REQUIRES: nvptx-registered-target
// REQUIRES: x86-registered-target

// RUN: %clang_cc1 "-aux-triple" "x86_64-unknown-linux-gnu" "-triple" "nvptx64-nvidia-cuda" \
// RUN:    -fcuda-is-device "-aux-target-cpu" "x86-64" -O1 -S -o - %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: .visible .func _Z8test_argPDF16bDF16b(
// CHECK:        .param .b64 _Z8test_argPDF16bDF16b_param_0,
// CHECK:        .param .align 2 .b8 _Z8test_argPDF16bDF16b_param_1[2]
//
__device__ void test_arg(__bf16 *out, __bf16 in) {
// CHECK-DAG:     ld.param.u64  %[[A:rd[0-9]+]], [_Z8test_argPDF16bDF16b_param_0];
// CHECK-DAG:     ld.param.b16  %[[R:rs[0-9]+]], [_Z8test_argPDF16bDF16b_param_1];
  __bf16 bf16 = in;
  *out = bf16;
// CHECK:         st.b16         [%[[A]]], %[[R]]
// CHECK:         ret;
}


// CHECK-LABEL: .visible .func (.param .align 2 .b8 func_retval0[2]) _Z8test_retDF16b(
// CHECK:        .param .align 2 .b8 _Z8test_retDF16b_param_0[2]
__device__ __bf16 test_ret( __bf16 in) {
// CHECK:        ld.param.b16    %[[R:rs[0-9]+]], [_Z8test_retDF16b_param_0];
  return in;
// CHECK:        st.param.b16    [func_retval0+0], %[[R]]
// CHECK:        ret;
}

__device__ __bf16 external_func( __bf16 in);

// CHECK-LABEL: .visible .func  (.param .align 2 .b8 func_retval0[2]) _Z9test_callDF16b(
// CHECK:        .param .align 2 .b8 _Z9test_callDF16b_param_0[2]
__device__ __bf16 test_call( __bf16 in) {
// CHECK:        ld.param.b16    %[[R:rs[0-9]+]], [_Z9test_callDF16b_param_0];
// CHECK:        st.param.b16    [param0+0], %[[R]];
// CHECK:        .param .align 2 .b8 retval0[2];
// CHECK:        call.uni (retval0),
// CHECK-NEXT:   _Z13external_funcDF16b,
// CHECK-NEXT:   (
// CHECK-NEXT:   param0
// CHECK-NEXT    );
// CHECK:        ld.param.b16    %[[RET:rs[0-9]+]], [retval0+0];
  return external_func(in);
// CHECK:        st.param.b16    [func_retval0+0], %[[RET]]
// CHECK:        ret;
}
