#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -check-prefixes=CHECK,GNU,GNU-HIP,HIP %s

// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s \
// RUN:     -fcuda-include-gpubinary %t -o - -x hip\
// RUN:   | FileCheck -check-prefix=NEG %s

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -emit-llvm %s \
// RUN:     -aux-triple amdgcn-amd-amdhsa -fcuda-include-gpubinary \
// RUN:     %t -o - -x hip\
// RUN:   | FileCheck -check-prefixes=CHECK,MSVC,MSVC-HIP,HIP %s

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -emit-llvm %s \
// RUN:     -aux-triple nvptx64 -fcuda-include-gpubinary \
// RUN:     %t -target-sdk-version=9.2 -o - \
// RUN:   | FileCheck -check-prefixes=CHECK,MSVC,CUDA %s

// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -emit-llvm %s \
// RUN:     -aux-triple amdgcn-amd-amdhsa -fcuda-include-gpubinary \
// RUN:     %t -o - -x hip\
// RUN:   | FileCheck -check-prefix=NEG %s

#include "Inputs/hip/hip_runtime.h"

// Check kernel handles are emitted for non-MSVC target but not for MSVC target.

// GNU-HIP: @[[HCKERN:ckernel]] = constant ptr @[[CSTUB:__device_stub__ckernel]], align 8
// GNU-HIP: @[[HNSKERN:_ZN2ns8nskernelEv]] = constant ptr @[[NSSTUB:_ZN2ns23__device_stub__nskernelEv]], align 8
// GNU-HIP: @[[HTKERN:_Z10kernelfuncIiEvv]] = linkonce_odr constant ptr @[[TSTUB:_Z25__device_stub__kernelfuncIiEvv]], comdat, align 8
// GNU-HIP: @[[HDKERN:_Z11kernel_declv]] = external constant ptr, align 8
// GNU-HIP: @[[HTDKERN:_Z20template_kernel_declIiEvT_]] = external constant ptr, align 8

// MSVC-HIP: @[[HCKERN:ckernel]] = dso_local constant ptr @[[CSTUB:__device_stub__ckernel]], align 8
// MSVC-HIP: @[[HNSKERN:"\?nskernel@ns@@YAXXZ.*"]] = dso_local constant ptr @[[NSSTUB:"\?__device_stub__nskernel@ns@@YAXXZ"]], align 8
// MSVC-HIP: @[[HTKERN:"\?\?\$kernelfunc@H@@YAXXZ.*"]] = linkonce_odr dso_local constant ptr @[[TSTUB:"\?\?\$__device_stub__kernelfunc@H@@YAXXZ.*"]], comdat, align 8
// MSVC-HIP: @[[HDKERN:"\?kernel_decl@@YAXXZ.*"]] = external dso_local constant ptr, align 8
// MSVC-HIP: @[[HTDKERN:"\?\?\$template_kernel_decl@H@@YAXH.*"]] = external dso_local constant ptr, align 8
extern "C" __global__ void ckernel() {}

// CUDA: @[[HCKERN:__device_stub__ckernel\.id]] = dso_local global i8 0
// CUDA: @[[HNSKERN:"\?__device_stub__nskernel@ns@@YAXXZ\.id"]] = dso_local global i8 0
// CUDA: @[[HTKERN:"\?\?\$__device_stub__kernelfunc@H@@YAXXZ\.id"]] = linkonce_odr dso_local global i8 0, comdat

namespace ns {
__global__ void nskernel() {}
} // namespace ns

template<class T>
__global__ void kernelfunc() {}

__global__ void kernel_decl();

template<class T>
__global__ void template_kernel_decl(T x);

extern "C" void (*kernel_ptr)();
extern "C" void *void_ptr;

extern "C" void launch(void *kern);

// Device side kernel names

// CHECK: @[[CKERN:[0-9]*]] = {{.*}} c"ckernel\00"
// CHECK: @[[NSKERN:[0-9]*]] = {{.*}} c"_ZN2ns8nskernelEv\00"
// CHECK: @[[TKERN:[0-9]*]] = {{.*}} c"_Z10kernelfuncIiEvv\00"

// Non-template kernel stub functions

// HIP: define{{.*}}@[[CSTUB]]
// CUDA: define{{.*}}@[[CSTUB:__device_stub__ckernel]]
// HIP: call{{.*}}@hipLaunchByPtr{{.*}}@[[HCKERN]]
// CUDA: call{{.*}}@hipLaunchByPtr{{.*}}@[[CSTUB]]
// CUDA: store volatile i8 1, ptr @[[HCKERN]], align 1
// CHECK: ret void

// HIP: define{{.*}}@[[NSSTUB]]
// CUDA: define{{.*}}@[[NSSTUB:"\?__device_stub__nskernel@ns@@YAXXZ"]]
// HIP: call{{.*}}@hipLaunchByPtr{{.*}}@[[HNSKERN]]
// CUDA: call{{.*}}@hipLaunchByPtr{{.*}}@[[NSSTUB]]
// CUDA: store volatile i8 1, ptr @[[HNSKERN]], align 1
// CHECK: ret void

// Check kernel stub is called for triple chevron.

// CHECK-LABEL: define{{.*}}@fun1()
// CHECK: call void @[[CSTUB]]()
// CHECK: call void @[[NSSTUB]]()
// HIP: call void @[[TSTUB]]()
// CUDA: call void @[[TSTUB:"\?\?\$__device_stub__kernelfunc@H@@YAXXZ.*"]]()
// GNU: call void @[[DSTUB:_Z26__device_stub__kernel_declv]]()
// GNU: call void @[[TDSTUB:_Z35__device_stub__template_kernel_declIiEvT_]](
// MSVC: call void @[[DSTUB:"\?__device_stub__kernel_decl@@YAXXZ"]]()
// MSVC: call void @[[TDSTUB:"\?\?\$__device_stub__template_kernel_decl@H@@YAXH@Z"]](

extern "C" void fun1(void) {
  ckernel<<<1, 1>>>();
  ns::nskernel<<<1, 1>>>();
  kernelfunc<int><<<1, 1>>>();
  kernel_decl<<<1, 1>>>();
  template_kernel_decl<<<1, 1>>>(1);
}

// Template kernel stub functions

// CHECK: define{{.*}}@[[TSTUB]]
// HIP: call{{.*}}@hipLaunchByPtr{{.*}}@[[HTKERN]]
// CUDA: call{{.*}}@hipLaunchByPtr{{.*}}@[[TSTUB]]
// CUDA: store volatile i8 1, ptr @[[HTKERN]], align 1
// CHECK: ret void

// Check declaration of stub function for external kernel.

// CHECK: declare{{.*}}@[[DSTUB]]
// CHECK: declare{{.*}}@[[TDSTUB]]

// Check kernel handle is used for passing the kernel as a function pointer.

// CHECK-LABEL: define{{.*}}@fun2()
// HIP: call void @launch({{.*}}[[HCKERN]]
// HIP: call void @launch({{.*}}[[HNSKERN]]
// HIP: call void @launch({{.*}}[[HTKERN]]
// HIP: call void @launch({{.*}}[[HDKERN]]
// HIP: call void @launch({{.*}}[[HTDKERN]]
extern "C" void fun2() {
  launch((void *)ckernel);
  launch((void *)ns::nskernel);
  launch((void *)kernelfunc<int>);
  launch((void *)kernel_decl);
  launch((void *)template_kernel_decl<int>);
}

// Check kernel handle is used for assigning a kernel to a function pointer.

// CHECK-LABEL: define{{.*}}@fun3()
// HIP:  store ptr @[[HCKERN]], ptr @kernel_ptr, align 8
// HIP:  store ptr @[[HCKERN]], ptr @kernel_ptr, align 8
// HIP:  store ptr @[[HCKERN]], ptr @void_ptr, align 8
// HIP:  store ptr @[[HCKERN]], ptr @void_ptr, align 8
extern "C" void fun3() {
  kernel_ptr = ckernel;
  kernel_ptr = &ckernel;
  void_ptr = (void *)ckernel;
  void_ptr = (void *)&ckernel;
}

// Check kernel stub is loaded from kernel handle when function pointer is
// used with triple chevron.

// CHECK-LABEL: define{{.*}}@fun4()
// HIP:  store ptr @[[HCKERN]], ptr @kernel_ptr
// HIP:  call noundef i32 @{{.*hipConfigureCall}}
// HIP:  %[[HANDLE:.*]] = load ptr, ptr @kernel_ptr, align 8
// HIP:  %[[STUB:.*]] = load ptr, ptr %[[HANDLE]], align 8
// HIP:  call void %[[STUB]]()
extern "C" void fun4() {
  kernel_ptr = ckernel;
  kernel_ptr<<<1,1>>>();
}

// Check kernel handle is passed to a function.

// CHECK-LABEL: define{{.*}}@fun5()
// HIP:  store ptr @[[HCKERN]], ptr @kernel_ptr
// HIP:  %[[HANDLE:.*]] = load ptr, ptr @kernel_ptr, align 8
// HIP:  call void @launch(ptr noundef %[[HANDLE]])
extern "C" void fun5() {
  kernel_ptr = ckernel;
  launch((void *)kernel_ptr);
}

// Check kernel handle is registered.

// HIP-LABEL: define{{.*}}@__hip_register_globals
// HIP: call{{.*}}@__hipRegisterFunction{{.*}}@[[HCKERN]]{{.*}}@[[CKERN]]
// HIP: call{{.*}}@__hipRegisterFunction{{.*}}@[[HNSKERN]]{{.*}}@[[NSKERN]]
// HIP: call{{.*}}@__hipRegisterFunction{{.*}}@[[HTKERN]]{{.*}}@[[TKERN]]
// NEG-NOT: call{{.*}}@__hipRegisterFunction{{.*}}__device_stub
// NEG-NOT: call{{.*}}@__hipRegisterFunction{{.*}}kernel_decl
// NEG-NOT: call{{.*}}@__hipRegisterFunction{{.*}}template_kernel_decl
