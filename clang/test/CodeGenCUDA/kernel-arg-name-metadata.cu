#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fhip-kernel-arg-name \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefix=NEG %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: define{{.*}} amdgpu_kernel void @_Z6kerneliPf({{.*}} !kernel_arg_name [[MD:![0-9]+]]
// NEG-NOT: define{{.*}} amdgpu_kernel void @_Z6kerneliPf({{.*}} !kernel_arg_name
__global__ void kernel(int arg1, float *arg2) {
}

// CHECK: [[MD]] = !{!"arg1", !"arg2"}
