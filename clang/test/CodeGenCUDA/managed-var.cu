#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target, amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -std=c++11 \
// RUN:   -emit-llvm -fgpu-rdc -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,NORDC %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -std=c++11 \
// RUN:   -emit-llvm -fgpu-rdc -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,RDC %s

#include "Inputs/hip/hip_runtime.h"

// DEV-DAG: @x = external addrspace(1) externally_initialized global i32
// NORDC-DAG: @x = internal global i32 1
// RDC-DAG: @x = dso_local global i32 1
// NORDC-DAG: @x.managed = internal global i32* null
// RDC-DAG: @x.managed = dso_local global i32* null
// HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"x\00"

struct vec {
  float x,y,z;
};

__managed__ int x = 1;
__managed__ vec v[100];
__managed__ vec v2[100] = {{1, 1, 1}};

// DEV-DAG: @ex = external addrspace(1) global i32
// HOST-DAG: @ex = external global i32
extern __managed__ int ex;

// DEV-DAG: @_ZL2sx = external addrspace(1) externally_initialized global i32
// HOST-DAG: @_ZL2sx = internal global i32 1
// HOST-DAG: @_ZL2sx.managed = internal global i32* null
static __managed__ int sx = 1;

// HOST-NOT: @ex.managed

// Force ex and sx mitted in device compilation.
__global__ void foo(int *z) {
  *z = x + ex + sx;
  v[1].x = 2;
}

// Force ex and sx emitted in host compilatioin.
int foo2() {
  return ex + sx;
}

// HOST-LABEL: define {{.*}}@_Z4loadv()
// HOST:  %ld.managed = load i32*, i32** @x.managed, align 4
// HOST:  %0 = load i32, i32* %ld.managed, align 4
// HOST:  ret i32 %0
int load() {
  return x;
}

// HOST-LABEL: define {{.*}}@_Z5storev()
// HOST:  %ld.managed = load i32*, i32** @x.managed, align 4
// HOST:  store i32 2, i32* %ld.managed, align 4
void store() {
  x = 2;
}

// HOST-LABEL: define {{.*}}@_Z10addr_takenv()
// HOST:  %ld.managed = load i32*, i32** @x.managed, align 4
// HOST:  store i32* %ld.managed, i32** %p, align 8
// HOST:  %0 = load i32*, i32** %p, align 8
// HOST:  store i32 3, i32* %0, align 4
void addr_taken() {
  int *p = &x;
  *p = 3;
}

// HOST-LABEL: define {{.*}}@_Z5load2v()
// HOST: %ld.managed = load [100 x %struct.vec]*, [100 x %struct.vec]** @v.managed, align 16
// HOST:  %0 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %ld.managed, i64 0, i64 1, i32 0
// HOST:  %1 = load float, float* %0, align 4
// HOST:  ret float %1
float load2() {
  return v[1].x;
}

// HOST-LABEL: define {{.*}}@_Z5load3v()
// HOST:  %ld.managed = load <{ %struct.vec, [99 x %struct.vec] }>*, <{ %struct.vec, [99 x %struct.vec] }>** @v2.managed, align 16
// HOST:  %0 = bitcast <{ %struct.vec, [99 x %struct.vec] }>* %ld.managed to [100 x %struct.vec]*
// HOST:  %1 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %0, i64 0, i64 1, i32 1
// HOST:  %2 = load float, float* %1, align 4
// HOST:  ret float %2
float load3() {
  return v2[1].y;
}

// HOST-LABEL: define {{.*}}@_Z11addr_taken2v()
// HOST:  %ld.managed = load [100 x %struct.vec]*, [100 x %struct.vec]** @v.managed, align 16
// HOST:  %0 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %ld.managed, i64 0, i64 1, i32 0
// HOST:  %1 = ptrtoint float* %0 to i64
// HOST:  %ld.managed1 = load <{ %struct.vec, [99 x %struct.vec] }>*, <{ %struct.vec, [99 x %struct.vec] }>** @v2.managed, align 16
// HOST:  %2 = bitcast <{ %struct.vec, [99 x %struct.vec] }>* %ld.managed1 to [100 x %struct.vec]*
// HOST:  %3 = getelementptr inbounds [100 x %struct.vec], [100 x %struct.vec]* %2, i64 0, i64 1, i32 1
// HOST:  %4 = ptrtoint float* %3 to i64
// HOST:  %5 = sub i64 %4, %1
// HOST:  %6 = sdiv i64 %5, 4
// HOST:  %7 = sitofp i64 %6 to float
// HOST:  ret float %7
float addr_taken2() {
  return (float)reinterpret_cast<long>(&(v2[1].y)-&(v[1].x));
}

// HOST-DAG: __hipRegisterManagedVar({{.*}}@x.managed {{.*}}@x {{.*}}@[[DEVNAMEX]]{{.*}}, i64 4, i32 4)
// HOST-DAG: __hipRegisterManagedVar({{.*}}@_ZL2sx.managed {{.*}}@_ZL2sx
// HOST-NOT: __hipRegisterManagedVar({{.*}}@ex.managed {{.*}}@ex
// HOST-DAG: declare void @__hipRegisterManagedVar(i8**, i8*, i8*, i8*, i64, i32)
