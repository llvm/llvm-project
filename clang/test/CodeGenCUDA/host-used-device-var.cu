#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++11 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// Check device variables used by neither host nor device functioins are not kept.

// CHECK-NOT: @v1
__device__ int v1;

// CHECK-NOT: @v2
__constant__ int v2;

// CHECK-NOT: @_ZL2v3
static __device__ int v3;

// Check device variables used by host functions are kept.

// CHECK-DAG: @u1
__device__ int u1;

// CHECK-DAG: @u2
__constant__ int u2;

// Check host-used static device var is in llvm.compiler.used.
// CHECK-DAG: @_ZL2u3
static __device__ int u3;

// Check device-used static device var is emitted but is not in llvm.compiler.used.
// CHECK-DAG: @_ZL2u4
static __device__ int u4;

// Check device variables with used attribute are always kept.
// CHECK-DAG: @u5
__device__ __attribute__((used)) int u5;

int fun1() {
  return u1 + u2 + u3;
}

__global__ void kern1(int **x) {
  *x = &u4;
}
// Check the exact list of variables to ensure @_ZL2u4 is not among them.
// CHECK: @llvm.compiler.used = {{[^@]*}} @_ZL2u3 {{[^@]*}} @u1 {{[^@]*}} @u2 {{[^@]*}} @u5
