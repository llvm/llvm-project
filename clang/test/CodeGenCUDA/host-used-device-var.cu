#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV-NEG %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST-NEG %s

#include "Inputs/hip/hip_runtime.h"

// Check device variables used by neither host nor device functioins are not kept.

// DEV-NEG-NOT: @v1
__device__ int v1;

// DEV-NEG-NOT: @v2
__constant__ int v2;

// DEV-NEG-NOT: @_ZL2v3
static __device__ int v3;

// Check device variables used by host functions are kept.

// DEV-DAG: @u1
__device__ int u1;

// DEV-DAG: @u2
__constant__ int u2;

// Check host-used static device var is in llvm.compiler.used.
// DEV-DAG: @_ZL2u3
static __device__ int u3;

// Check device-used static device var is emitted but is not in llvm.compiler.used.
// DEV-DAG: @_ZL2u4
static __device__ int u4;

// Check device variables with used attribute are always kept.
// DEV-DAG: @u5
__device__ __attribute__((used)) int u5;

// Test external device variable ODR-used by host code is not emitted or registered.
// DEV-NEG-NOT: @ext_var
extern __device__ int ext_var;

// DEV-DAG: @inline_var = linkonce_odr addrspace(1) externally_initialized global i32 0
__device__ inline int inline_var;

template<typename T>
using func_t = T (*) (T, T);

template <typename T>
__device__ T add_func (T x, T y)
{
  return x + y;
}

// DEV-DAG: @_Z10p_add_funcIiE = linkonce_odr addrspace(1) externally_initialized global i32 (i32, i32)* @_Z8add_funcIiET_S0_S0_
template <typename T>
__device__ func_t<T> p_add_func = add_func<T>;

void use(func_t<int> p);
void use(int *p);

void fun1() {
  use(&u1);
  use(&u2);
  use(&u3);
  use(&ext_var);
  use(&inline_var);
  use(p_add_func<int>);
}

__global__ void kern1(int **x) {
  *x = &u4;
}

// Check the exact list of variables to ensure @_ZL2u4 is not among them.
// DEV: @llvm.compiler.used = {{[^@]*}} @_Z10p_add_funcIiE {{[^@]*}} @_ZL2u3 {{[^@]*}} @inline_var {{[^@]*}} @u1 {{[^@]*}} @u2 {{[^@]*}} @u5

// HOST-DAG: hipRegisterVar{{.*}}@u1
// HOST-DAG: hipRegisterVar{{.*}}@u2
// HOST-DAG: hipRegisterVar{{.*}}@_ZL2u3
// HOST-DAG: hipRegisterVar{{.*}}@u5
// HOST-DAG: hipRegisterVar{{.*}}@inline_var
// HOST-DAG: hipRegisterVar{{.*}}@_Z10p_add_funcIiE
// HOST-NEG-NOT: hipRegisterVar{{.*}}@ext_var
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZL2u4
