
#include <hip/hip_runtime.h>
// Create module flag for code object version.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -o - %s | FileCheck %s -check-prefix=V4

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=2 -o - %s | FileCheck -check-prefix=V2 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=3 -o - %s | FileCheck -check-prefix=V3 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4 -o - %s | FileCheck -check-prefix=V4 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=5 -o - %s | FileCheck -check-prefix=V5 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=none -o - %s | FileCheck %s -check-prefix=NONE

// RUN: not %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4.1 -o - %s 2>&1| FileCheck %s -check-prefix=INV

// V2: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 200}
// V3: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 300}
// V4: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 400}
// V5: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 500}
// NONE-NOT: !{{.*}} = !{i32 1, !"amdgpu_code_object_version",
// INV: error: invalid value '4.1' in '-mcode-object-version=4.1'
