
#include <hip/hip_runtime.h>
// Create module flag for code object version.

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -o - %s | FileCheck %s -check-prefix=V4

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4 -o - %s | FileCheck -check-prefix=V4 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=5 -o - %s | FileCheck -check-prefix=V5 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=none -o - %s | FileCheck %s -check-prefix=NONE

// RUN: not %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4.1 -o - %s 2>&1| FileCheck %s -check-prefix=INV

// V4: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 400}
// V5: !{{.*}} = !{i32 1, !"amdgpu_code_object_version", i32 500}
// NONE-NOT: !{{.*}} = !{i32 1, !"amdgpu_code_object_version",
// INV: error: invalid value '4.1' in '-mcode-object-version=4.1'
