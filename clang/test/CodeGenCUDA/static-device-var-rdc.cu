#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV,INT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux \
// RUN:   -fgpu-rdc -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,INT-HOST %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=abc \
// RUN:   -fgpu-rdc -emit-llvm -o - -x hip %s > %t.dev
// RUN: cat %t.dev | FileCheck -check-prefixes=DEV,EXT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=abc \
// RUN:   -fgpu-rdc -emit-llvm -o - -x hip %s > %t.host
// RUN: cat %t.host | FileCheck -check-prefixes=HOST,EXT-HOST %s

// Check host and device compilations use the same postfixes for static
// variable names.

// RUN: cat %t.dev %t.host | FileCheck -check-prefix=POSTFIX %s

#include "Inputs/hip/hip_runtime.h"

// Test function scope static device variable, which should not be externalized.
// DEV-DAG: @_ZZ6kernelPiPPKiE1w = internal addrspace(4) constant i32 1


// HOST-DAG: @_ZL1x = internal global i32 undef
// HOST-DAG: @_ZL1y = internal global i32 undef

// Test normal static device variables
// INT-DEV-DAG: @_ZL1x = dso_local addrspace(1) externally_initialized global i32 0
// INT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1x.static.[[HASH:.*]] = dso_local addrspace(1) externally_initialized global i32 0
// EXT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x.static.[[HASH:.*]]\00"

// POSTFIX: @_ZL1x.static.[[HASH:.*]] = dso_local addrspace(1) externally_initialized global i32 0
// POSTFIX: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x.static.[[HASH]]\00"

static __device__ int x;

// Test static device variables not used by host code should not be externalized
// DEV-DAG: @_ZL2x2 = internal addrspace(1) global i32 0

static __device__ int x2;

// Test normal static device variables
// INT-DEV-DAG: @_ZL1y = dso_local addrspace(4) externally_initialized global i32 0
// INT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1y.static.[[HASH]] = dso_local addrspace(4) externally_initialized global i32 0
// EXT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y.static.[[HASH]]\00"

static __constant__ int y;

// Test static host variable, which should not be externalized nor registered.
// HOST-DAG: @_ZL1z = internal global i32 0
// DEV-NOT: @_ZL1z
static int z;

// Test static device variable in inline function, which should not be
// externalized nor registered.
// DEV-DAG: @_ZZ6devfunPPKiE1p = linkonce_odr addrspace(4) constant i32 2, comdat

inline __device__ void devfun(const int ** b) {
  const static int p = 2;
  b[0] = &p;
}

__global__ void kernel(int *a, const int **b) {
  const static int w = 1;
  a[0] = x;
  a[1] = y;
  b[0] = &w;
  b[1] = &x2;
  devfun(b);
}

int* getDeviceSymbol(int *x);

void foo() {
  getDeviceSymbol(&x);
  getDeviceSymbol(&y);
  z = 123;
}

// HOST: __hipRegisterVar({{.*}}@_ZL1x {{.*}}@[[DEVNAMEX]]
// HOST: __hipRegisterVar({{.*}}@_ZL1y {{.*}}@[[DEVNAMEY]]
// HOST-NOT: __hipRegisterVar({{.*}}@_ZL2x2
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6kernelPiPPKiE1w
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6devfunPPKiE1p
