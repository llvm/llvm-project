#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -x hip -triple x86_64-linux-gnu -aux-triple amdgcn-amd-amdhsa -emit-llvm %s -o - | FileCheck %s --check-prefix=HOST
// RUN: %clang_cc1 -std=c++11 -x hip -triple amdgcn-amd-amdhsa -fcuda-is-device -emit-llvm %s -o - | FileCheck %s --check-prefix=DEVICE

#include "Inputs/hip/hip_runtime.h"

// HOST: @0 = private unnamed_addr constant [43 x i8] c"_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_\00", align 1

__device__ float d0(float x) {
  return [](float x) { return x + 2.f; }(x);
}

__device__ float d1(float x) {
  return [](float x) { return x * 2.f; }(x);
}

// DEVICE: amdgpu_kernel void @_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_(
template <typename F>
__global__ void k0(float *p, F f) {
  p[0] = f(p[0]) + d0(p[1]) + d1(p[2]);
}

void f0(float *p) {
  [](float *p) {
    *p = 1.f;
  }(p);
}

// The inner/outer lambdas are required to be mangled following ODR but their
// linkages are still required to keep the original `internal` linkage.

// HOST: define internal void @_ZZ2f1PfENKUlS_E_clES_(
// DEVICE: define internal float @_ZZZ2f1PfENKUlS_E_clES_ENKUlfE_clEf(
void f1(float *p) {
  [](float *p) {
    k0<<<1,1>>>(p, [] __device__ (float x) { return x + 1.f; });
  }(p);
}
// HOST: @__hip_register_globals
// HOST: __hipRegisterFunction{{.*}}@_Z17__device_stub__k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_{{.*}}@0
