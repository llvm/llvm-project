#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=CHECK,DEFAULT %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa --gpu-max-threads-per-block=1024 \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=CHECK,MAX1024 %s
// RUN: %clang_cc1 -triple nvptx \
// RUN:     -fcuda-is-device -emit-llvm -o - %s | FileCheck %s \
// RUN:     -check-prefix=NAMD
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:     -verify -o - -x hip %s | FileCheck -check-prefix=NAMD %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -foffload-uniform-block \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=CHECK,DEFAULT %s
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fno-offload-uniform-block \
// RUN:     -fcuda-is-device -emit-llvm -o - -x hip %s \
// RUN:     | FileCheck -check-prefixes=NOUB %s

#include "Inputs/hip/hip_runtime.h"

__global__ void flat_work_group_size_default() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z28flat_work_group_size_defaultv() [[FLAT_WORK_GROUP_SIZE_DEFAULT:#[0-9]+]]
// NOUB: define{{.*}} void @_Z28flat_work_group_size_defaultv() [[NOUB:#[0-9]+]]
}

__attribute__((amdgpu_flat_work_group_size(32, 64))) // expected-no-diagnostics
__global__ void flat_work_group_size_32_64() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z26flat_work_group_size_32_64v() [[FLAT_WORK_GROUP_SIZE_32_64:#[0-9]+]]
}
__attribute__((amdgpu_waves_per_eu(2))) // expected-no-diagnostics
__global__ void waves_per_eu_2() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z14waves_per_eu_2v() [[WAVES_PER_EU_2:#[0-9]+]]
}
__attribute__((amdgpu_num_sgpr(32))) // expected-no-diagnostics
__global__ void num_sgpr_32() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z11num_sgpr_32v() [[NUM_SGPR_32:#[0-9]+]]
}
__attribute__((amdgpu_num_vgpr(64))) // expected-no-diagnostics
__global__ void num_vgpr_64() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z11num_vgpr_64v() [[NUM_VGPR_64:#[0-9]+]]
}
__attribute__((amdgpu_max_num_work_groups(32, 4, 2))) // expected-no-diagnostics
__global__ void max_num_work_groups_32_4_2() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z26max_num_work_groups_32_4_2v() [[MAX_NUM_WORK_GROUPS_32_4_2:#[0-9]+]]
}
__attribute__((amdgpu_max_num_work_groups(32))) // expected-no-diagnostics
__global__ void max_num_work_groups_32() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z22max_num_work_groups_32v() [[MAX_NUM_WORK_GROUPS_32_1_1:#[0-9]+]]
}
__attribute__((amdgpu_max_num_work_groups(32,1))) // expected-no-diagnostics
__global__ void max_num_work_groups_32_1() {
// CHECK: define{{.*}} amdgpu_kernel void @_Z24max_num_work_groups_32_1v() [[MAX_NUM_WORK_GROUPS_32_1_1:#[0-9]+]]
}



template<unsigned a>
__attribute__((amdgpu_max_num_work_groups(a, 4, 2)))
__global__ void template_a_4_2_max_num_work_groups() {}
template __global__ void template_a_4_2_max_num_work_groups<32>();
// CHECK: define{{.*}} amdgpu_kernel void @_Z34template_a_4_2_max_num_work_groupsILj32EEvv() [[MAX_NUM_WORK_GROUPS_32_4_2:#[0-9]+]]

template<unsigned a>
__attribute__((amdgpu_max_num_work_groups(32, a, 2)))
__global__ void template_32_a_2_max_num_work_groups() {}
template __global__ void template_32_a_2_max_num_work_groups<4>();
// CHECK: define{{.*}} amdgpu_kernel void @_Z35template_32_a_2_max_num_work_groupsILj4EEvv() [[MAX_NUM_WORK_GROUPS_32_4_2:#[0-9]+]]

template<unsigned a>
__attribute__((amdgpu_max_num_work_groups(32, 4, a)))
__global__ void template_32_4_a_max_num_work_groups() {}
template __global__ void template_32_4_a_max_num_work_groups<2>();
// CHECK: define{{.*}} amdgpu_kernel void @_Z35template_32_4_a_max_num_work_groupsILj2EEvv() [[MAX_NUM_WORK_GROUPS_32_4_2:#[0-9]+]]

// Make sure this is silently accepted on other targets.
// NAMD-NOT: "amdgpu-flat-work-group-size"
// NAMD-NOT: "amdgpu-waves-per-eu"
// NAMD-NOT: "amdgpu-num-vgpr"
// NAMD-NOT: "amdgpu-num-sgpr"
// NAMD-NOT: "amdgpu-max-num-work-groups"

// DEFAULT-DAG: attributes [[FLAT_WORK_GROUP_SIZE_DEFAULT]] = {{.*}}"amdgpu-flat-work-group-size"="1,1024"{{.*}}"uniform-work-group-size"="true"
// MAX1024-DAG: attributes [[FLAT_WORK_GROUP_SIZE_DEFAULT]] = {{.*}}"amdgpu-flat-work-group-size"="1,1024"
// CHECK-DAG: attributes [[FLAT_WORK_GROUP_SIZE_32_64]] = {{.*}}"amdgpu-flat-work-group-size"="32,64"
// CHECK-DAG: attributes [[WAVES_PER_EU_2]] = {{.*}}"amdgpu-waves-per-eu"="2"
// CHECK-DAG: attributes [[NUM_SGPR_32]] = {{.*}}"amdgpu-num-sgpr"="32"
// CHECK-DAG: attributes [[NUM_VGPR_64]] = {{.*}}"amdgpu-num-vgpr"="64"
// CHECK-DAG: attributes [[MAX_NUM_WORK_GROUPS_32_4_2]] = {{.*}}"amdgpu-max-num-workgroups"="32,4,2"
// CHECK-DAG: attributes [[MAX_NUM_WORK_GROUPS_32_1_1]] = {{.*}}"amdgpu-max-num-workgroups"="32,1,1"

// NOUB-NOT: "uniform-work-group-size"="true"
