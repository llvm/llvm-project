#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -fgpu-rdc -emit-llvm %s -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

__global__ void g2(int x) {}

// CHECK-LABEL: define{{.*}}g1
__global__ void g1(void) {
  // CHECK: [[CONFIG:%.*]] = call{{.*}}_Z22cudaGetParameterBuffermm(i64{{.*}}64, i64{{.*}}4)
  // CHECK-NEXT: [[FLAG:%.*]] = icmp ne ptr [[CONFIG]], null
  // CHECK-NEXT: br i1 [[FLAG]], label %[[THEN:.*]], label %[[ENDIF:.*]]
  // CHECK: [[THEN]]:
  // CHECK-NEXT: [[PPTR:%.*]] = getelementptr{{.*}}i8, ptr [[CONFIG]], i64 0
  // CHECK-NEXT: store i32 42, ptr [[PPTR]]
  // CHECK: = call{{.*}} i32 @_Z16cudaLaunchDevicePvS_4dim3S0_jP10cudaStream(ptr{{.*}} @_Z2g2i, ptr{{.*}} [[CONFIG]],
  g2<<<1, 1>>>(42);
}
