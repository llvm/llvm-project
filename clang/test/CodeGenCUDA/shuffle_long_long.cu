#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm -target-cpu sm_30 %s -o - | FileCheck %s --check-prefix=NO_SYNC
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -emit-llvm -target-cpu sm_30 -target-feature +ptx70 -DSYNC -DCUDA_VERSION=9000 %s -o - | FileCheck %s --check-prefix=SYNC

#include "Inputs/hip/hip_runtime.h"

__device__ void *memcpy(void *dest, const void *src, size_t n);

#define warpSize 32
#include <__clang_cuda_intrinsics.h>

__device__ void use(unsigned long long, long long);

// Test function, 4 shfl calls.
// NO_SYNC: define{{.*}} @_Z14test_long_longv
// NO_SYNC:     call noundef i64 @_Z6__shflyii(
// NO_SYNC:     call noundef i64 @_Z6__shflxii(

// SYNC: define{{.*}} @_Z14test_long_longv
// SYNC:        call noundef i64 @_Z11__shfl_syncjyii(
// SYNC:        call noundef i64 @_Z11__shfl_syncjxii(

// unsigned long long -> long long
// NO_SYNC: define{{.*}} @_Z6__shflyii
// NO_SYNC:     call noundef i64 @_Z6__shflxii(

// long long -> int + int
// NO_SYNC: define{{.*}} @_Z6__shflxii
// NO_SYNC:     call noundef i32 @_Z6__shfliii(
// NO_SYNC:     call noundef i32 @_Z6__shfliii(

// NO_SYNC: define{{.*}} @_Z6__shfliii
// NO_SYNC:   call i32 @llvm.nvvm.shfl.idx.i32

// unsigned long long -> long long
// SYNC: _Z11__shfl_syncjyii
// SYNC:     call noundef i64 @_Z11__shfl_syncjxii(

// long long -> int + int
// SYNC: define{{.*}} @_Z11__shfl_syncjxii
// SYNC:     call noundef i32 @_Z11__shfl_syncjiii(
// SYNC:     call noundef i32 @_Z11__shfl_syncjiii(

// SYNC: define{{.*}} @_Z11__shfl_syncjiii
// SYNC:      call i32 @llvm.nvvm.shfl.sync.idx.i32

__device__ void test_long_long() {
  unsigned long long ull = 13;
  long long ll = 17;
#ifndef SYNC
  ull = __shfl(ull, 7, 32);
  ll = __shfl(ll, 7, 32);
  use(ull, ll);
#else
  ull = __shfl_sync(0x11, ull, 7, 32);
  ll = __shfl_sync(0x11, ll, 7, 32);
  use(ull, ll);
#endif
}

