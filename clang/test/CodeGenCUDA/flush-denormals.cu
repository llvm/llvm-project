// RUN: %clang_cc1 -fcuda-is-device \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=NOFTZ %s

// RUN: %clang_cc1 -fcuda-is-device -fdenormal-fp-math-f32=ieee \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=NOFTZ %s

// RUN: %clang_cc1 -fcuda-is-device -fdenormal-fp-math-f32=preserve-sign \
// RUN:   -triple nvptx-nvidia-cuda -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=FTZ %s

// RUN: %clang_cc1 -fcuda-is-device -x hip \
// RUN:   -triple amdgcn-amd-amdhsa -target-cpu gfx900 -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=AMDNOFTZ %s

// RUN: %clang_cc1 -fcuda-is-device -x hip \
// RUN:   -triple amdgcn-amd-amdhsa -target-cpu gfx900 -fdenormal-fp-math-f32=ieee -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=AMDNOFTZ %s

// RUN: %clang_cc1 -fcuda-is-device -x hip -fdenormal-fp-math-f32=preserve-sign \
// RUN:   -triple amdgcn-amd-amdhsa -target-cpu gfx900 -emit-llvm -o - %s | \
// RUN:   FileCheck -check-prefix=AMDFTZ %s

#include "Inputs/hip/hip_runtime.h"

// Checks that device function calls get emitted with the "denormal-fp-math-f32"
// attribute set when we compile CUDA device code with
// -fdenormal-fp-math-f32. Further, check that we reflect the presence or
// absence of -fcuda-flush-denormals-to-zero in a module flag.

// AMDGCN targets always have +fp64-fp16-denormals.
// AMDGCN targets without fast FMAF (e.g. gfx803) always have +fp32-denormals.
// For AMDGCN target with fast FMAF (e.g. gfx900), it has +fp32-denormals
// by default and -fp32-denormals when there is option
// -fcuda-flush-denormals-to-zero.

// CHECK-LABEL: define void @foo() #0
extern "C" __device__ void foo() {}

// FTZ: attributes #0 = {{.*}} "denormal-fp-math-f32"="preserve-sign,preserve-sign"
// NOFTZ-NOT: "denormal-fp-math-f32"

// AMDNOFTZ: attributes #0 = {{.*}}+fp32-denormals{{.*}}+fp64-fp16-denormals
// AMDFTZ: attributes #0 = {{.*}}+fp64-fp16-denormals{{.*}}-fp32-denormals

// FTZ:!llvm.module.flags = !{{{.*}}[[MODFLAG:![0-9]+]]}
// FTZ:[[MODFLAG]] = !{i32 4, !"nvvm-reflect-ftz", i32 1}

// NOFTZ:!llvm.module.flags = !{{{.*}}[[MODFLAG:![0-9]+]]}
// NOFTZ:[[MODFLAG]] = !{i32 4, !"nvvm-reflect-ftz", i32 0}
