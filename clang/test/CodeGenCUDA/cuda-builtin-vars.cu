// RUN: %clang_cc1 "-triple" "nvptx-nvidia-cuda" -emit-llvm -fcuda-is-device -o - %s | FileCheck %s


#include <hip/hip_runtime.h>
#include "__clang_cuda_builtin_vars.h"

// CHECK: define{{.*}} void @_Z6kernelPi(ptr noundef %out)
__attribute__((global))
void kernel(int *out) {
  int i = 0;
  out[i++] = threadIdx.x; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.tid.x()
  out[i++] = threadIdx.y; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.tid.y()
  out[i++] = threadIdx.z; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.tid.z()

  out[i++] = blockIdx.x; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ctaid.x()
  out[i++] = blockIdx.y; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ctaid.y()
  out[i++] = blockIdx.z; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ctaid.z()

  out[i++] = blockDim.x; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ntid.x()
  out[i++] = blockDim.y; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ntid.y()
  out[i++] = blockDim.z; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.ntid.z()

  out[i++] = gridDim.x; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.nctaid.x()
  out[i++] = gridDim.y; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.nctaid.y()
  out[i++] = gridDim.z; // CHECK: call noundef i32 @llvm.nvvm.read.ptx.sreg.nctaid.z()

  out[i++] = warpSize; // CHECK: store i32 32,

  // CHECK: ret void
}
