#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -fgpu-rdc -std=c++11 -emit-llvm -o - -target-cpu gfx906 | FileCheck %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -fgpu-rdc -std=c++11 -emit-llvm -o - -target-cpu gfx906 \
// RUN:   | FileCheck -check-prefix=NEG %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++11 -emit-llvm -o - -target-cpu gfx906 \
// RUN:   | FileCheck -check-prefixes=NEG,NORDC %s

#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: @gpu.used.external = appending {{.*}}global
// CHECK-DAG: @_Z7kernel1v
// CHECK-DAG: @_Z7kernel4v
// CHECK-DAG: @var1
// CHECK-LABEL: @llvm.compiler.used = {{.*}} @gpu.used.external

// NEG-NOT: @gpu.used.external = {{.*}} @_Z7kernel2v
// NEG-NOT: @gpu.used.external = {{.*}} @_Z7kernel3v
// NEG-NOT: @gpu.used.external = {{.*}} @var2
// NEG-NOT: @gpu.used.external = {{.*}} @var3
// NORDC-NOT: @gpu.used.external = {{.*}} @_Z7kernel1v
// NORDC-NOT: @gpu.used.external = {{.*}} @_Z7kernel4v
// NORDC-NOT: @gpu.used.external = {{.*}} @var1

__global__ void kernel1();

// kernel2 is not marked as used since it is a definition.
__global__ void kernel2() {}

// kernel3 is not marked as used since it is not called by host function.
__global__ void kernel3();

// kernel4 is marked as used even though it is not called.
__global__ void kernel4();

extern __device__ int var1;

__device__ int var2;

extern __device__ int var3;

void use(int *p);

void test() {
  kernel1<<<1, 1>>>();
  void *p = (void*)kernel4;
  use(&var1);
}
