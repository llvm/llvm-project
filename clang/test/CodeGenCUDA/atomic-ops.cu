#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -x hip -std=c++11 -triple amdgcn -fcuda-is-device -emit-llvm %s -o - | FileCheck %s
#include "Inputs/hip/hip_runtime.h"

// CHECK-LABEL: @_Z24atomic32_op_singlethreadPiii
// CHECK: cmpxchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: load atomic i32, ptr {{%[0-9]+}} syncscope("singlethread-one-as") monotonic, align 4
// CHECK: store atomic i32 %{{.*}}, ptr %{{.*}} syncscope("singlethread-one-as") monotonic, align 4
__device__ int atomic32_op_singlethread(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z25atomicu32_op_singlethreadPjjj
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("singlethread-one-as")
__device__ unsigned int atomicu32_op_singlethread(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return val;
}

// CHECK-LABEL: @_Z21atomic32_op_wavefrontPiii
// CHECK: cmpxchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i32, ptr {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 4
// CHECK: store atomic i32 %{{.*}}, ptr %{{.*}} syncscope("wavefront-one-as") monotonic, align 4
__device__ int atomic32_op_wavefront(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu32_op_wavefrontPjjj
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("wavefront-one-as")
__device__ unsigned int atomicu32_op_wavefront(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return val;
}

// CHECK-LABEL: @_Z21atomic32_op_workgroupPiii
// CHECK: cmpxchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i32 %{{.*}}, ptr %{{.*}} syncscope("workgroup-one-as") monotonic, align 4
__device__ int atomic32_op_workgroup(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu32_op_workgroupPjjj
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("workgroup-one-as")
__device__ unsigned int atomicu32_op_workgroup(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return val;
}

// CHECK-LABEL: @_Z17atomic32_op_agentPiii
// CHECK: cmpxchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i32 %{{.*}}, ptr %{{.*}} syncscope("agent-one-as") monotonic, align 4
__device__ int atomic32_op_agent(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z18atomicu32_op_agentPjjj
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("agent-one-as")
__device__ unsigned int atomicu32_op_agent(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return val;
}

// CHECK-LABEL: @_Z18atomic32_op_systemPiii
// CHECK: cmpxchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i32 {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as") monotonic monotonic, align 4
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i32, ptr %{{.*}}, align 4
// CHECK: store atomic i32 %{{.*}}, ptr %{{.*}} syncscope("one-as") monotonic, align 4
__device__ int atomic32_op_system(int *ptr, int val, int desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z19atomicu32_op_systemPjjj
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i32 {{%[0-9]+}} syncscope("one-as")
__device__ unsigned int atomicu32_op_system(unsigned int *ptr, unsigned int val, unsigned int desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return val;
}

// CHECK-LABEL: @_Z24atomic64_op_singlethreadPxS_xx
// CHECK: cmpxchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
__device__ long long atomic64_op_singlethread(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z25atomicu64_op_singlethreadPyS_yy
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("singlethread-one-as")
// CHECK: load atomic i64, ptr %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("singlethread-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_singlethread(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SINGLETHREAD);
  return val;
}

// CHECK-LABEL: @_Z21atomic64_op_wavefrontPxS_xx
// CHECK: cmpxchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i64, ptr {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("wavefront-one-as") monotonic, align 8
__device__ long long atomic64_op_wavefront(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu64_op_wavefrontPyS_yy
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("wavefront-one-as")
// CHECK: load atomic i64, ptr {{%[0-9]+}} syncscope("wavefront-one-as") monotonic, align 8
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("wavefront-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_wavefront(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WAVEFRONT);
  return val;
}

// CHECK-LABEL: @_Z21atomic64_op_workgroupPxS_xx
// CHECK: cmpxchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("workgroup-one-as") monotonic, align 8
__device__ long long atomic64_op_workgroup(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z22atomicu64_op_workgroupPyS_yy
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("workgroup-one-as")
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("workgroup-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_workgroup(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
  return val;
}

// CHECK-LABEL: @_Z17atomic64_op_agentPxS_xx
// CHECK: cmpxchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("agent-one-as") monotonic, align 8
__device__ long long atomic64_op_agent(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z18atomicu64_op_agentPyS_yy
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("agent-one-as")
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("agent-one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_agent(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
  return val;
}

// CHECK-LABEL: @_Z18atomic64_op_systemPxS_xx
// CHECK: cmpxchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: cmpxchg weak ptr {{%[0-9]+}}, i64 {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as") monotonic monotonic, align 8
// CHECK: atomicrmw xchg ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw add ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw and ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw or ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw xor ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw min ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw max ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i64, ptr %{{.*}}, align 8
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("one-as") monotonic, align 8
__device__ long long atomic64_op_system(long long *ptr, long long *ptr2, long long val, long long desired) {
  bool flag = __hip_atomic_compare_exchange_strong(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  flag = __hip_atomic_compare_exchange_weak(ptr, &val, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_exchange(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_add(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_and(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_or(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_xor(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return flag ? val : desired;
}

// CHECK-LABEL: @_Z19atomicu64_op_systemPyS_yy
// CHECK: atomicrmw umin ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: atomicrmw umax ptr {{%[0-9]+}}, i64 {{%[0-9]+}} syncscope("one-as")
// CHECK: load i64, ptr %{{.*}}, align 8
// CHECK: store atomic i64 %{{.*}}, ptr %{{.*}} syncscope("one-as") monotonic, align 8
__device__ unsigned long long atomicu64_op_system(unsigned long long *ptr, unsigned long long *ptr2, unsigned long long val, unsigned long long desired) {
  val = __hip_atomic_fetch_min(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_fetch_max(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  val = __hip_atomic_load(ptr, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  __hip_atomic_store(ptr, val, __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_SYSTEM);
  return val;
}
