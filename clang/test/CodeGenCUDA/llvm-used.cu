
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple nvptx64-unknown-unknown | FileCheck %s


// Make sure we emit the proper addrspacecast for llvm.used.  PR22383 exposed an
// issue where we were generating a bitcast instead of an addrspacecast.

// CHECK: @llvm.compiler.used = appending global [1 x ptr] [ptr addrspacecast (ptr addrspace(1) @a to ptr)], section "llvm.metadata"
__attribute__((device)) __attribute__((__used__)) int a[] = {};
