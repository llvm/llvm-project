#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple x86_64-pc-windows-msvc -fdump-record-layouts \
// RUN:   -emit-llvm -o %t -xhip %s 2>&1 | FileCheck %s --check-prefix=AST
// RUN: cat %t | FileCheck --check-prefixes=CHECK,HOST %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -target-cpu gfx1100 \
// RUN:   -emit-llvm -fdump-record-layouts -aux-triple x86_64-pc-windows-msvc \
// RUN:   -o %t -xhip %s | FileCheck %s --check-prefix=AST
// RUN: cat %t | FileCheck --check-prefixes=CHECK,DEV %s

#include "Inputs/hip/hip_runtime.h"

// AST: *** Dumping AST Record Layout
// AST-LABEL:         0 | struct C
// AST-NEXT:          0 |   struct A (base) (empty)
// AST-NEXT:          1 |   struct B (base) (empty)
// AST-NEXT:          4 |   int i
// AST-NEXT:            | [sizeof=8, align=4,
// AST-NEXT:            |  nvsize=8, nvalign=4]

// CHECK: %struct.C = type { [4 x i8], i32 }

struct A {};
struct B {};
struct C : A, B {
    int i;
};

// AST: *** Dumping AST Record Layout
// AST-LABEL:          0 | struct I
// AST-NEXT:           0 |   (I vftable pointer)
// AST-NEXT:           8 |   int i
// AST-NEXT:             | [sizeof=16, align=8,
// AST-NEXT:             |  nvsize=16, nvalign=8]

// AST: *** Dumping AST Record Layout
// AST-LABEL:          0 | struct J
// AST-NEXT:           0 |   struct I (primary base)
// AST-NEXT:           0 |     (I vftable pointer)
// AST-NEXT:           8 |     int i
// AST-NEXT:          16 |   int j
// AST-NEXT:             | [sizeof=24, align=8,
// AST-NEXT:             |  nvsize=24, nvalign=8]

// CHECK: %struct.I = type { ptr, i32 }
// CHECK: %struct.J = type { %struct.I, i32 }

// HOST: @0 = private unnamed_addr constant { [4 x ptr] } { [4 x ptr] [ptr @"??_R4J@@6B@", ptr @"?f@J@@UEAAXXZ", ptr null, ptr @"?h@J@@UEAAXXZ"] }, comdat($"??_7J@@6B@")
// HOST: @1 = private unnamed_addr constant { [4 x ptr] } { [4 x ptr] [ptr @"??_R4I@@6B@", ptr @_purecall, ptr null, ptr @_purecall] }, comdat($"??_7I@@6B@")
// HOST: @"??_7J@@6B@" = unnamed_addr alias ptr, getelementptr inbounds ({ [4 x ptr] }, ptr @0, i32 0, i32 0, i32 1)
// HOST: @"??_7I@@6B@" = unnamed_addr alias ptr, getelementptr inbounds ({ [4 x ptr] }, ptr @1, i32 0, i32 0, i32 1)

// DEV: @_ZTV1J = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @_ZN1J1gEv to ptr addrspace(1)), ptr addrspace(1) addrspacecast (ptr @_ZN1J1hEv to ptr addrspace(1))] }, comdat, align 8
// DEV: @_ZTV1I = linkonce_odr unnamed_addr addrspace(1) constant { [5 x ptr addrspace(1)] } { [5 x ptr addrspace(1)] [ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) null, ptr addrspace(1) addrspacecast (ptr @__cxa_pure_virtual to ptr addrspace(1)), ptr addrspace(1) addrspacecast (ptr @__cxa_pure_virtual to ptr addrspace(1))] }, comdat, align 8
struct I {
    virtual void f() = 0;
    __device__ virtual void g() = 0;
    __device__ __host__ virtual void h() = 0;
    int i;
};

struct J : I {
    void f() override {}
    __device__ void g() override {}
    __device__ __host__ void h() override {}
    int j;
};

// DEV: define dso_local amdgpu_kernel void @_Z8C_kernel1C(ptr addrspace(4) noundef byref(%struct.C) align 4 %0)
// DEV:  %coerce = alloca %struct.C, align 4, addrspace(5)
// DEV:  %c = addrspacecast ptr addrspace(5) %coerce to ptr
// DEV:  call void @llvm.memcpy.p0.p4.i64(ptr align 4 %c, ptr addrspace(4) align 4 %0, i64 8, i1 false)
// DEV:  %i = getelementptr inbounds %struct.C, ptr %c, i32 0, i32 1
// DEV:  store i32 1, ptr %i, align 4

__global__ void C_kernel(C c)
{
  c.i = 1;
}

// HOST-LABEL: define dso_local void @"?test_C@@YAXXZ"()
// HOST:  %c = alloca %struct.C, align 4
// HOST:  %i = getelementptr inbounds %struct.C, ptr %c, i32 0, i32 1
// HOST:  store i32 11, ptr %i, align 4

void test_C() {
  C c;
  c.i = 11;
  C_kernel<<<1, 1>>>(c);
}

// DEV: define dso_local void @_Z5J_devP1J(ptr noundef %j)
// DEV:  %j.addr = alloca ptr, align 8, addrspace(5)
// DEV:  %j.addr.ascast = addrspacecast ptr addrspace(5) %j.addr to ptr
// DEV:  store ptr %j, ptr %j.addr.ascast, align 8
// DEV:  %0 = load ptr, ptr %j.addr.ascast, align 8
// DEV:  %i = getelementptr inbounds %struct.I, ptr %0, i32 0, i32 1
// DEV:  store i32 2, ptr %i, align 8
// DEV:  %1 = load ptr, ptr %j.addr.ascast, align 8
// DEV:  %j1 = getelementptr inbounds %struct.J, ptr %1, i32 0, i32 1
// DEV:  store i32 3, ptr %j1, align 8
// DEV:  %2 = load ptr, ptr %j.addr.ascast, align 8
// DEV:  %vtable = load ptr addrspace(1), ptr %2, align 8
// DEV:  %vfn = getelementptr inbounds ptr addrspace(1), ptr addrspace(1) %vtable, i64 1
// DEV:  %3 = load ptr addrspace(1), ptr addrspace(1) %vfn, align 8
// DEV:  call addrspace(1) void %3(ptr noundef nonnull align 8 dereferenceable(24) %2)
// DEV:  %4 = load ptr, ptr %j.addr.ascast, align 8
// DEV:  %vtable2 = load ptr addrspace(1), ptr %4, align 8
// DEV:  %vfn3 = getelementptr inbounds ptr addrspace(1), ptr addrspace(1) %vtable2, i64 2
// DEV:  %5 = load ptr addrspace(1), ptr addrspace(1) %vfn3, align 8
// DEV:  call addrspace(1) void %5(ptr noundef nonnull align 8 dereferenceable(24) %4)

__device__ void J_dev(J *j) {
  j->i = 2;
  j->j = 3;
  j->g();
  j->h();
}

// DEV: define dso_local amdgpu_kernel void @_Z8J_kernelv()
// DEV:  %j = alloca %struct.J, align 8, addrspace(5)
// DEV:  %j.ascast = addrspacecast ptr addrspace(5) %j to ptr
// DEV:  call void @_ZN1JC1Ev(ptr noundef nonnull align 8 dereferenceable(24) %j.ascast)
// DEV:  call void @_Z5J_devP1J(ptr noundef %j.ascast)

__global__ void J_kernel() {
  J j;
  J_dev(&j);
}

// HOST-LABEL: define dso_local void @"?J_host@@YAXPEAUJ@@@Z"(ptr noundef %j)
// HOST:  %0 = load ptr, ptr %j.addr, align 8
// HOST:  %i = getelementptr inbounds %struct.I, ptr %0, i32 0, i32 1
// HOST:  store i32 12, ptr %i, align 8
// HOST:  %1 = load ptr, ptr %j.addr, align 8
// HOST:  %j1 = getelementptr inbounds %struct.J, ptr %1, i32 0, i32 1
// HOST:  store i32 13, ptr %j1, align 8
// HOST:  %2 = load ptr, ptr %j.addr, align 8
// HOST:  %vtable = load ptr, ptr %2, align 8
// HOST:  %vfn = getelementptr inbounds ptr, ptr %vtable, i64 0
// HOST:  %3 = load ptr, ptr %vfn, align 8
// HOST:  call void %3(ptr noundef nonnull align 8 dereferenceable(24) %2)
// HOST:  %4 = load ptr, ptr %j.addr, align 8
// HOST:  %vtable2 = load ptr, ptr %4, align 8
// HOST:  %vfn3 = getelementptr inbounds ptr, ptr %vtable2, i64 2
// HOST:  %5 = load ptr, ptr %vfn3, align 8
// HOST:  call void %5(ptr noundef nonnull align 8 dereferenceable(24) %4)

void J_host(J *j) {
  j->i = 12;
  j->j = 13;
  j->f();
  j->h();
}

// HOST: define dso_local void @"?test_J@@YAXXZ"()
// HOST:  %j = alloca %struct.J, align 8
// HOST:  %call = call noundef ptr @"??0J@@QEAA@XZ"(ptr noundef nonnull align 8 dereferenceable(24) %j)
// HOST:  call void @"?J_host@@YAXPEAUJ@@@Z"(ptr noundef %j)

void test_J() {
  J j;
  J_host(&j);
  J_kernel<<<1, 1>>>();
}

// HOST: define linkonce_odr dso_local noundef ptr @"??0J@@QEAA@XZ"(ptr noundef nonnull returned align 8 dereferenceable(24) %this)
// HOST:  %this.addr = alloca ptr, align 8
// HOST:  store ptr %this, ptr %this.addr, align 8
// HOST:  %this1 = load ptr, ptr %this.addr, align 8
// HOST:  %call = call noundef ptr @"??0I@@QEAA@XZ"(ptr noundef nonnull align 8 dereferenceable(16) %this1) #5
// HOST:  store ptr @"??_7J@@6B@", ptr %this1, align 8
// HOST:  ret ptr %this1

// HOST: define linkonce_odr dso_local noundef ptr @"??0I@@QEAA@XZ"(ptr noundef nonnull returned align 8 dereferenceable(16) %this)
// HOST:  %this.addr = alloca ptr, align 8
// HOST:  store ptr %this, ptr %this.addr, align 8
// HOST:  %this1 = load ptr, ptr %this.addr, align 8
// HOST:  store ptr @"??_7I@@6B@", ptr %this1, align 8
// HOST:  ret ptr %this1

// DEV: define linkonce_odr void @_ZN1JC1Ev(ptr noundef nonnull align 8 dereferenceable(24) %this)
// DEV:  %this.addr = alloca ptr, align 8, addrspace(5)
// DEV:  %this.addr.ascast = addrspacecast ptr addrspace(5) %this.addr to ptr
// DEV:  store ptr %this, ptr %this.addr.ascast, align 8
// DEV:  %this1 = load ptr, ptr %this.addr.ascast, align 8
// DEV:  call void @_ZN1JC2Ev(ptr noundef nonnull align 8 dereferenceable(24) %this1)

// DEV: define linkonce_odr void @_ZN1JC2Ev(ptr noundef nonnull align 8 dereferenceable(24) %this)
// DEV:  %this.addr = alloca ptr, align 8, addrspace(5)
// DEV:  %this.addr.ascast = addrspacecast ptr addrspace(5) %this.addr to ptr
// DEV:  store ptr %this, ptr %this.addr.ascast, align 8
// DEV:  %this1 = load ptr, ptr %this.addr.ascast, align 8
// DEV:  call void @_ZN1IC2Ev(ptr noundef nonnull align 8 dereferenceable(16) %this1)
// DEV:  store ptr addrspace(1) getelementptr inbounds inrange(-16, 24) ({ [5 x ptr addrspace(1)] }, ptr addrspace(1) @_ZTV1J, i32 0, i32 0, i32 2), ptr %this1, align 8

// DEV: define linkonce_odr void @_ZN1IC2Ev(ptr noundef nonnull align 8 dereferenceable(16) %this)
// DEV:  %this.addr = alloca ptr, align 8, addrspace(5)
// DEV:  %this.addr.ascast = addrspacecast ptr addrspace(5) %this.addr to ptr
// DEV:  store ptr %this, ptr %this.addr.ascast, align 8
// DEV:  %this1 = load ptr, ptr %this.addr.ascast, align 8
// DEV:  store ptr addrspace(1) getelementptr inbounds inrange(-16, 24) ({ [5 x ptr addrspace(1)] }, ptr addrspace(1) @_ZTV1I, i32 0, i32 0, i32 2), ptr %this1, align 8
