// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -emit-llvm \
// RUN:   -o - %s | FileCheck %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -emit-llvm -target-sdk-version=10.1 \
// RUN:   -o - %s | FileCheck %s
// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -o - %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

extern "C" {
__device__ int foo() { return 1; }
}

[[gnu::alias("foo")]] __device__ int alias();

// CHECK: @_Z5aliasv = alias i32 (), ptr @foo
//
//      CHECK: define dso_local i32 @foo() #[[ATTR0:[0-9]+]] {
// CHECK-NEXT: entry:
//      CHECK:   ret i32 1
// CHECK-NEXT: }

// RUN: not %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -emit-llvm -target-sdk-version=9.0 \
// RUN:   -o - %s 2>&1 | FileCheck %s --check-prefix=NO_SUPPORT
// NO_SUPPORT: CUDA older than 10.0 does not support .alias
