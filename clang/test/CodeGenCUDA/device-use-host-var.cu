// RUN: %clang_cc1 -std=c++11 -triple amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -emit-llvm -o - -x hip %s | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

struct A {
  int x;
};

constexpr int constexpr_var = 1;
constexpr A constexpr_struct{2};
constexpr A constexpr_array[4] = {0, 0, 0, 3};
constexpr char constexpr_str[] = "abcd";
const int const_var = 4;

// CHECK-DAG: @_ZL13constexpr_str.const = private unnamed_addr addrspace(4) constant [5 x i8] c"abcd\00"
// CHECK-DAG: @_ZL13constexpr_var = internal addrspace(4) constant i32 1
// CHECK-DAG: @_ZL16constexpr_struct = internal addrspace(4) constant %struct.A { i32 2 }
// CHECK-DAG: @_ZL15constexpr_array = internal addrspace(4) constant [4 x %struct.A] [%struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A { i32 3 }]
// CHECK-NOT: external

// CHECK-LABEL: define{{.*}}@_Z7dev_funPiPPKi
// CHECK: store i32 1
// CHECK: store i32 2
// CHECK: store i32 3
// CHECK: store i32 4
// CHECK: load i8, i8* getelementptr {{.*}} @_ZL13constexpr_str.const
// CHECK: store i32* {{.*}}@_ZL13constexpr_var
// CHECK: store i32* getelementptr {{.*}} @_ZL16constexpr_struct
// CHECK: store i32* getelementptr {{.*}} @_ZL15constexpr_array
__device__ void dev_fun(int *out, const int **out2) {
  *out = constexpr_var;
  *out = constexpr_struct.x;
  *out = constexpr_array[3].x;
  *out = const_var;
  *out = constexpr_str[3];
  *out2 = &constexpr_var;
  *out2 = &constexpr_struct.x;
  *out2 = &constexpr_array[3].x;
}
