#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm -x hip %s -o - -triple x86_64-linux-gnu \
// RUN:   | FileCheck -check-prefixes=CHECK,HOST %s
// RUN: %clang_cc1 -emit-llvm -x hip %s -o - -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   | FileCheck -check-prefixes=CHECK,DEV %s

#include "Inputs/hip/hip_runtime.h"

// CHECK: %class.anon = type { ptr, float, ptr, ptr }
// CHECK: %class.anon.0 = type { ptr, float, ptr, ptr }
// CHECK: %class.anon.1 = type { ptr, ptr, ptr }
// CHECK: %class.anon.2 = type { ptr, float, ptr, ptr }

// HOST: call void @_ZN8DevByVal21__device_stub__kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr noundef byval(%class.anon)
// DEV: define amdgpu_kernel void @_ZN8DevByVal6kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr addrspace(4) noundef byref(%class.anon)

// Only the device function passes arugments by value.
namespace DevByVal {
__device__ float fun(float x, float y) {
  return x;
}

float fun(const float &x, const float &y) {
  return x;
}

template<typename F>
void __global__ kernel(F f)
{
  f(1);
}

void test(float const * fl, float const * A, float * Vf)
{
  float constexpr small(1.0e-25);

  auto lambda = [=] __device__ __host__ (unsigned int n) {
    float const value = fun(small, fl[0]);
    Vf[0] = value * A[0];
  };
  kernel<<<1, 1>>>(lambda);
}
}

// HOST: call void @_ZN9HostByVal21__device_stub__kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr noundef byval(%class.anon.0)
// DEV: define amdgpu_kernel void @_ZN9HostByVal6kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr addrspace(4) noundef byref(%class.anon.0)

// Only the host function passes arugments by value.
namespace HostByVal {
float fun(float x, float y) {
  return x;
}

__device__ float fun(const float &x, const float &y) {
  return x;
}

template<typename F>
void __global__ kernel(F f)
{
  f(1);
}

void test(float const * fl, float const * A, float * Vf)
{
  float constexpr small(1.0e-25);

  auto lambda = [=] __device__ __host__ (unsigned int n) {
    float const value = fun(small, fl[0]);
    Vf[0] = value * A[0];
  };
  kernel<<<1, 1>>>(lambda);
}
}

// HOST: call void @_ZN9BothByVal21__device_stub__kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr noundef byval(%class.anon.1)
// DEV: define amdgpu_kernel void @_ZN9BothByVal6kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr addrspace(4) noundef byref(%class.anon.1)

// Both the host and device functions pass arugments by value.
namespace BothByVal {
float fun(float x, float y) {
  return x;
}

__device__ float fun(float x, float y) {
  return x;
}

template<typename F>
void __global__ kernel(F f)
{
  f(1);
}

void test(float const * fl, float const * A, float * Vf)
{
  float constexpr small(1.0e-25);

  auto lambda = [=] __device__ __host__ (unsigned int n) {
    float const value = fun(small, fl[0]);
    Vf[0] = value * A[0];
  };
  kernel<<<1, 1>>>(lambda);
}
}

// HOST: call void @_ZN12NeitherByVal21__device_stub__kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr noundef byval(%class.anon.2)
// DEV: define amdgpu_kernel void @_ZN12NeitherByVal6kernelIZNS_4testEPKfS2_PfEUljE_EEvT_(ptr addrspace(4) noundef byref(%class.anon.2)

// Neither the host nor device function passes arugments by value.
namespace NeitherByVal {
float fun(const float& x, const float& y) {
  return x;
}

__device__ float fun(const float& x, const float& y) {
  return x;
}

template<typename F>
void __global__ kernel(F f)
{
  f(1);
}

void test(float const * fl, float const * A, float * Vf)
{
  float constexpr small(1.0e-25);

  auto lambda = [=] __device__ __host__ (unsigned int n) {
    float const value = fun(small, fl[0]);
    Vf[0] = value * A[0];
  };
  kernel<<<1, 1>>>(lambda);
}
}
