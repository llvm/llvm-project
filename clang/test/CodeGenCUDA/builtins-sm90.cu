
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 "-triple" "nvptx64-nvidia-cuda" "-target-feature" "+ptx80" "-target-cpu" "sm_90" -emit-llvm -fcuda-is-device -o - %s | FileCheck %s

// CHECK: define{{.*}} void @_Z6kernelPlPvj(
__attribute__((global)) void kernel(long *out, void *ptr, unsigned u) {
  int i = 0;
  // CHECK: call i1 @llvm.nvvm.isspacep.shared.cluster
  out[i++] = __nvvm_isspacep_shared_cluster(ptr);

  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.clusterid.x()
  out[i++] = __nvvm_read_ptx_sreg_clusterid_x();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.clusterid.y()
  out[i++] = __nvvm_read_ptx_sreg_clusterid_y();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.clusterid.z()
  out[i++] = __nvvm_read_ptx_sreg_clusterid_z();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.clusterid.w()
  out[i++] = __nvvm_read_ptx_sreg_clusterid_w();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nclusterid.x()
  out[i++] = __nvvm_read_ptx_sreg_nclusterid_x();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nclusterid.y()
  out[i++] = __nvvm_read_ptx_sreg_nclusterid_y();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nclusterid.z()
  out[i++] = __nvvm_read_ptx_sreg_nclusterid_z();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.nclusterid.w()
  out[i++] = __nvvm_read_ptx_sreg_nclusterid_w();

  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.ctaid.x()
  out[i++] = __nvvm_read_ptx_sreg_cluster_ctaid_x();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.ctaid.y()
  out[i++] = __nvvm_read_ptx_sreg_cluster_ctaid_y();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.ctaid.z()
  out[i++] = __nvvm_read_ptx_sreg_cluster_ctaid_z();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.ctaid.w()
  out[i++] = __nvvm_read_ptx_sreg_cluster_ctaid_w();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.nctaid.x()
  out[i++] = __nvvm_read_ptx_sreg_cluster_nctaid_x();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.nctaid.y()
  out[i++] = __nvvm_read_ptx_sreg_cluster_nctaid_y();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.nctaid.z()
  out[i++] = __nvvm_read_ptx_sreg_cluster_nctaid_z();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.nctaid.w()
  out[i++] = __nvvm_read_ptx_sreg_cluster_nctaid_w();

  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.ctarank()
  out[i++] = __nvvm_read_ptx_sreg_cluster_ctarank();
  // CHECK: call i32 @llvm.nvvm.read.ptx.sreg.cluster.nctarank()
  out[i++] = __nvvm_read_ptx_sreg_cluster_nctarank();
  // CHECK: call i1 @llvm.nvvm.is_explicit_cluster()
  out[i++] = __nvvm_is_explicit_cluster();

  auto * sptr = (__attribute__((address_space(3))) void *)ptr;
  // CHECK: call ptr @llvm.nvvm.mapa(ptr %{{.*}}, i32 %{{.*}})
  out[i++] = (long) __nvvm_mapa(ptr, u);
  // CHECK: call ptr addrspace(3) @llvm.nvvm.mapa.shared.cluster(ptr addrspace(3) %{{.*}}, i32 %{{.*}})
  out[i++] = (long) __nvvm_mapa_shared_cluster(sptr, u);
  // CHECK: call i32 @llvm.nvvm.getctarank(ptr {{.*}})
  out[i++] = __nvvm_getctarank(ptr);
  // CHECK: call i32 @llvm.nvvm.getctarank.shared.cluster(ptr addrspace(3) {{.*}})
  out[i++] = __nvvm_getctarank_shared_cluster(sptr);

  // CHECK: call void @llvm.nvvm.barrier.cluster.arrive()
  __nvvm_barrier_cluster_arrive();
  // CHECK: call void @llvm.nvvm.barrier.cluster.arrive.relaxed()
  __nvvm_barrier_cluster_arrive_relaxed();
  // CHECK: call void @llvm.nvvm.barrier.cluster.wait()
  __nvvm_barrier_cluster_wait();
  // CHECK: call void @llvm.nvvm.fence.sc.cluster()
  __nvvm_fence_sc_cluster();

  // CHECK: ret void
}
