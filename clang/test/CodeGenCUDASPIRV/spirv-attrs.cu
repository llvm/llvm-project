#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64 -o - -emit-llvm -x cuda %s  | FileCheck %s
// RUN: %clang_cc1 -fcuda-is-device -triple spirv32 -o - -emit-llvm -x cuda %s  | FileCheck %s

#define __global__ __attribute__((global))

__attribute__((reqd_work_group_size(0, 0, 0)))
__global__ void reqd_work_group_size_0_0_0() {}

__attribute__((reqd_work_group_size(128, 1, 1)))
__global__ void reqd_work_group_size_128_1_1() {}

__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}

__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}

__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}

template <unsigned a, unsigned b, unsigned c>
__attribute__((reqd_work_group_size(a, b, c)))
__global__ void reqd_work_group_size_a_b_c() {}

template __global__ void reqd_work_group_size_a_b_c<256,1,1>(void);

// CHECK: define spir_kernel void @_Z26reqd_work_group_size_0_0_0v() #[[ATTR:[0-9]+]] !reqd_work_group_size ![[WG_SIZE_ZEROS:[0-9]+]]
// CHECK: define spir_kernel void @_Z28reqd_work_group_size_128_1_1v() #[[ATTR:[0-9]+]] !reqd_work_group_size ![[WG_SIZE:[0-9]+]]
// CHECK: define spir_kernel void @_Z26work_group_size_hint_2_2_2v() #[[ATTR]] !work_group_size_hint ![[WG_HINT:[0-9]+]]
// CHECK: define spir_kernel void @_Z17vec_type_hint_intv() #[[ATTR]] !vec_type_hint ![[VEC_HINT:[0-9]+]]
// CHECK: define spir_kernel void @_Z28intel_reqd_sub_group_size_64v() #[[ATTR]] !intel_reqd_sub_group_size ![[SUB_GRP:[0-9]+]]
// CHECK: define spir_kernel void @_Z26reqd_work_group_size_a_b_cILj256ELj1ELj1EEvv() #[[ATTR]] comdat !reqd_work_group_size ![[WG_SIZE_TMPL:[0-9]+]]

// CHECK: attributes #[[ATTR]] = { {{.*}} }

// CHECK: ![[WG_SIZE_ZEROS]] = !{i32 0, i32 0, i32 0}
// CHECK: ![[WG_SIZE]] = !{i32 128, i32 1, i32 1}
// CHECK: ![[WG_HINT]] = !{i32 2, i32 2, i32 2}
// CHECK: ![[VEC_HINT]] = !{i32 poison, i32 1}
// CHECK: ![[SUB_GRP]] = !{i32 64}
// CHECK: ![[WG_SIZE_TMPL]] = !{i32 256, i32 1, i32 1}
