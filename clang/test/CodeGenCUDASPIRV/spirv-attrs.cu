#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64 -o - -emit-llvm -x cuda %s  | FileCheck %s
// RUN: %clang_cc1 -fcuda-is-device -triple spirv32 -o - -emit-llvm -x cuda %s  | FileCheck %s

#define __global__ __attribute__((global))

__attribute__((reqd_work_group_size(128, 1, 1)))
__global__ void reqd_work_group_size_128_1_1() {}

__attribute__((work_group_size_hint(2, 2, 2)))
__global__ void work_group_size_hint_2_2_2() {}

__attribute__((vec_type_hint(int)))
__global__ void vec_type_hint_int() {}

__attribute__((intel_reqd_sub_group_size(64)))
__global__ void intel_reqd_sub_group_size_64() {}

// CHECK: define spir_kernel void @_Z28reqd_work_group_size_128_1_1v() #[[ATTR:[0-9]+]] !reqd_work_group_size ![[WG_SIZE:[0-9]+]]
// CHECK: define spir_kernel void @_Z26work_group_size_hint_2_2_2v() #[[ATTR]] !work_group_size_hint ![[WG_HINT:[0-9]+]]
// CHECK: define spir_kernel void @_Z17vec_type_hint_intv() #[[ATTR]] !vec_type_hint ![[VEC_HINT:[0-9]+]]
// CHECK: define spir_kernel void @_Z28intel_reqd_sub_group_size_64v() #[[ATTR]] !intel_reqd_sub_group_size ![[SUB_GRP:[0-9]+]]

// CHECK: attributes #[[ATTR]] = { {{.*}} }

// CHECK: ![[WG_SIZE]] = !{i32 128, i32 1, i32 1}
// CHECK: ![[WG_HINT]] = !{i32 2, i32 2, i32 2}
// CHECK: ![[VEC_HINT]] = !{i32 undef, i32 1}
// CHECK: ![[SUB_GRP]] = !{i32 64}
