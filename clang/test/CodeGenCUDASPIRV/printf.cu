
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fcuda-is-device -triple spirv32 -o - -emit-llvm -x cuda %s  | FileCheck --check-prefix=CHECK-SPIRV32 %s
// RUN: %clang_cc1 -fcuda-is-device -triple spirv64 -o - -emit-llvm -x cuda %s  | FileCheck --check-prefix=CHECK-SPIRV64 %s

// CHECK-SPIRV32: @.str = private unnamed_addr addrspace(4) constant [13 x i8] c"Hello World\0A\00", align 1
// CHECK-SPIRV64: @.str = private unnamed_addr addrspace(1) constant [13 x i8] c"Hello World\0A\00", align 1

extern "C" __attribute__((device)) int printf(const char* format, ...);

__attribute__((global)) void printf_kernel() {
  printf("Hello World\n");
}
