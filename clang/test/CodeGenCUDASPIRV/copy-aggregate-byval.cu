
#include <hip/hip_runtime.h>
// Tests CUDA kernel arguments get copied by value when targeting SPIR-V, even with
// destructor, copy constructor or move constructor defined by user.

// RUN: %clang -Xclang -no-opaque-pointers -emit-llvm --cuda-device-only --offload=spirv32 \
// RUN:   -nocudalib -nocudainc %s -o %t.bc -c 2>&1
// RUN: llvm-dis %t.bc -o %t.ll
// RUN: FileCheck %s --input-file=%t.ll

// RUN: %clang -Xclang -no-opaque-pointers -emit-llvm --cuda-device-only --offload=spirv64 \
// RUN:   -nocudalib -nocudainc %s -o %t.bc -c 2>&1
// RUN: llvm-dis %t.bc -o %t.ll
// RUN: FileCheck %s --input-file=%t.ll

class GpuData {
 public:
  __attribute__((host)) __attribute__((device)) GpuData(int* src) {}
  __attribute__((host)) __attribute__((device)) ~GpuData() {}
  __attribute__((host)) __attribute__((device)) GpuData(const GpuData& other) {}
  __attribute__((host)) __attribute__((device)) GpuData(GpuData&& other) {}
};

// CHECK: define
// CHECK-SAME: spir_kernel void @_Z6kernel7GpuData(%class.GpuData* noundef byval(%class.GpuData) align

__attribute__((global)) void kernel(GpuData output) {}
