
#include <hip/hip_runtime.h>
// Tests that "sm_XX" gets correctly converted to "compute_YY" when we invoke
// fatbinary.

// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM20 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_21 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM21 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_30 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM30 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_32 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM32 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_35 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM35 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_37 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM37 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_50 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM50 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_52 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM52 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_53 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM53 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM60 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_61 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM61 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_62 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM62 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_70 --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM70 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx600 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX600 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx601 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX601 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx602 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX602 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx700 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX700 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx701 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX701 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx702 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX702 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx703 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX703 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx704 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX704 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx705 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX705 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx801 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX801 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx802 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX802 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx803 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX803 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx805 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX805 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx810 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX810 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx900 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX900 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx902 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX902 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=amdgcnspirv -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,SPIRV %s

// CUDA: ptxas
// CUDA-SAME: -m64
// CUDA: fatbinary

// HIP: clang-offload-bundler

// SM20:--image3=kind=elf,sm=20{{.*}}
// SM21:--image3=kind=elf,sm=21{{.*}}
// SM30:--image3=kind=elf,sm=30{{.*}}
// SM32:--image3=kind=elf,sm=32{{.*}}
// SM35:--image3=kind=elf,sm=35{{.*}}
// SM37:--image3=kind=elf,sm=37{{.*}}
// SM50:--image3=kind=elf,sm=50{{.*}}
// SM52:--image3=kind=elf,sm=52{{.*}}
// SM53:--image3=kind=elf,sm=53{{.*}}
// SM60:--image3=kind=elf,sm=60{{.*}}
// SM61:--image3=kind=elf,sm=61{{.*}}
// SM62:--image3=kind=elf,sm=62{{.*}}
// SM70:--image3=kind=elf,sm=70{{.*}}
// GFX600:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx600
// GFX601:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx601
// GFX602:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx602
// GFX700:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx700
// GFX701:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx701
// GFX702:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx702
// GFX703:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx703
// GFX704:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx704
// GFX705:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx705
// GFX801:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx801
// GFX802:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx802
// GFX803:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx803
// GFX805:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx805
// GFX810:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx810
// GFX900:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx900
// GFX902:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx902
// SPIRV:-targets=host-x86_64-unknown-linux-gnu,hip-spirv64-amd-amdhsa--amdgcnspirv
