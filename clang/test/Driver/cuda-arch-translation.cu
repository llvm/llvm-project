
#include <hip/hip_runtime.h>
// Tests that "sm_XX" gets correctly converted to "compute_YY" when we invoke
// fatbinary.

// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM20 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_21 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM21 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_30 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM30 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_32 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM32 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_35 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM35 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_37 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM37 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_50 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM50 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_52 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM52 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_53 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM53 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM60 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_61 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM61 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_62 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM62 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_70 --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM70 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx600 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX600 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx601 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX601 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx602 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX602 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx700 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX700 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx701 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX701 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx702 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX702 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx703 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX703 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx704 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX704 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx705 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX705 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx801 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX801 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx802 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX802 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx803 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX803 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx805 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX805 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx810 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX810 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx900 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX900 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx902 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX902 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=amdgcnspirv -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,SPIRV %s

// CUDA: ptxas
// CUDA-SAME: -m64
// CUDA: fatbinary

// HIP: llvm-offload-binary

// SM20:--image3=kind=elf,sm=20{{.*}}
// SM21:--image3=kind=elf,sm=21{{.*}}
// SM30:--image3=kind=elf,sm=30{{.*}}
// SM32:--image3=kind=elf,sm=32{{.*}}
// SM35:--image3=kind=elf,sm=35{{.*}}
// SM37:--image3=kind=elf,sm=37{{.*}}
// SM50:--image3=kind=elf,sm=50{{.*}}
// SM52:--image3=kind=elf,sm=52{{.*}}
// SM53:--image3=kind=elf,sm=53{{.*}}
// SM60:--image3=kind=elf,sm=60{{.*}}
// SM61:--image3=kind=elf,sm=61{{.*}}
// SM62:--image3=kind=elf,sm=62{{.*}}
// SM70:--image3=kind=elf,sm=70{{.*}}
// GFX600:triple=amdgcn-amd-amdhsa,arch=gfx600
// GFX601:triple=amdgcn-amd-amdhsa,arch=gfx601
// GFX602:triple=amdgcn-amd-amdhsa,arch=gfx602
// GFX700:triple=amdgcn-amd-amdhsa,arch=gfx700
// GFX701:triple=amdgcn-amd-amdhsa,arch=gfx701
// GFX702:triple=amdgcn-amd-amdhsa,arch=gfx702
// GFX703:triple=amdgcn-amd-amdhsa,arch=gfx703
// GFX704:triple=amdgcn-amd-amdhsa,arch=gfx704
// GFX705:triple=amdgcn-amd-amdhsa,arch=gfx705
// GFX801:triple=amdgcn-amd-amdhsa,arch=gfx801
// GFX802:triple=amdgcn-amd-amdhsa,arch=gfx802
// GFX803:triple=amdgcn-amd-amdhsa,arch=gfx803
// GFX805:triple=amdgcn-amd-amdhsa,arch=gfx805
// GFX810:triple=amdgcn-amd-amdhsa,arch=gfx810
// GFX900:triple=amdgcn-amd-amdhsa,arch=gfx900
// GFX902:triple=amdgcn-amd-amdhsa,arch=gfx902
// SPIRV:triple=spirv64-amd-amdhsa,arch=amdgcnspirv
