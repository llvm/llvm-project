
#include <hip/hip_runtime.h>
// Tests the phases generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.

// REQUIRES: clang-driver
// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target
// REQUIRES: amdgpu-registered-target
//
// Test single gpu architecture with complete compilation.
//
// Test CUDA NVPTX phases.
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN,BIN_NV %s
//
// Test HIP AMDGPU -fgpu-rdc phases.
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 -fgpu-rdc %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN,BIN_AMD,BIN_AMD_RDC %s
//
// Test HIP AMDGPU -fno-gpu-rdc phases (default).
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN,BIN_AMD,BIN_AMD_NRDC %s
//
// BIN_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// BIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN_NV-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH:sm_30]])
// BIN_AMD-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH:gfx803]])
// BIN-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// BIN-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH]])
// BIN_NV-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH]])
// BIN_NV-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH]])
// BIN_NV-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P7]]}, object
// BIN_NV-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH]])" {[[P6]]}, assembler
// BIN_NV-DAG: [[P10:[0-9]+]]: linker, {[[P8]], [[P9]]}, cuda-fatbin, (device-[[T]])
// BIN_NV-DAG: [[P11:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P10]]}, ir
// BIN_NV-DAG: [[P12:[0-9]+]]: backend, {[[P11]]}, assembler, (host-[[T]])
// BIN_AMD_RDC-DAG: [[P12:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// BIN_AMD_NRDC-DAG: [[P6:[0-9]+]]: linker, {[[P5]]}, image, (device-hip, [[ARCH]])
// BIN_AMD_NRDC-DAG: [[P7:[0-9]+]]: offload, "device-hip (amdgcn-amd-amdhsa:[[ARCH]])" {[[P6]]}, image
// BIN_AMD_NRDC-DAG: [[P8:[0-9]+]]: linker, {[[P7]]}, hip-fatbin, (device-hip)
// BIN_AMD_NRDC-DAG: [[P11:[0-9]+]]: offload, "host-hip (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-hip (amdgcn-amd-amdhsa)" {[[P8]]}, ir
// BIN_AMD_NRDC-DAG: [[P12:[0-9]+]]: backend, {[[P11]]}, assembler, (host-[[T]])
// BIN-DAG: [[P13:[0-9]+]]: assembler, {[[P12]]}, object, (host-[[T]])
// BIN-DAG: [[P14:[0-9]+]]: linker, {[[P13]]}, image, (host-[[T]])
// BIN_AMD_RDC-DAG: [[P15:[0-9]+]]: backend, {[[P5]]}, ir, (device-[[T]], [[ARCH]])
// BIN_AMD_RDC-DAG: [[P16:[0-9]+]]: linker, {[[P15]]}, image, (device-[[T]], [[ARCH]])
// BIN_AMD_RDC-DAG: [[P17:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P14]]},
// BIN_AMD_RDC-DAG-SAME:  "device-[[T]] ([[TRIPLE:amdgcn-amd-amdhsa]]:[[ARCH]])" {[[P16]]}, object

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM,ASM_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 -fgpu-rdc %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM,ASM_AMD %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 -fcuda-rdc %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM,ASM_AMD %s
// ASM_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// ASM_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH:gfx803]])
// ASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// ASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// ASM_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// ASM_NV-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda|amdgcn-amd-amdhsa]]:[[ARCH]])" {[[P3]]}, assembler
// ASM-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (host-[[T]])
// ASM-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (host-[[T]])
// ASM-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (host-[[T]])

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,BIN2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -fgpu-rdc %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,BIN2_AMD %s
// BIN2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// BIN2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// BIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// BIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// BIN2-DAG: [[P3:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH1:sm_30|gfx803]])
// BIN2-DAG: [[P4:[0-9]+]]: preprocessor, {[[P3]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// BIN2-DAG: [[P5:[0-9]+]]: compiler, {[[P4]]}, ir, (device-[[T]], [[ARCH1]])
// BIN2_NV-DAG: [[P6:[0-9]+]]: backend, {[[P5]]}, assembler, (device-[[T]], [[ARCH1]])
// BIN2_NV-DAG: [[P7:[0-9]+]]: assembler, {[[P6]]}, object, (device-[[T]], [[ARCH1]])
// BIN2_NV-DAG: [[P8:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH1]])" {[[P7]]}, object
// BIN2_NV-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH1]])" {[[P6]]}, assembler
// BIN2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35|gfx900]])
// BIN2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// BIN2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (device-[[T]], [[ARCH2]])
// BIN2_NV-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (device-[[T]], [[ARCH2]])
// BIN2_NV-DAG: [[P14:[0-9]+]]: assembler, {[[P13]]}, object, (device-[[T]], [[ARCH2]])
// BIN2_NV-DAG: [[P15:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P14]]}, object
// BIN2_NV-DAG: [[P16:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P13]]}, assembler
// BIN2_NV-DAG: [[P17:[0-9]+]]: linker, {[[P8]], [[P9]], [[P15]], [[P16]]}, cuda-fatbin, (device-[[T]])
// BIN2_NV-DAG: [[P18:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P2]]}, "device-[[T]] ([[TRIPLE]])" {[[P17]]}, ir
// BIN2_NV-DAG: [[P19:[0-9]+]]: backend, {[[P18]]}, assembler, (host-[[T]])
// BIN2_AMD-DAG: [[P19:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// BIN2-DAG: [[P20:[0-9]+]]: assembler, {[[P19]]}, object, (host-[[T]])
// BIN2-DAG: [[P21:[0-9]+]]: linker, {[[P20]]}, image, (host-[[T]])
// BIN2_AMD-DAG: [[P22:[0-9]+]]: backend, {[[P5]]}, ir, (device-[[T]], [[ARCH1]])
// BIN2_AMD-DAG: [[P23:[0-9]+]]: backend, {[[P12]]}, ir, (device-[[T]], [[ARCH2]])
// BIN2_AMD-DAG: [[P24:[0-9]+]]: linker, {[[P22]]}, image, (device-[[T]], [[ARCH1]])
// BIN2_AMD-DAG: [[P25:[0-9]+]]: linker, {[[P23]]}, image, (device-[[T]], [[ARCH2]])
// BIN2_AMD-DAG: [[P26:[0-9]+]]: offload, "host-[[T]] (powerpc64le-ibm-linux-gnu)" {[[P21]]},
// BIN2_AMD-DAG-SAME:  "device-[[T]] ([[TRIPLE:amdgcn-amd-amdhsa]]:[[ARCH1]])" {[[P24]]},
// BIN2_AMD-DAG-SAME:  "device-[[T]] ([[TRIPLE:amdgcn-amd-amdhsa]]:[[ARCH2]])" {[[P25]]}, object

//
// Test two gpu architecturess up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM2,ASM2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -fgpu-rdc %s -S 2>&1 \
// RUN: | FileCheck -check-prefixes=ASM2,ASM2_AMD %s
// ASM2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH1:sm_30]])
// ASM2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH1:gfx803]])
// ASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH1]])
// ASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH1]])
// ASM2_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH1]])
// ASM2_NV-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda|amdgcn-amd-amdhsa]]:[[ARCH1]])" {[[P3]]}, assembler
// ASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35|gfx900]])
// ASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// ASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// ASM2_NV-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// ASM2_NV-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// ASM2-DAG: [[P10:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (host-[[T]])
// ASM2-DAG: [[P11:[0-9]+]]: preprocessor, {[[P10]]}, [[T]]-cpp-output, (host-[[T]])
// ASM2-DAG: [[P12:[0-9]+]]: compiler, {[[P11]]}, ir, (host-[[T]])
// ASM2-DAG: [[P13:[0-9]+]]: backend, {[[P12]]}, assembler, (host-[[T]])

//
// Test single gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN,HBIN_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN,HBIN_AMD %s
// HBIN_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// HBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN-NOT: device
//
// Test single gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=HASM,HASM_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=HASM,HASM_AMD %s
// HASM_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// HASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM-NOT: device

//
// Test two gpu architectures with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN2,HBIN2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefixes=HBIN2,HBIN2_AMD %s
// HBIN2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HBIN2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// HBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HBIN2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HBIN2-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (host-[[T]])
// HBIN2-DAG: [[P5:[0-9]+]]: linker, {[[P4]]}, image, (host-[[T]])
// HBIN2-NOT: device

//
// Test two gpu architectures up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=HASM2,HASM2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 %s --cuda-host-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=HASM2,HASM2_AMD %s
// HASM2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (host-[[T]])
// HASM2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (host-[[T]])
// HASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (host-[[T]])
// HASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (host-[[T]])
// HASM2-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (host-[[T]])
// HASM2-NOT: device

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN,DBIN_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN,DBIN_AMD %s
// DBIN_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH:gfx803]])
// DBIN-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN_NV-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN_NV-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] (nvptx64-nvidia-cuda:[[ARCH]])" {[[P4]]}, object
// DBIN-NOT: host
//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=DASM,DASM_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=DASM,DASM_AMD %s
// DASM_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH:gfx803]])
// DASM-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM_NV-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda|amdgcn-amd-amdhsa]]:[[ARCH]])" {[[P3]]}, assembler
// DASM-NOT: host

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefixes=DBIN2,DBIN2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 %s --cuda-device-only \
// RUN: 2>&1 | FileCheck -check-prefixes=DBIN2,DBIN2_AMD %s
// DBIN2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DBIN2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH:gfx803]])
// DBIN2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DBIN2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DBIN2_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DBIN2_NV-DAG: [[P4:[0-9]+]]: assembler, {[[P3]]}, object, (device-[[T]], [[ARCH]])
// DBIN2_NV-DAG: [[P5:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda]]:[[ARCH]])" {[[P4]]}, object
// DBIN2-DAG: [[P6:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35|gfx900]])
// DBIN2-DAG: [[P7:[0-9]+]]: preprocessor, {[[P6]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DBIN2-DAG: [[P8:[0-9]+]]: compiler, {[[P7]]}, ir, (device-[[T]], [[ARCH2]])
// DBIN2_NV-DAG: [[P9:[0-9]+]]: backend, {[[P8]]}, assembler, (device-[[T]], [[ARCH2]])
// DBIN2_NV-DAG: [[P10:[0-9]+]]: assembler, {[[P9]]}, object, (device-[[T]], [[ARCH2]])
// DBIN2_NV-DAG: [[P11:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P10]]}, object
// DBIN2-NOT: host
//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-phases \
// RUN: --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S \
// RUN: 2>&1 | FileCheck -check-prefixes=DASM2,DASM2_NV %s
// RUN: %clang -x hip -target powerpc64le-ibm-linux-gnu \
// RUN: -ccc-print-phases --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 %s \
// RUN: --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefixes=DASM2,DASM2_AMD %s
// DASM2_NV-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:cuda]], (device-[[T]], [[ARCH:sm_30]])
// DASM2_AMD-DAG: [[P0:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T:hip]], (device-[[T]], [[ARCH:gfx803]])
// DASM2-DAG: [[P1:[0-9]+]]: preprocessor, {[[P0]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH]])
// DASM2-DAG: [[P2:[0-9]+]]: compiler, {[[P1]]}, ir, (device-[[T]], [[ARCH]])
// DASM2_NV-DAG: [[P3:[0-9]+]]: backend, {[[P2]]}, assembler, (device-[[T]], [[ARCH]])
// DASM2_NV-DAG: [[P4:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE:nvptx64-nvidia-cuda|amdgcn-amd-amdhsa]]:[[ARCH]])" {[[P3]]}, assembler
// DASM2-DAG: [[P5:[0-9]+]]: input, "{{.*}}cuda-phases.cu", [[T]], (device-[[T]], [[ARCH2:sm_35|gfx900]])
// DASM2-DAG: [[P6:[0-9]+]]: preprocessor, {[[P5]]}, [[T]]-cpp-output, (device-[[T]], [[ARCH2]])
// DASM2-DAG: [[P7:[0-9]+]]: compiler, {[[P6]]}, ir, (device-[[T]], [[ARCH2]])
// DASM2_NV-DAG: [[P8:[0-9]+]]: backend, {[[P7]]}, assembler, (device-[[T]], [[ARCH2]])
// DASM2_NV-DAG: [[P9:[0-9]+]]: offload, "device-[[T]] ([[TRIPLE]]:[[ARCH2]])" {[[P8]]}, assembler
// DASM2-NOT: host
