
#include <hip/hip_runtime.h>
// REQUIRES: !system-solaris

// The test cannot be run on Darwin because /Users will be treated as a MSVC option.

// RUN: %clang_cl -### -target x86_64-pc-windows-msvc --offload-arch=sm_35 -fgpu-rdc \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda \
// RUN:   /Wall -x cuda -- %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix=CUDA

// RUN: %clang_cl -### -target x86_64-pc-windows-msvc --offload-arch=gfx1010 -fgpu-rdc --hip-link \
// RUN:   --rocm-path=%S/Inputs/rocm /Wall -x hip -- %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix=HIP

// CUDA: "-cc1" "-triple" "nvptx64-nvidia-cuda" "-aux-triple" "x86_64-pc-windows-msvc"
// CUDA-SAME: "-Weverything"
// CUDA: ptxas
// CUDA: "-cc1" "-triple" "x86_64-pc-windows-msvc{{.*}}" "-aux-triple" "nvptx64-nvidia-cuda"
// CUDA-SAME: "-Weverything"
// CUDA: link

// HIP: "-cc1" "-triple" "x86_64-pc-windows-msvc{{.*}}" "-aux-triple" "amdgcn-amd-amdhsa"
// HIP-SAME: "-Weverything"
// HIP: "-cc1" "-triple" "amdgcn-amd-amdhsa" "-aux-triple" "x86_64-pc-windows-msvc"
// HIP-SAME: "-Weverything"
// HIP: {{lld.* "-flavor" "gnu" "-m" "elf64_amdgpu"}}
// HIP: {{link.* "amdhip64.lib"}}

// CMake uses this option when finding packages for HIP, so
// make sure it does not cause error.

// RUN: %clang_cl --print-libgcc-file-name
