
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed -fcuda-short-ptr

// RUN: %clang -### --target=x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fcuda-short-ptr -nocudainc -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 | FileCheck %s

// CHECK: "-mllvm" "--nvptx-short-ptr"
// CHECK-SAME: "-fcuda-short-ptr"
