
#include <hip/hip_runtime.h>
// Check that -fno-thread-safe-statics get passed down to device-side
// compilation only.
//
// RUN: %clang -### -x cuda --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 %s \
// RUN:            -nocudainc -nocudalib --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda \
// RUN:            2>&1 | FileCheck %s

// RUN: %clang -### -x hip --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx1010 %s \
// RUN:            -nocudainc -nocudalib 2>&1 | FileCheck %s
//
// CHECK: "-fcuda-is-device"
// CHECK-SAME: "-fno-threadsafe-statics"
// CHECK: "-triple" "x86_64-unknown-linux-gnu"
// CHECK-NOT: "-fno-threadsafe-statics"
