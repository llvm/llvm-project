
#include <hip/hip_runtime.h>

// Check that we raise an error if we're trying to compile CUDA code but can't
// find a CUDA install, unless -nocudainc was passed.

// RUN: not %clang -### --sysroot=%s/no-cuda-there --cuda-path-ignore-env %s 2>&1 | FileCheck %s --check-prefix ERR
// RUN: not %clang -### --cuda-path=%s/no-cuda-there %s 2>&1 | FileCheck %s --check-prefix ERR
// ERR: cannot find CUDA installation

// RUN: not %clang -### -nocudainc --sysroot=%s/no-cuda-there --cuda-path-ignore-env %s 2>&1 | FileCheck %s --check-prefix OK
// RUN: not %clang -### -nocudainc --cuda-path=%s/no-cuda-there %s 2>&1 | FileCheck %s --check-prefix OK
// OK-NOT: cannot find CUDA installation
