
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed
// -fcuda-flush-denormals-to-zero. This should be translated to
// -fdenormal-fp-math-f32=preserve-sign

// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_20 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_20 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_10 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_10 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// CPUFTZ-NOT: -fdenormal-fp-math

// FTZ-NOT: -fdenormal-fp-math-f32=
// FTZ: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"

// The default of ieee is omitted
// NOFTZ-NOT: "-fdenormal-fp-math"
// NOFTZ-NOT: "-fdenormal-fp-math-f32"
