
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed
// -fcuda-flush-denormals-to-zero. This should be translated to
// -fdenormal-fp-math-f32=preserve-sign

// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test explicit argument.
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fcuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fno-cuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fcuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fno-cuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test the default changing with no argument based on the subtarget.
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// CPUFTZ-NOT: -fdenormal-fp-math

// FTZ-NOT: -fdenormal-fp-math-f32=
// FTZ: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"

// The default of ieee is omitted
// NOFTZ-NOT: "-fdenormal-fp-math"
// NOFTZ-NOT: "-fdenormal-fp-math-f32"
