
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed
// -fcuda-flush-denormals-to-zero. This should be translated to
// -fdenormal-fp-math-f32=preserve-sign

// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_20 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_20 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_10 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell--cuda-gpu-arch=sm_10 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// CPUFTZ-NOT: -fdenormal-fp-math

// FTZ: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// NOFTZ: "-fdenormal-fp-math=ieee,ieee"
