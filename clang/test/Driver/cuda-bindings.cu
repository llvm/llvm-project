
#include <hip/hip_runtime.h>
// Tests the bindings generated for a CUDA offloading target for different
// combinations of:
// - Number of gpu architectures;
// - Host/device-only compilation;
// - User-requested final phase - binary or assembly.
// It parallels cuda-phases.cu test, but verifies whether output file is temporary or not.

// It's hard to check whether file name is temporary in a portable
// way. Instead we check whether we've generated a permanent name on
// device side, which appends '-device-cuda-<triple>' suffix.

// REQUIRES: powerpc-registered-target
// REQUIRES: nvptx-registered-target

//
// Test single gpu architecture with complete compilation.
// No intermediary device files should have "-device-cuda..." in the name.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings --cuda-gpu-arch=sm_30 %s 2>&1 \
// RUN: | FileCheck -check-prefix=BIN %s
// BIN: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "nvptx64-nvidia-cuda" - "NVPTX::Linker",{{.*}} output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// BIN-NOT: cuda-bindings-device-cuda-nvptx64
// BIN: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"

//
// Test single gpu architecture up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings --cuda-gpu-arch=sm_30 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM %s
// ASM-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// ASM-DAG: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test two gpu architectures with complete compilation.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,AOUT %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:       --offload-arch=sm_30,sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,AOUT %s
// .. same, but with explicitly specified output.
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:       --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,TOUT %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --offload-arch=sm_30,sm_35 %s -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefixes=BIN2,TOUT %s
// BIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Linker",{{.*}} output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// BIN2: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// BIN2-NOT: cuda-bindings-device-cuda-nvptx64
// AOUT: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"
// TOUT: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "{{.*}}/out"

// .. same, but with -fsyntax-only
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:       --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefix=SYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:        --offload-arch=sm_30,sm_35 %s -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefix=SYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:       --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s 2>&1 \
// RUN: | FileCheck -check-prefix=SYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:        --offload-arch=sm_30,sm_35 %s -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefix=SYN %s
// SYN-NOT: inputs:
// SYN: # "powerpc64le-ibm-linux-gnu" - "clang", inputs: [{{.*}}], output: (nothing)
// SYN-NEXT: # "nvptx64-nvidia-cuda" - "clang", inputs: [{{.*}}], output: (nothing)
// SYN-NEXT: # "nvptx64-nvidia-cuda" - "clang", inputs: [{{.*}}], output: (nothing)
// SYN-NOT: inputs

// .. and with --offload-new-driver
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:       --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 --offload-new-driver %s 2>&1 \
// RUN: | FileCheck -check-prefix=NDSYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:        --offload-arch=sm_30,sm_35 %s --offload-new-driver -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefix=NDSYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:       --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --offload-new-driver 2>&1 \
// RUN: | FileCheck -check-prefix=NDSYN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings -fsyntax-only \
// RUN:        --offload-arch=sm_30,sm_35 %s --offload-new-driver -o %t/out 2>&1 \
// RUN: | FileCheck -check-prefix=NDSYN %s
// NDSYN-NOT: inputs:
// NDSYN: # "nvptx64-nvidia-cuda" - "clang", inputs: [{{.*}}], output: (nothing)
// NDSYN-NEXT: # "nvptx64-nvidia-cuda" - "clang", inputs: [{{.*}}], output: (nothing)
// NDSYN-NEXT: # "powerpc64le-ibm-linux-gnu" - "clang", inputs: [{{.*}}], output: (nothing)
// NDSYN-NOT: inputs:


//
// Test two gpu architectures up to the assemble phase.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s -S 2>&1 \
// RUN: | FileCheck -check-prefix=ASM2 %s
// ASM2-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// ASM2-DAG: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.s"
// ASM2-DAG: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test one or more gpu architecture with complete compilation in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only 2>&1 \
// RUN: | FileCheck -check-prefix=HBIN %s
// HBIN: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}}  output:
// HBIN-NOT: cuda-bindings-device-cuda-nvptx64
// HBIN: # "powerpc64le-ibm-linux-gnu" - "GNU::Linker", inputs:{{.*}}, output: "a.out"

//
// Test one or more gpu architecture up to the assemble phase in host-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-host-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=HASM %s
// HASM: # "powerpc64le-ibm-linux-gnu" - "clang",{{.*}} output: "cuda-bindings.s"

//
// Test single gpu architecture with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN %s
// DBIN: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.o"

//
// Test single gpu architecture up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM %s
// DASM: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"

//
// Test two gpu architectures with complete compilation in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only 2>&1 \
// RUN: | FileCheck -check-prefix=DBIN2 %s
// DBIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN2-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.o"
// DBIN2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output:
// DBIN2-NOT: cuda-bindings-device-cuda-nvptx64
// DBIN2: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.o"

//
// Test two gpu architectures up to the assemble phase in device-only
// compilation mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -ccc-print-bindings \
// RUN:        --cuda-gpu-arch=sm_30 --cuda-gpu-arch=sm_35 %s --cuda-device-only -S 2>&1 \
// RUN: | FileCheck -check-prefix=DASM2 %s
// DASM2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_30.s"
// DASM2: # "nvptx64-nvidia-cuda" - "clang",{{.*}} output: "cuda-bindings-cuda-nvptx64-nvidia-cuda-sm_35.s"

//
// Ensure we output the user's specified name in device-only mode.
//
// RUN: %clang -target powerpc64le-ibm-linux-gnu -### \
// RUN:        --cuda-gpu-arch=sm_52 --cuda-device-only -c -o foo.o %s 2>&1 \
// RUN: | FileCheck -check-prefix=D_ONLY %s
// RUN: %clang -target powerpc64le-ibm-linux-gnu -### --offload-new-driver \
// RUN:        --cuda-gpu-arch=sm_52 --cuda-device-only -c -o foo.o %s 2>&1 \
// RUN: | FileCheck -check-prefix=D_ONLY %s
// D_ONLY: "foo.o"

//
// Check to make sure we can generate multiple outputs for device-only
// compilation and fail with '-o'.
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu --offload-new-driver -ccc-print-bindings \
// RUN:        --offload-arch=sm_70 --offload-arch=sm_52 --offload-device-only -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=MULTI-D-ONLY %s
//      MULTI-D-ONLY: # "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX_70:.+]]"
// MULTI-D-ONLY-NEXT: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_70]]"], output: "[[CUBIN_70:.+]]"
// MULTI-D-ONLY-NEXT: # "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]"], output: "[[PTX_52:.+]]"
// MULTI-D-ONLY-NEXT: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_52]]"], output: "[[CUBIN_52:.+]]"
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu --offload-new-driver -ccc-print-bindings \
// RUN:        --offload-arch=sm_70 --offload-arch=sm_52 --offload-device-only -c -o %t %s 2>&1 \
// RUN: | FileCheck -check-prefix=MULTI-D-ONLY-O %s
// MULTI-D-ONLY-O: error: cannot specify -o when generating multiple output files

//
// Check to ensure that we can use '-fsyntax-only' for CUDA output with the new
// driver.
// 
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu --offload-new-driver \
// RUN:        -fsyntax-only --offload-arch=sm_70 --offload-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=SYNTAX-ONLY %s
// SYNTAX-ONLY: "-cc1" "-triple" "nvptx64-nvidia-cuda"{{.*}}"-fsyntax-only"
// SYNTAX-ONLY: "-cc1" "-triple" "nvptx64-nvidia-cuda"{{.*}}"-fsyntax-only"
// SYNTAX-ONLY: "-cc1" "-triple" "powerpc64le-ibm-linux-gnu"{{.*}}"-fsyntax-only"

//
// Check to ensure that we can use '-save-temps' when operating in RDC-mode.
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu -save-temps --offload-new-driver \
// RUN:        -fgpu-rdc --offload-arch=sm_70 --offload-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=SAVE-TEMPS %s
// SAVE-TEMPS: "-cc1" "-triple" "nvptx64-nvidia-cuda"{{.*}}"-target-cpu" "sm_52"
// SAVE-TEMPS: "-cc1" "-triple" "nvptx64-nvidia-cuda"{{.*}}"-target-cpu" "sm_70"
// SAVE-TEMPS: "-cc1" "-triple" "powerpc64le-ibm-linux-gnu"

//
// Check to ensure that we cannot use '-foffload' when not operating in RDC-mode.
//
// RUN: %clang -### -target powerpc64le-ibm-linux-gnu -fno-gpu-rdc --offload-new-driver \
// RUN:        -foffload-lto --offload-arch=sm_70 --offload-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix=LTO-NO-RDC %s
// LTO-NO-RDC: error: unsupported option '-foffload-lto' for language mode '-fno-gpu-rdc'
