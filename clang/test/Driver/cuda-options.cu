
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case. Compile device-side to PTX assembly and make sure
// we use it on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -c -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Typical compilation + link case.
// RUN: %clang -### -target x86_64-linux-gnu -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-host-only disables device-side compilation, but doesn't
// disable host-side compilation/linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Verify that --cuda-device-only disables host-side compilation and linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Check that the last of --cuda-compile-host-device, --cuda-host-only, and
// --cuda-device-only wins.

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:    --cuda-host-only -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-host-only -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-host-only \
// RUN:    -nogpulib -nogpuinc --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    -nogpulib -nogpuinc --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-host-only \
// RUN:   -nogpulib -nogpuinc --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-gpu-arch option passes the correct GPU architecture to
// device compilation.
// RUN: %clang -### -nogpulib -nogpuinc --target=x86_64-linux-gnu --cuda-gpu-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE-SM52 -check-prefix HOST \
// RUN:    -check-prefix INCLUDES-DEVICE -check-prefix NOLINK %s

// Verify that there is one device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### --target=x86_64-linux-gnu \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes DEVICE,DEVICE-NOSAVE,DEVICE2 \
// RUN:             -check-prefixes DEVICE-SM52,DEVICE2-SM60 \
// RUN:             -check-prefixes INCLUDES-DEVICE,INCLUDES-DEVICE2 \
// RUN:             -check-prefixes HOST,HOST-NOSAVE,NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -save-temps is used.
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc -save-temps -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-SAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-SAVE -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -fno-integrated-as is used.
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc -fno-integrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-NOSAVE \
// RUN:    -check-prefix HOST-AS -check-prefix NOLINK %s

// Verify that --[no-]cuda-gpu-arch arguments are handled correctly.
// a) --no-cuda-gpu-arch=X negates preceding --cuda-gpu-arch=X
// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_70 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-gpu-arch=sm_70 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM52,NOARCH-SM60,NOARCH-SM70 %s

// b) --no-cuda-gpu-arch=X negates more than one preceding --cuda-gpu-arch=X
// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_70 --cuda-gpu-arch=sm_70 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-gpu-arch=sm_70 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM52,NOARCH-SM60,NOARCH-SM70 %s

// c) if --no-cuda-gpu-arch=X negates all preceding --cuda-gpu-arch=X
//    we default to sm_52 -- same as if no --cuda-gpu-arch were passed.
// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_70 --cuda-gpu-arch=sm_60 \
// RUN:   --no-cuda-gpu-arch=sm_70 --no-cuda-gpu-arch=sm_60 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM52,NOARCH-SM60,NOARCH-SM70 %s

// d) --no-cuda-gpu-arch=X is a no-op if there's no preceding --cuda-gpu-arch=X
// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52\
// RUN:   --no-cuda-gpu-arch=sm_70 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM52,ARCH-SM60,NOARCH-SM70 %s

// e) --no-cuda-gpu-arch=X does not affect following --cuda-gpu-arch=X
// RUN: %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --no-cuda-gpu-arch=sm_70 --no-cuda-gpu-arch=sm_52 \
// RUN:   --cuda-gpu-arch=sm_70 --cuda-gpu-arch=sm_52 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes ARCH-SM52,NOARCH-SM60,ARCH-SM70 %s

// f) --no-cuda-gpu-arch=all negates all preceding --cuda-gpu-arch=X
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-gpu-arch=all \
// RUN:   --cuda-gpu-arch=sm_70 \
// RUN:   -c -nogpulib -nogpuinc %s 2>&1 \
// RUN: | FileCheck -check-prefixes NOARCH-SM52,NOARCH-SM60,ARCH-SM70 %s

// g) There's no --cuda-gpu-arch=all
// RUN: not %clang -### --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   -nogpulib -nogpuinc --cuda-gpu-arch=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefix ARCHALLERROR %s


// Verify that --[no-]cuda-include-ptx arguments are handled correctly.
// a) by default we're including PTX for all GPUs.
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM60,PTX-SM52 %s

// b) --no-cuda-include-ptx=all disables PTX inclusion for all GPUs
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,NOPTX-SM60,NOPTX-SM52 %s

// c) --no-cuda-include-ptx=sm_XX disables PTX inclusion for that GPU only.
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-include-ptx=sm_60 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,NOPTX-SM60,PTX-SM52 %s
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-include-ptx=sm_52 \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM60,NOPTX-SM52 %s

// d) --cuda-include-ptx=all overrides preceding --no-cuda-include-ptx=all
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-include-ptx=all --cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM60,PTX-SM52 %s

// e) --cuda-include-ptx=all overrides preceding --no-cuda-include-ptx=sm_XX
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc \
// RUN:   --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_52 \
// RUN:   --no-cuda-include-ptx=sm_52 --cuda-include-ptx=all \
// RUN:   -c %s 2>&1 \
// RUN: | FileCheck -check-prefixes FATBIN-COMMON,PTX-SM60,PTX-SM52 %s

// Verify -flto=thin -fwhole-program-vtables handling. This should result in
// both options being passed to the host compilation, with neither passed to
// the device compilation.
// RUN: %clang -### --target=x86_64-linux-gnu -nogpulib -nogpuinc -c -flto=thin -fwhole-program-vtables %s 2>&1 \
// RUN: | FileCheck -check-prefixes DEVICE,DEVICE-NOSAVE,HOST,INCLUDES-DEVICE,NOLINK,THINLTOWPD %s
// THINLTOWPD-NOT: error: invalid argument '-fwhole-program-vtables' only allowed with '-flto'

// ARCH-SM52: "-cc1"{{.*}}"-target-cpu" "sm_52"
// NOARCH-SM52-NOT: "-cc1"{{.*}}"-target-cpu" "sm_52"
// ARCH-SM60: "-cc1"{{.*}}"-target-cpu" "sm_60"
// NOARCH-SM60-NOT: "-cc1"{{.*}}"-target-cpu" "sm_60"
// ARCH-SM70: "-cc1"{{.*}}"-target-cpu" "sm_70"
// NOARCH-SM70-NOT: "-cc1"{{.*}}"-target-cpu" "sm_70"
// ARCHALLERROR: error: unsupported CUDA gpu architecture: all

// Match device-side preprocessor and compiler phases with -save-temps.
// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda"

// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly.
// DEVICE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-NOSAVE-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// THINLTOWPD-NOT: "-flto=thin"
// DEVICE-SAME: "-fcuda-is-device"
// DEVICE-SM52-SAME: "-target-cpu" "sm_52"
// THINLTOWPD-NOT: "-fwhole-program-vtables"
// DEVICE-SAME: "-o" "[[PTXFILE:[^"]*]]"
// DEVICE-NOSAVE-SAME: "-x" "cuda"
// DEVICE-SAVE-SAME: "-x" "ir"

// Match the call to ptxas (which assembles PTX to SASS).
// DEVICE:ptxas
// DEVICE-SM52-DAG: "--gpu-name" "sm_52"
// DEVICE-DAG: "--output-file" "[[CUBINFILE:[^"]*]]"
// DEVICE-DAG: "[[PTXFILE]]"

// Match another device-side compilation.
// DEVICE2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE2-SAME: "-aux-triple" "x86_64-unknown-linux-gnu"
// DEVICE2-SAME: "-fcuda-is-device"
// DEVICE2-SM60-SAME: "-target-cpu" "sm_60"
// DEVICE2-SAME: "-o" "[[PTXFILE2:[^"]*]]"
// DEVICE2-SAME: "-x" "cuda"

// Match another call to ptxas.
// DEVICE2: ptxas
// DEVICE2-SM60-DAG: "--gpu-name" "sm_60"
// DEVICE2-DAG: "--output-file" "[[CUBINFILE2:[^"]*]]"
// DEVICE2-DAG: "[[PTXFILE2]]"

// Match no device-side compilation.
// NODEVICE-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// NODEVICE-NOT: "-fcuda-is-device"

// INCLUDES-DEVICE:fatbinary
// INCLUDES-DEVICE-DAG: "--create" "[[FATBINARY:[^"]*]]"
// INCLUDES-DEVICE-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE]]"
// INCLUDES-DEVICE-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE]]"
// INCLUDES-DEVICE2-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE2]]"
// INCLUDES-DEVICE2-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE2]]"

// Match host-side preprocessor job with -save-temps.
// HOST-SAVE: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// HOST-SAVE-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAVE-NOT: "-fcuda-is-device"
// HOST-SAVE-SAME: "-x" "cuda"

// Match host-side compilation.
// HOST: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// HOST-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// THINLTOWPD-SAME: "-flto=thin"
// HOST-NOT: "-fcuda-is-device"
// There is only one GPU binary after combining it with fatbinary!
// INCLUDES-DEVICE2-NOT: "-fcuda-include-gpubinary"
// INCLUDES-DEVICE-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"
// There is only one GPU binary after combining it with fatbinary.
// INCLUDES-DEVICE2-NOT: "-fcuda-include-gpubinary"
// THINLTOWPD-SAME: "-fwhole-program-vtables"
// HOST-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// HOST-NOSAVE-SAME: "-x" "cuda"
// HOST-SAVE-SAME: "-x" "cuda-cpp-output"

// Match external assembler that uses compilation output.
// HOST-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// NOINCLUDES-DEVICE-NOT: "-fcuda-include-gpubinary"

// Match no host compilation.
// NOHOST-NOT: "-cc1" "-triple"
// NOHOST-NOT: "-x" "cuda"

// Match linker.
// LINK: "{{.*}}{{ld|link}}{{(.exe)?}}"
// LINK-SAME: "[[HOSTOUTPUT]]"

// Match no linker.
// NOLINK-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"

// FATBIN-COMMON:fatbinary
// FATBIN-COMMON: "--create" "[[FATBINARY:[^"]*]]"
// FATBIN-COMMON: "--image=profile=sm_52,file=
// PTX-SM52: "--image=profile=compute_52,file=
// NOPTX-SM52-NOT: "--image=profile=compute_52,file=
// FATBIN-COMMON: "--image=profile=sm_60,file=
// PTX-SM60: "--image=profile=compute_60,file=
// NOPTX-SM60-NOT: "--image=profile=compute_60,file=
