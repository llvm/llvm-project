
#include <hip/hip_runtime.h>
// RUN: %clang -### -target x86_64-linux-gnu -foffload-via-llvm -ccc-print-bindings \
// RUN:        --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS %s

//      BINDINGS: "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[HOST_BC:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]", "[[HOST_BC]]"], output: "[[PTX_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_35]]"], output: "[[CUBIN_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]", "[[HOST_BC]]"], output: "[[PTX_SM_70:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_70:.+]]"], output: "[[CUBIN_SM_70:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Packager", inputs: ["[[CUBIN_SM_35]]", "[[CUBIN_SM_70]]"], output: "[[BINARY:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[HOST_BC]]", "[[BINARY]]"], output: "[[HOST_OBJ:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[HOST_OBJ]]"], output: "a.out"

// RUN: %clang -### -target x86_64-linux-gnu -foffload-via-llvm -ccc-print-bindings \
// RUN:        --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS-DEVICE %s

// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX:.+]]"
// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX]]"], output: "[[CUBIN:.+]]"

// RUN: %clang -### -target x86_64-linux-gnu -ccc-print-bindings --offload-link -foffload-via-llvm %s 2>&1 | FileCheck -check-prefix DEVICE-LINK %s

// DEVICE-LINK: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[INPUT:.+]]"], output: "a.out"
