#include "hip/hip_runtime.h"
// RUN: cat %s | clang-repl --cuda | FileCheck %s

extern "C" int printf(const char*, ...);

__global__ void test_func() {}

test_func<<<1,1>>>();
printf("CUDA Error: %d", hipGetLastError());
// CHECK: CUDA Error: 0

%quit
