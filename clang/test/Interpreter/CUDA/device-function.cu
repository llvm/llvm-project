#include "hip/hip_runtime.h"
// Tests __device__ function calls
// RUN: cat %s | clang-repl --cuda | FileCheck %s

extern "C" int printf(const char*, ...);

__device__ inline void test_device(int* value) { *value = 42; }
__global__ void test_kernel(int* value) { test_device(value); }

int var;
int* devptr = nullptr;
printf("hipMalloc: %d\n", hipMalloc((void **) &devptr, sizeof(int)));
// CHECK: hipMalloc: 0

test_kernel<<<1,1>>>(devptr);
printf("CUDA Error: %d\n", hipGetLastError());
// CHECK-NEXT: CUDA Error: 0

printf("hipMemcpy: %d\n", hipMemcpy(&var, devptr, sizeof(int), hipMemcpyDeviceToHost));
// CHECK-NEXT: hipMemcpy: 0

printf("Value: %d\n", var);
// CHECK-NEXT: Value: 42

%quit
