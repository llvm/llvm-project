#include "hip/hip_runtime.h"
// Tests hipMemcpy and writes from kernel
// RUN: cat %s | clang-repl --cuda | FileCheck %s

extern "C" int printf(const char*, ...);

__global__ void test_func(int* value) { *value = 42; }

int var;
int* devptr = nullptr;
printf("hipMalloc: %d\n", hipMalloc((void **) &devptr, sizeof(int)));
// CHECK: hipMalloc: 0

test_func<<<1,1>>>(devptr);
printf("CUDA Error: %d\n", hipGetLastError());
// CHECK-NEXT: CUDA Error: 0

printf("hipMemcpy: %d\n", hipMemcpy(&var, devptr, sizeof(int), hipMemcpyDeviceToHost));
// CHECK-NEXT: hipMemcpy: 0

printf("Value: %d\n", var);
// CHECK-NEXT: Value: 42

%quit
