#include "hip/hip_runtime.h"
// Tests device function templates
// RUN: cat %s | clang-repl --cuda | FileCheck %s

extern "C" int printf(const char*, ...);

template <typename T> __device__ inline T sum(T a, T b) { return a + b; }
__global__ void test_kernel(int* value) { *value = sum(40, 2); }

int var;
int* devptr = nullptr;
printf("hipMalloc: %d\n", hipMalloc((void **) &devptr, sizeof(int)));
// CHECK: hipMalloc: 0

test_kernel<<<1,1>>>(devptr);
printf("CUDA Error: %d\n", hipGetLastError());
// CHECK-NEXT: CUDA Error: 0

printf("hipMemcpy: %d\n", hipMemcpy(&var, devptr, sizeof(int), hipMemcpyDeviceToHost));
// CHECK-NEXT: hipMemcpy: 0

printf("Value: %d\n", var);
// CHECK-NEXT: Value: 42

%quit
