
#include <hip/hip_runtime.h>
// Check that types, widths, __CLANG_ATOMIC* macros, etc. match on the host and
// device sides of CUDA compilations.  Note that we filter out long double, as
// this is intentionally different on host and device.
//
// Also ignore __CLANG_ATOMIC_LLONG_LOCK_FREE on i386. The default host CPU for
// an i386 triple is typically at least an i586, which has cmpxchg8b (Clang
// feature, "cx8"). Therefore, __CLANG_ATOMIC_LLONG_LOCK_FREE is 2 on the host,
// but the value should be 1 for the device.
//
// FIXME: We really should make __GCC_HAVE_SYNC_COMPARE_AND_SWAP identical on
// host and device, but architecturally this is difficult at the moment.

// RUN: mkdir -p %t

// RUN: %clang --cuda-host-only -nocudainc -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE|_ATOMIC_LLONG_LOCK_FREE' > %t/i386-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE|_ATOMIC_LLONG_LOCK_FREE' > %t/i386-device-defines-filtered
// RUN: diff %t/i386-host-defines-filtered %t/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-device-defines-filtered
// RUN: diff %t/x86_64-host-defines-filtered %t/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/powerpc64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target powerpc64-unknown-linux-gnu -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/powerpc64-device-defines-filtered
// RUN: diff %t/powerpc64-host-defines-filtered %t/powerpc64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE|_ATOMIC_LLONG_LOCK_FREE' > %t/i386-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target i386-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE|_ATOMIC_LLONG_LOCK_FREE' > %t/i386-msvc-device-defines-filtered
// RUN: diff %t/i386-msvc-host-defines-filtered %t/i386-msvc-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib -target x86_64-windows-msvc -x cuda -E -dM -o - /dev/null \
// RUN:   | grep -E 'define __[^ ]*(TYPE|MAX|SIZEOF|WIDTH)|define __CLANG_ATOMIC' \
// RUN:   | grep -Ev '__LDBL|_LONG_DOUBLE' > %t/x86_64-msvc-device-defines-filtered
// RUN: diff %t/x86_64-msvc-host-defines-filtered %t/x86_64-msvc-device-defines-filtered
