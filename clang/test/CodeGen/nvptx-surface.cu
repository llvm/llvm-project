#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple nvptx-unknown-unknown -fcuda-is-device -O3 -o - %s -emit-llvm | FileCheck %s
// RUN: %clang_cc1 -triple nvptx64-unknown-unknown -fcuda-is-device -O3 -o - %s -emit-llvm | FileCheck %s
#include "../Headers/Inputs/include/hip/hip_runtime.h"

#include "__clang_cuda_texture_intrinsics.h"

__device__ void surfchar(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  char val;

  // CHECK: %0 = tail call i8 asm "suld.b.1d.b8.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i8 asm "suld.b.1d.b8.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i8 asm "suld.b.1d.b8.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i8 asm "suld.b.2d.b8.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i8 asm "suld.b.2d.b8.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i8 asm "suld.b.2d.b8.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i8 asm "suld.b.3d.b8.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i8 asm "suld.b.3d.b8.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i8 asm "suld.b.3d.b8.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i8 asm "suld.b.a1d.b8.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i8 asm "suld.b.a1d.b8.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i8 asm "suld.b.a1d.b8.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfsignedchar(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  signed char val;

  // CHECK: %0 = tail call i8 asm "suld.b.1d.b8.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i8 asm "suld.b.1d.b8.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i8 asm "suld.b.1d.b8.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i8 asm "suld.b.2d.b8.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i8 asm "suld.b.2d.b8.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i8 asm "suld.b.2d.b8.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i8 asm "suld.b.3d.b8.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i8 asm "suld.b.3d.b8.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i8 asm "suld.b.3d.b8.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i8 asm "suld.b.a1d.b8.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i8 asm "suld.b.a1d.b8.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i8 asm "suld.b.a1d.b8.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfchar1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  char1 val;

  // CHECK: %0 = tail call i8 asm "suld.b.1d.b8.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i8 asm "suld.b.1d.b8.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i8 asm "suld.b.1d.b8.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i8 asm "suld.b.2d.b8.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i8 asm "suld.b.2d.b8.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i8 asm "suld.b.2d.b8.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i8 asm "suld.b.3d.b8.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i8 asm "suld.b.3d.b8.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i8 asm "suld.b.3d.b8.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i8 asm "suld.b.a1d.b8.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i8 asm "suld.b.a1d.b8.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i8 asm "suld.b.a1d.b8.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfunsignedchar(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  unsigned char val;

  // CHECK: %0 = tail call i8 asm "suld.b.1d.b8.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i8 asm "suld.b.1d.b8.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i8 asm "suld.b.1d.b8.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i8 asm "suld.b.2d.b8.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i8 asm "suld.b.2d.b8.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i8 asm "suld.b.2d.b8.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i8 asm "suld.b.3d.b8.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i8 asm "suld.b.3d.b8.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i8 asm "suld.b.3d.b8.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i8 asm "suld.b.a1d.b8.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i8 asm "suld.b.a1d.b8.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i8 asm "suld.b.a1d.b8.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuchar1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uchar1 val;

  // CHECK: %0 = tail call i8 asm "suld.b.1d.b8.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i8 asm "suld.b.1d.b8.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i8 asm "suld.b.1d.b8.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b8.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i8 asm "suld.b.2d.b8.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i8 asm "suld.b.2d.b8.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i8 asm "suld.b.2d.b8.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b8.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i8 asm "suld.b.3d.b8.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i8 asm "suld.b.3d.b8.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i8 asm "suld.b.3d.b8.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b8.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i8 asm "suld.b.a1d.b8.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i8 asm "suld.b.a1d.b8.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i8 asm "suld.b.a1d.b8.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b8.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i8 asm "suld.b.a2d.b8.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i8 asm "suld.b.a2d.b8.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i8 asm "suld.b.a2d.b8.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b8.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfshort(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  short val;

  // CHECK: %0 = tail call i16 asm "suld.b.1d.b16.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i16 asm "suld.b.1d.b16.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i16 asm "suld.b.1d.b16.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i16 asm "suld.b.2d.b16.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i16 asm "suld.b.2d.b16.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i16 asm "suld.b.2d.b16.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i16 asm "suld.b.3d.b16.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i16 asm "suld.b.3d.b16.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i16 asm "suld.b.3d.b16.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i16 asm "suld.b.a1d.b16.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i16 asm "suld.b.a1d.b16.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i16 asm "suld.b.a1d.b16.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfshort1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  short1 val;

  // CHECK: %0 = tail call i16 asm "suld.b.1d.b16.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i16 asm "suld.b.1d.b16.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i16 asm "suld.b.1d.b16.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i16 asm "suld.b.2d.b16.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i16 asm "suld.b.2d.b16.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i16 asm "suld.b.2d.b16.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i16 asm "suld.b.3d.b16.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i16 asm "suld.b.3d.b16.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i16 asm "suld.b.3d.b16.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i16 asm "suld.b.a1d.b16.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i16 asm "suld.b.a1d.b16.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i16 asm "suld.b.a1d.b16.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfunsignedshort(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  unsigned short val;

  // CHECK: %0 = tail call i16 asm "suld.b.1d.b16.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i16 asm "suld.b.1d.b16.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i16 asm "suld.b.1d.b16.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i16 asm "suld.b.2d.b16.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i16 asm "suld.b.2d.b16.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i16 asm "suld.b.2d.b16.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i16 asm "suld.b.3d.b16.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i16 asm "suld.b.3d.b16.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i16 asm "suld.b.3d.b16.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i16 asm "suld.b.a1d.b16.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i16 asm "suld.b.a1d.b16.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i16 asm "suld.b.a1d.b16.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfushort1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  ushort1 val;

  // CHECK: %0 = tail call i16 asm "suld.b.1d.b16.zero {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.zero [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i16 asm "suld.b.1d.b16.clamp {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.clamp [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i16 asm "suld.b.1d.b16.trap {$0}, [$1, {$2}];", "=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b16.trap [$0, {$1}], {$2};", "l,r,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i16 asm "suld.b.2d.b16.zero {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.zero [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i16 asm "suld.b.2d.b16.clamp {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.clamp [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i16 asm "suld.b.2d.b16.trap {$0}, [$1, {$2, $3}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b16.trap [$0, {$1, $2}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i16 asm "suld.b.3d.b16.zero {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i16 asm "suld.b.3d.b16.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i16 asm "suld.b.3d.b16.trap {$0}, [$1, {$2, $3, $4, $4}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b16.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i16 asm "suld.b.a1d.b16.zero {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.zero [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i16 asm "suld.b.a1d.b16.clamp {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.clamp [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i16 asm "suld.b.a1d.b16.trap {$0}, [$1, {$3, $2}];", "=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b16.trap [$0, {$2, $1}], {$3};", "l,r,r,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i16 asm "suld.b.a2d.b16.zero {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i16 asm "suld.b.a2d.b16.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i16 asm "suld.b.a2d.b16.trap {$0}, [$1, {$4, $2, $3, $3}];", "=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b16.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfint(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  int val;

  // CHECK: %0 = tail call i32 asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i32 asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i32 asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i32 asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i32 asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i32 asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i32 asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i32 asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i32 asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i32 asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i32 asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i32 asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfint1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  int1 val;

  // CHECK: %0 = tail call i32 asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i32 asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i32 asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i32 asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i32 asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i32 asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i32 asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i32 asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i32 asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i32 asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i32 asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i32 asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfunsignedint(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  unsigned int val;

  // CHECK: %0 = tail call i32 asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i32 asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i32 asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i32 asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i32 asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i32 asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i32 asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i32 asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i32 asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i32 asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i32 asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i32 asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuint1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uint1 val;

  // CHECK: %0 = tail call i32 asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i32 asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i32 asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i32 asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i32 asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i32 asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i32 asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i32 asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i32 asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i32 asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i32 asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i32 asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i32 asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i32 asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i32 asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surflonglong(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  long long val;

  // CHECK: %0 = tail call i64 asm "suld.b.1d.b64.zero {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.zero [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i64 asm "suld.b.1d.b64.clamp {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.clamp [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i64 asm "suld.b.1d.b64.trap {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.trap [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i64 asm "suld.b.2d.b64.zero {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.zero [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i64 asm "suld.b.2d.b64.clamp {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.clamp [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i64 asm "suld.b.2d.b64.trap {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.trap [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i64 asm "suld.b.3d.b64.zero {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i64 asm "suld.b.3d.b64.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i64 asm "suld.b.3d.b64.trap {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i64 asm "suld.b.a1d.b64.zero {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.zero [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i64 asm "suld.b.a1d.b64.clamp {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.clamp [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i64 asm "suld.b.a1d.b64.trap {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.trap [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surflonglong1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  longlong1 val;

  // CHECK: %0 = tail call i64 asm "suld.b.1d.b64.zero {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.zero [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i64 asm "suld.b.1d.b64.clamp {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.clamp [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i64 asm "suld.b.1d.b64.trap {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.trap [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i64 asm "suld.b.2d.b64.zero {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.zero [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i64 asm "suld.b.2d.b64.clamp {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.clamp [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i64 asm "suld.b.2d.b64.trap {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.trap [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i64 asm "suld.b.3d.b64.zero {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i64 asm "suld.b.3d.b64.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i64 asm "suld.b.3d.b64.trap {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i64 asm "suld.b.a1d.b64.zero {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.zero [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i64 asm "suld.b.a1d.b64.clamp {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.clamp [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i64 asm "suld.b.a1d.b64.trap {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.trap [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfunsignedlonglong(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  unsigned long long val;

  // CHECK: %0 = tail call i64 asm "suld.b.1d.b64.zero {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.zero [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i64 asm "suld.b.1d.b64.clamp {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.clamp [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i64 asm "suld.b.1d.b64.trap {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.trap [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i64 asm "suld.b.2d.b64.zero {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.zero [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i64 asm "suld.b.2d.b64.clamp {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.clamp [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i64 asm "suld.b.2d.b64.trap {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.trap [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i64 asm "suld.b.3d.b64.zero {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i64 asm "suld.b.3d.b64.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i64 asm "suld.b.3d.b64.trap {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i64 asm "suld.b.a1d.b64.zero {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.zero [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i64 asm "suld.b.a1d.b64.clamp {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.clamp [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i64 asm "suld.b.a1d.b64.trap {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.trap [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfulonglong1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  ulonglong1 val;

  // CHECK: %0 = tail call i64 asm "suld.b.1d.b64.zero {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.zero [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call i64 asm "suld.b.1d.b64.clamp {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.clamp [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call i64 asm "suld.b.1d.b64.trap {$0}, [$1, {$2}];", "=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b64.trap [$0, {$1}], {$2};", "l,r,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call i64 asm "suld.b.2d.b64.zero {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.zero [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call i64 asm "suld.b.2d.b64.clamp {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.clamp [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call i64 asm "suld.b.2d.b64.trap {$0}, [$1, {$2, $3}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b64.trap [$0, {$1, $2}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call i64 asm "suld.b.3d.b64.zero {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call i64 asm "suld.b.3d.b64.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call i64 asm "suld.b.3d.b64.trap {$0}, [$1, {$2, $3, $4, $4}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b64.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call i64 asm "suld.b.a1d.b64.zero {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.zero [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call i64 asm "suld.b.a1d.b64.clamp {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.clamp [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call i64 asm "suld.b.a1d.b64.trap {$0}, [$1, {$3, $2}];", "=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b64.trap [$0, {$2, $1}], {$3};", "l,r,r,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call i64 asm "suld.b.a2d.b64.zero {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call i64 asm "suld.b.a2d.b64.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call i64 asm "suld.b.a2d.b64.trap {$0}, [$1, {$4, $2, $3, $3}];", "=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b64.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surffloat(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  float val;

  // CHECK: %0 = tail call contract float asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call contract float asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call contract float asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call contract float asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call contract float asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call contract float asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call contract float asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call contract float asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call contract float asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call contract float asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call contract float asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call contract float asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surffloat1(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  float1 val;

  // CHECK: %0 = tail call contract float asm "suld.b.1d.b32.zero {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.zero [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call contract float asm "suld.b.1d.b32.clamp {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.clamp [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call contract float asm "suld.b.1d.b32.trap {$0}, [$1, {$2}];", "=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.b32.trap [$0, {$1}], {$2};", "l,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call contract float asm "suld.b.2d.b32.zero {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.zero [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call contract float asm "suld.b.2d.b32.clamp {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.clamp [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call contract float asm "suld.b.2d.b32.trap {$0}, [$1, {$2, $3}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.b32.trap [$0, {$1, $2}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call contract float asm "suld.b.3d.b32.zero {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.zero [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call contract float asm "suld.b.3d.b32.clamp {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.clamp [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call contract float asm "suld.b.3d.b32.trap {$0}, [$1, {$2, $3, $4, $4}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.b32.trap [$0, {$1, $2, $3, $3}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call contract float asm "suld.b.a1d.b32.zero {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.zero [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call contract float asm "suld.b.a1d.b32.clamp {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.clamp [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call contract float asm "suld.b.a1d.b32.trap {$0}, [$1, {$3, $2}];", "=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.b32.trap [$0, {$2, $1}], {$3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call contract float asm "suld.b.a2d.b32.zero {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.zero [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call contract float asm "suld.b.a2d.b32.clamp {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.clamp [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call contract float asm "suld.b.a2d.b32.trap {$0}, [$1, {$4, $2, $3, $3}];", "=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.b32.trap [$0, {$3, $1, $2, $2}], {$4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfchar2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  char2 val;

  // CHECK: %0 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.zero {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.zero [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.clamp {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.clamp [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.trap {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.trap [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.zero {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.clamp {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.trap {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.zero {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.clamp {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.trap {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuchar2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uchar2 val;

  // CHECK: %0 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.zero {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.zero [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.clamp {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.clamp [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i8, i8 } asm "suld.b.1d.v2.b8.trap {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b8.trap [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.zero {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.clamp {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i8, i8 } asm "suld.b.2d.v2.b8.trap {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b8.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i8, i8 } asm "suld.b.3d.v2.b8.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b8.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.zero {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.clamp {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i8, i8 } asm "suld.b.a1d.v2.b8.trap {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b8.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i8, i8 } asm "suld.b.a2d.v2.b8.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfshort2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  short2 val;

  // CHECK: %0 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.zero {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.zero [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.clamp {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.clamp [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.trap {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.trap [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.zero {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.clamp {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.trap {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.zero {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.clamp {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.trap {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}
__device__ void surfushort2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  ushort2 val;

  // CHECK: %0 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.zero {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.zero [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.clamp {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.clamp [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i16, i16 } asm "suld.b.1d.v2.b16.trap {$0, $1}, [$2, {$3}];", "=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b16.trap [$0, {$1}], {$2, $3};", "l,r,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.zero {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.clamp {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i16, i16 } asm "suld.b.2d.v2.b16.trap {$0, $1}, [$2, {$3, $4}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b16.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i16, i16 } asm "suld.b.3d.v2.b16.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b16.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.zero {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.clamp {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i16, i16 } asm "suld.b.a1d.v2.b16.trap {$0, $1}, [$2, {$4, $3}];", "=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b16.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i16, i16 } asm "suld.b.a2d.v2.b16.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfint2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  int2 val;

  // CHECK: %0 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.zero {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.zero [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.clamp {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.clamp [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.trap {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.trap [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.zero {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.clamp {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.trap {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.zero {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.clamp {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.trap {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuint2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uint2 val;

  // CHECK: %0 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.zero {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.zero [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.clamp {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.clamp [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i32, i32 } asm "suld.b.1d.v2.b32.trap {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.trap [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.zero {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.clamp {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i32, i32 } asm "suld.b.2d.v2.b32.trap {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i32, i32 } asm "suld.b.3d.v2.b32.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.zero {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.clamp {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i32, i32 } asm "suld.b.a1d.v2.b32.trap {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i32, i32 } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surflonglong2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  longlong2 val;

  // CHECK: %0 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.zero {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.zero [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.clamp {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.clamp [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.trap {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.trap [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.zero {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.clamp {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.trap {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.zero {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.clamp {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.trap {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfulonglong2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  ulonglong2 val;

  // CHECK: %0 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.zero {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.zero [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.clamp {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.clamp [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i64, i64 } asm "suld.b.1d.v2.b64.trap {$0, $1}, [$2, {$3}];", "=l,=l,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b64.trap [$0, {$1}], {$2, $3};", "l,r,l,l"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.zero {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.clamp {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i64, i64 } asm "suld.b.2d.v2.b64.trap {$0, $1}, [$2, {$3, $4}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b64.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i64, i64 } asm "suld.b.3d.v2.b64.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b64.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.zero {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.clamp {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i64, i64 } asm "suld.b.a1d.v2.b64.trap {$0, $1}, [$2, {$4, $3}];", "=l,=l,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b64.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,l,l"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i64, i64 } asm "suld.b.a2d.v2.b64.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=l,=l,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b64.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,l,l"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surffloat2(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  float2 val;

  // CHECK: %0 = tail call contract { float, float } asm "suld.b.1d.v2.b32.zero {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.zero [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call contract { float, float } asm "suld.b.1d.v2.b32.clamp {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.clamp [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call contract { float, float } asm "suld.b.1d.v2.b32.trap {$0, $1}, [$2, {$3}];", "=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v2.b32.trap [$0, {$1}], {$2, $3};", "l,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call contract { float, float } asm "suld.b.2d.v2.b32.zero {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.zero [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call contract { float, float } asm "suld.b.2d.v2.b32.clamp {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.clamp [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call contract { float, float } asm "suld.b.2d.v2.b32.trap {$0, $1}, [$2, {$3, $4}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v2.b32.trap [$0, {$1, $2}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call contract { float, float } asm "suld.b.3d.v2.b32.zero {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call contract { float, float } asm "suld.b.3d.v2.b32.clamp {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call contract { float, float } asm "suld.b.3d.v2.b32.trap {$0, $1}, [$2, {$3, $4, $5, $5}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v2.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call contract { float, float } asm "suld.b.a1d.v2.b32.zero {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.zero [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call contract { float, float } asm "suld.b.a1d.v2.b32.clamp {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.clamp [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call contract { float, float } asm "suld.b.a1d.v2.b32.trap {$0, $1}, [$2, {$4, $3}];", "=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v2.b32.trap [$0, {$2, $1}], {$3, $4};", "l,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.zero {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.clamp {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call contract { float, float } asm "suld.b.a2d.v2.b32.trap {$0, $1}, [$2, {$5, $3, $4, $4}];", "=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v2.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfchar4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  char4 val;

  // CHECK: %0 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuchar4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uchar4 val;

  // CHECK: %0 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i8, i8, i8, i8 } asm "suld.b.1d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b8.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i8, i8, i8, i8 } asm "suld.b.2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b8.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i8, i8, i8, i8 } asm "suld.b.3d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b8.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i8, i8, i8, i8 } asm "suld.b.a1d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b8.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i8, i8, i8, i8 } asm "suld.b.a2d.v4.b8.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b8.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfshort4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  short4 val;

  // CHECK: %0 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfushort4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  ushort4 val;

  // CHECK: %0 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i16, i16, i16, i16 } asm "suld.b.1d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5}];", "=h,=h,=h,=h,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b16.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i16, i16, i16, i16 } asm "suld.b.2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b16.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i16, i16, i16, i16 } asm "suld.b.3d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b16.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i16, i16, i16, i16 } asm "suld.b.a1d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=h,=h,=h,=h,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b16.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i16, i16, i16, i16 } asm "suld.b.a2d.v4.b16.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=h,=h,=h,=h,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b16.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,h,h,h,h"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfint4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  int4 val;

  // CHECK: %0 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surfuint4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  uint4 val;

  // CHECK: %0 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call { i32, i32, i32, i32 } asm "suld.b.1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call { i32, i32, i32, i32 } asm "suld.b.2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call { i32, i32, i32, i32 } asm "suld.b.3d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call { i32, i32, i32, i32 } asm "suld.b.a1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call { i32, i32, i32, i32 } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}

__device__ void surffloat4(hipSurfaceObject_t surf, int x, int y, int z, int layer, int face, int layerface) {
  float4 val;

  // CHECK: %0 = tail call contract { float, float, float, float } asm "suld.b.1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.zero [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeZero);
  // CHECK: %1 = tail call contract { float, float, float, float } asm "suld.b.1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.clamp [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeClamp);
  // CHECK: %2 = tail call contract { float, float, float, float } asm "suld.b.1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5}];", "=r,=r,=r,=r,l,r"
  __nv_tex_surf_handler("__isurf1Dread", &val, surf, x, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.1d.v4.b32.trap [$0, {$1}], {$2, $3, $4, $5};", "l,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1Dwrite_v2", &val, surf, x, hipBoundaryModeTrap);

  // CHECK: %3 = tail call contract { float, float, float, float } asm "suld.b.2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.zero [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeZero);
  // CHECK: %4 = tail call contract { float, float, float, float } asm "suld.b.2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.clamp [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeClamp);
  // CHECK: %5 = tail call contract { float, float, float, float } asm "suld.b.2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf2Dread", &val, surf, x, y, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.2d.v4.b32.trap [$0, {$1, $2}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2Dwrite_v2", &val, surf, x, y, hipBoundaryModeTrap);

  // CHECK: %6 = tail call contract { float, float, float, float } asm "suld.b.3d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.zero [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeZero);
  // CHECK: %7 = tail call contract { float, float, float, float } asm "suld.b.3d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.clamp [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeClamp);
  // CHECK: %8 = tail call contract { float, float, float, float } asm "suld.b.3d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$5, $6, $7, $7}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf3Dread", &val, surf, x, y, z, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.3d.v4.b32.trap [$0, {$1, $2, $3, $3}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf3Dwrite_v2", &val, surf, x, y, z, hipBoundaryModeTrap);

  // CHECK: %9 = tail call contract { float, float, float, float } asm "suld.b.a1d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.zero [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeZero);
  // CHECK: %10 = tail call contract { float, float, float, float } asm "suld.b.a1d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.clamp [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeClamp);
  // CHECK: %11 = tail call contract { float, float, float, float } asm "suld.b.a1d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$6, $5}];", "=r,=r,=r,=r,l,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredread", &val, surf, x, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a1d.v4.b32.trap [$0, {$2, $1}], {$3, $4, $5, $6};", "l,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf1DLayeredwrite_v2", &val, surf, x, layer, hipBoundaryModeTrap);

  // CHECK: %12 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeZero);
  // CHECK: %13 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeClamp);
  // CHECK: %14 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredread", &val, surf, x, y, layer, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurf2DLayeredwrite_v2", &val, surf, x, y, layer, hipBoundaryModeTrap);

  // CHECK: %15 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeZero);
  // CHECK: %16 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeClamp);
  // CHECK: %17 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapread", &val, surf, x, y, face, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapwrite_v2", &val, surf, x, y, face, hipBoundaryModeTrap);

  // CHECK: %18 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.zero {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.zero [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeZero);
  // CHECK: %19 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.clamp {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.clamp [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeClamp);
  // CHECK: %20 = tail call contract { float, float, float, float } asm "suld.b.a2d.v4.b32.trap {$0, $1, $2, $3}, [$4, {$7, $5, $6, $6}];", "=r,=r,=r,=r,l,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredread", &val, surf, x, y, layerface, hipBoundaryModeTrap);
  // CHECK: tail call void asm sideeffect "sust.b.a2d.v4.b32.trap [$0, {$3, $1, $2, $2}], {$4, $5, $6, $7};", "l,r,r,r,r,r,r,r"
  __nv_tex_surf_handler("__isurfCubemapLayeredwrite_v2", &val, surf, x, y, layerface, hipBoundaryModeTrap);
}
