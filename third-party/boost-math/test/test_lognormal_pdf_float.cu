#include "hip/hip_runtime.h"
//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#define BOOST_MATH_OVERFLOW_ERROR_POLICY ignore_error

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/math/distributions/lognormal.hpp>
#include <boost/math/special_functions/relative_difference.hpp>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_real_distribution.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef float float_type;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in1, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = pdf(boost::math::lognormal_distribution<float_type>(), in1[i]);
    }
}

/**
 * Host main routine
 */
int main(void)
{
  try{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector1(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    boost::random::mt19937 gen;
    boost::random::uniform_real_distribution<float_type> dist;
    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        input_vector1[i] = dist(gen);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;
    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector1.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();
    std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch lognormal distribution kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
       results.push_back(pdf(boost::math::lognormal_distribution<float_type>(), input_vector1[i]));
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (boost::math::epsilon_difference(output_vector[i], results[i]) > 100.0)
        {
            std::cerr << "Result verification failed at element " << i << "!" << std::endl;
            std::cerr << "Error rate was: " << boost::math::epsilon_difference(output_vector[i], results[i]) << "eps" << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";
  }
  catch(const std::exception& e)
  {
    std::cerr << "Stopped with exception: " << e.what() << std::endl;
  }
  return 0;
}
