#include "hip/hip_runtime.h"
//  Copyright John Maddock 2016.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/math/special_functions/round.hpp>
#include <boost/math/special_functions/relative_difference.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef float float_type;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = boost::math::round(in[i]) + boost::math::iround(in[i]) + boost::math::lround(in[i]) + boost::math::llround(in[i]);
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector addition of " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> h_A(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> h_C(numElements);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float_type)RAND_MAX;
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;
    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(h_A.get(), h_C.get(), numElements);
    hipDeviceSynchronize();
    std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
       results.push_back(4 * boost::math::round(h_A[i]));
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (boost::math::epsilon_difference(h_C[i], results[i]) > 10)
        {
            std::cerr << "Result verification failed at element " << i << "!" << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";

    return 0;
}

