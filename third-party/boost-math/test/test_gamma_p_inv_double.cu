#include "hip/hip_runtime.h"

//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <iostream>
#include <iomanip>
#include <vector>
#include <random>
#include <boost/math/special_functions.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef double float_type;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in1, const float_type *in2, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = boost::math::gamma_p_inv(in1[i], in2[i]);
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector1(numElements);

    // Allocate the managed input vector B
    cuda_managed_ptr<float_type> input_vector2(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    // Initialize the input vectors
    std::mt19937_64 gen(42);
    std::uniform_real_distribution<float_type> dist(0, 1);
    for (int i = 0; i < numElements; ++i)
    {
        input_vector1[i] = dist(gen);
        input_vector2[i] = dist(gen);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;

    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector1.get(), input_vector2.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();

    std::cout << "CUDA kernal done in: " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch CUDA kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
       results.push_back(boost::math::gamma_p_inv(input_vector1[i], input_vector2[i]));
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (boost::math::epsilon_difference(output_vector[i], results[i]) > 1000)
        {
            std::cerr << "Result verification failed at element " << i << "!\n"
                      << "Error found was: " << boost::math::epsilon_difference(output_vector[i], results[i]) << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED, normal calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";

    return 0;
}
