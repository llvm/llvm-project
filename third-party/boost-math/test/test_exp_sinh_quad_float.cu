#include "hip/hip_runtime.h"

//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/math/quadrature/exp_sinh.hpp>
#include <boost/math/special_functions.hpp>
#include <boost/math/tools/precision.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef float float_type;

__host__ __device__ float_type func(float_type x)
{
    BOOST_MATH_STD_USING
    return 1/(1+x*x);
}

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(float_type *out, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float_type tol = boost::math::tools::root_epsilon<float_type>();
    float_type error;
    float_type L1;
    boost::math::size_t levels;

    if (i < numElements)
    {
        out[i] = boost::math::quadrature::exp_sinh_integrate(func, tol, &error, &L1, &levels);
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        input_vector[i] = M_PI * (static_cast<float_type>(i) / numElements);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 512;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;

    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(output_vector.get(), numElements);
    hipDeviceSynchronize();

    std::cout << "CUDA kernal done in: " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    float_type tol = boost::math::tools::root_epsilon<float_type>();
    float_type error;
    float_type L1;
    boost::math::quadrature::exp_sinh<float_type> integrator;
    for(int i = 0; i < numElements; ++i)
    {
       results.push_back(integrator.integrate(func, tol, &error, &L1));
    }
    double t = w.elapsed();
    // check the results
    int failed_count = 0;
    for(int i = 0; i < numElements; ++i)
    {
        const auto eps = boost::math::epsilon_difference(output_vector[i], results[i]);
        if (eps > 10)
        {
            std::cerr   << std::setprecision(std::numeric_limits<float_type>::digits10)
                        << "Result verification failed at element " << i << "!\n"
                        << "Device: " << output_vector[i]
                        << "\n  Host: " << results[i]
                        << "\n   Eps: " << eps << "\n";
            failed_count++;
        }
        if (failed_count > 100)
        {
            break;
        }
    }

    if (failed_count != 0)
    {
        std::cout << "Test FAILED" << std::endl;
        return EXIT_FAILURE;
    }

    std::cout << "Test PASSED, normal calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";

    return 0;
}
