#include "hip/hip_runtime.h"
//  Copyright John Maddock 2016.
//  Copyright Matt Borland 2024.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#define BOOST_MATH_OVERFLOW_ERROR_POLICY ignore_error
#define BOOST_MATH_PROMOTE_DOUBLE_POLICY false

// floating-point value does not fit in required floating-point type
#pragma nv_diag_suppress 221

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/math/special_functions/beta.hpp>
#include <boost/math/special_functions/relative_difference.hpp>
#include <boost/array.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef float float_type;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in1, const float_type *in2, const float_type *in3, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = boost::math::ibetac_inv(in1[i], in2[i], in3[i]);
    }
}

template <class T> struct table_type { typedef T type; };
typedef float_type T;
#define SC_(x) static_cast<T>(x)

#include "ibeta_data.ipp"
#include "ibeta_small_data.ipp"

/**
 * Host main routine
 */
int main(void)
{
  try{
    // Consolidate the test data:
    std::vector<float_type> v1, v2, v3;

    for(unsigned i = 0; i < ibeta_data.size(); ++i)
    {
       v1.push_back(ibeta_data[i][0]);
       v2.push_back(ibeta_data[i][1]);
       v3.push_back(ibeta_data[i][2]);
    }
    for(unsigned i = 0; i < ibeta_small_data.size(); ++i)
    {
       v1.push_back(ibeta_small_data[i][0]);
       v2.push_back(ibeta_small_data[i][1]);
       v3.push_back(ibeta_small_data[i][2]);
    }

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector1(numElements);
    cuda_managed_ptr<float_type> input_vector2(numElements);
    cuda_managed_ptr<float_type> input_vector3(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        int table_id = i % v1.size();
        input_vector1[i] = v1[table_id];
        input_vector2[i] = v2[table_id];
        input_vector3[i] = v3[table_id];
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;

    watch w;
    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector1.get(), input_vector2.get(), input_vector3.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();
    std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
       results.push_back(boost::math::ibetac_inv(input_vector1[i], input_vector2[i], input_vector3[i]));
    double t = w.elapsed();
    bool failed = false;
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (boost::math::isfinite(output_vector[i]))
        {
            if (boost::math::epsilon_difference(output_vector[i], results[i]) > 300)
            {
                std::cerr << "Result verification failed at element " << i << "!" << std::endl;
                std::cerr << "Error rate was: " << boost::math::epsilon_difference(output_vector[i], results[i]) << "eps" << std::endl;
                failed = true;
            }
        }
    }

    if (failed)
    {
        return EXIT_FAILURE;
    }

    std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";
  }
  catch(const std::exception& e)
  {
    std::cerr << "Stopped with exception: " << e.what() << std::endl;
  }
  return 0;
}


