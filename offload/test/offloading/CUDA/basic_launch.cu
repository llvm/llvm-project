// RUN: %clang++ -foffload-via-llvm --offload-arch=native %s -o %t
// RUN: %t | %fcheck-generic
// RUN: %clang++ -foffload-via-llvm --offload-arch=native %s -o %t -fopenmp
// RUN: %t | %fcheck-generic

// UNSUPPORTED: aarch64-unknown-linux-gnu
// UNSUPPORTED: aarch64-unknown-linux-gnu-LTO
// UNSUPPORTED: x86_64-pc-linux-gnu
// UNSUPPORTED: x86_64-pc-linux-gnu-LTO


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" {
void *llvm_omp_target_alloc_shared(size_t Size, int DeviceNum);
void llvm_omp_target_free_shared(void *DevicePtr, int DeviceNum);
}

__global__ void square(int *A) { *A = 42; }

int main(int argc, char **argv) {
  int DevNo = 0;
  int *Ptr = reinterpret_cast<int *>(llvm_omp_target_alloc_shared(4, DevNo));
  *Ptr = 7;
  printf("Ptr %p, *Ptr: %i\n", Ptr, *Ptr);
  // CHECK: Ptr [[Ptr:0x.*]], *Ptr: 7
  square<<<1, 1>>>(Ptr);
  printf("Ptr %p, *Ptr: %i\n", Ptr, *Ptr);
  // CHECK: Ptr [[Ptr]], *Ptr: 42
  llvm_omp_target_free_shared(Ptr, DevNo);
}
