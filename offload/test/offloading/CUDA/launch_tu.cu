// clang-format off
// RUN: %clang++ -foffload-via-llvm --offload-arch=native %s -o %t.launch_tu.o -c
// RUN: %clang++ -foffload-via-llvm --offload-arch=native -x cuda %S/kernel_tu.cu.inc -o %t.kernel_tu.o -c
// RUN: %clang++ -foffload-via-llvm --offload-arch=native %t.launch_tu.o %t.kernel_tu.o -o %t
// RUN: %t | %fcheck-generic
// clang-format on

// UNSUPPORTED: aarch64-unknown-linux-gnu
// UNSUPPORTED: aarch64-unknown-linux-gnu-LTO
// UNSUPPORTED: x86_64-pc-linux-gnu
// UNSUPPORTED: x86_64-pc-linux-gnu-LTO


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" {
void *llvm_omp_target_alloc_shared(size_t Size, int DeviceNum);
void llvm_omp_target_free_shared(void *DevicePtr, int DeviceNum);
}

extern __global__ void square(int *A);

int main(int argc, char **argv) {
  int DevNo = 0;
  int *Ptr = reinterpret_cast<int *>(llvm_omp_target_alloc_shared(4, DevNo));
  *Ptr = 7;
  printf("Ptr %p, *Ptr: %i\n", Ptr, *Ptr);
  // CHECK: Ptr [[Ptr:0x.*]], *Ptr: 7
  square<<<1, 1>>>(Ptr);
  printf("Ptr %p, *Ptr: %i\n", Ptr, *Ptr);
  // CHECK: Ptr [[Ptr]], *Ptr: 42
  llvm_omp_target_free_shared(Ptr, DevNo);
}
